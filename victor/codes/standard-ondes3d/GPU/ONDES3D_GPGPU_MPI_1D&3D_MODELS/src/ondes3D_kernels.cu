#include "hip/hip_runtime.h"
// THIS FILE CONTAINS THE KERNELS FOR ONDES3D

/* VIM FOLDING QUICK HELP {{{ 
 * Des marqueurs sont places dans le code pour le repliage de code (code folding) sous vim.
 *
 * Pour replier / deplier le code sous vim : 
 * set foldmethod=marker (deja dans .vimrc pour le compte cudauser)
 *
 * en mode commande (esc) :
 * 	zo : deplie un niveau
 * 	zc : replie un niveau
 * 	zO : deplie recursivement
 * 	zC : replie recursivement
 * 	zM : replie tout
 * 	zR : deplie tout
 * 	za : replie / deplie
 *
 * David.
 *
 */// }}}

#define DEVICE_SIDE_INCLUDE

#include <stdio.h>
#ifdef EMU_DGN
#include <math.h>
#endif
#include <hip/hip_runtime.h>
#include "ondes3D_kernels.h"

#define MASK_FIRST_X 1
#define MASK_LAST_X 2
#define MASK_FIRST_Y 4
#define MASK_LAST_Y 8
#define DUMMY_VALUE 100

// UTILS {{{
void printCudaErr(hipError_t err, char* where)
{   if (err != hipSuccess) {
        printf("\nError <%s> detected in section %s\n",hipGetErrorString(err),where);fflush(stdout);
    }
}
// }}}

// CONSTANT MEM (FOR 1D MODELS) {{{
__constant__ float c_rho[CONSTANT_MAX_SIZE];
__constant__ float c_vp[CONSTANT_MAX_SIZE];
__constant__ float c_vs[CONSTANT_MAX_SIZE];

#define RHO(z) c_rho[(z)+2]
#define VP(z) c_vp[(z)+2]
#define VS(z) c_vs[(z)+2]
#define LAM(z) (c_vp[(z)+2]*c_vp[(z)+2]*c_rho[(z)+2] - 2.f*c_vs[(z)+2]*c_vs[(z)+2]*c_rho[(z)+2])
#define MU(z) (c_vs[(z)+2]*c_vs[(z)+2]*c_rho[(z)+2])

void setConstRho(float* array, int size)
{
	printCudaErr(hipMemcpyToSymbol(HIP_SYMBOL(c_rho), array, size*sizeof(float)),"__FUNCTION__");
}

void setConstVp(float* array, int size)
{
	printCudaErr(hipMemcpyToSymbol(HIP_SYMBOL(c_vp), array, size*sizeof(float)),"__FUNCTION__");
}

void setConstVs(float* array, int size)
{
	printCudaErr(hipMemcpyToSymbol(HIP_SYMBOL(c_vs), array, size*sizeof(float)),"__FUNCTION__");
}
// }}}

// TEXTURES DECLARATION {{{
// 1D arrays textures
texture<float, 1, hipReadModeElementType> tex_dumpx;
texture<float, 1, hipReadModeElementType> tex_dumpy;
texture<float, 1, hipReadModeElementType> tex_dumpz;
texture<float, 1, hipReadModeElementType> tex_dumpx2;
texture<float, 1, hipReadModeElementType> tex_dumpy2;
texture<float, 1, hipReadModeElementType> tex_dumpz2;

texture<float, 1, hipReadModeElementType> tex_alphax;
texture<float, 1, hipReadModeElementType> tex_alphay;
texture<float, 1, hipReadModeElementType> tex_alphaz;
texture<float, 1, hipReadModeElementType> tex_alphax2;
texture<float, 1, hipReadModeElementType> tex_alphay2;
texture<float, 1, hipReadModeElementType> tex_alphaz2;

texture<float, 1, hipReadModeElementType> tex_kappax;
texture<float, 1, hipReadModeElementType> tex_kappay;
texture<float, 1, hipReadModeElementType> tex_kappaz;
texture<float, 1, hipReadModeElementType> tex_kappax2;
texture<float, 1, hipReadModeElementType> tex_kappay2;
texture<float, 1, hipReadModeElementType> tex_kappaz2;
// }}}

// TEXTURES BINDING {{{

void bindTexturesCpmlVector(	float* d_dumpx, float* d_alphax, float* d_kappax, float* d_dumpx2, float* d_alphax2, float* d_kappax2,
				float* d_dumpy, float* d_alphay, float* d_kappay, float* d_dumpy2, float* d_alphay2, float* d_kappay2,
				float* d_dumpz, float* d_alphaz, float* d_kappaz, float* d_dumpz2, float* d_alphaz2, float* d_kappaz2,
				int size_x, int size_y, int size_z) {

	hipChannelFormatDesc channelDescFloat = hipCreateChannelDesc<float>();
	printCudaErr(hipBindTexture(NULL,tex_dumpx, d_dumpx, channelDescFloat, size_x*sizeof(float)),"hipBindTexture d_dumpx on tex_dumpx");
	printCudaErr(hipBindTexture(NULL,tex_alphax, d_alphax, channelDescFloat, size_x*sizeof(float)),"hipBindTexture d_alphax on tex_alphax");
	printCudaErr(hipBindTexture(NULL,tex_kappax, d_kappax, channelDescFloat, size_x*sizeof(float)),"hipBindTexture d_kappax on tex_kappax");
	printCudaErr(hipBindTexture(NULL,tex_dumpx2, d_dumpx2, channelDescFloat, size_x*sizeof(float)),"hipBindTexture d_dumpx2 on tex_dumpx2");
	printCudaErr(hipBindTexture(NULL,tex_alphax2, d_alphax2, channelDescFloat, size_x*sizeof(float)),"hipBindTexture d_alphax2 on tex_alphax2");
	printCudaErr(hipBindTexture(NULL,tex_kappax2, d_kappax2, channelDescFloat, size_x*sizeof(float)),"hipBindTexture d_kappax2 on tex_kappax2");

	printCudaErr(hipBindTexture(NULL,tex_dumpy, d_dumpy, channelDescFloat, size_y*sizeof(float)),"hipBindTexture d_dumpy on tex_dumpy");
	printCudaErr(hipBindTexture(NULL,tex_alphay, d_alphay, channelDescFloat, size_y*sizeof(float)),"hipBindTexture d_alphay on tex_alphay");
	printCudaErr(hipBindTexture(NULL,tex_kappay, d_kappay, channelDescFloat, size_y*sizeof(float)),"hipBindTexture d_kappay on tex_kappay");
	printCudaErr(hipBindTexture(NULL,tex_dumpy2, d_dumpy2, channelDescFloat, size_y*sizeof(float)),"hipBindTexture d_dumpy2 on tex_dumpy2");
	printCudaErr(hipBindTexture(NULL,tex_alphay2, d_alphay2, channelDescFloat, size_y*sizeof(float)),"hipBindTexture d_alphay2 on tex_alphay2");
	printCudaErr(hipBindTexture(NULL,tex_kappay2, d_kappay2, channelDescFloat, size_y*sizeof(float)),"hipBindTexture d_kappay2 on tex_kappay2");

	printCudaErr(hipBindTexture(NULL,tex_dumpz, d_dumpz, channelDescFloat, size_z*sizeof(float)),"hipBindTexture d_dumpz on tex_dumpz");
	printCudaErr(hipBindTexture(NULL,tex_alphaz, d_alphaz, channelDescFloat, size_z*sizeof(float)),"hipBindTexture d_alphaz on tex_alphaz");
	printCudaErr(hipBindTexture(NULL,tex_kappaz, d_kappaz, channelDescFloat, size_z*sizeof(float)),"hipBindTexture d_kappaz on tex_kappaz");
	printCudaErr(hipBindTexture(NULL,tex_dumpz2, d_dumpz2, channelDescFloat, size_z*sizeof(float)),"hipBindTexture d_dumpz2 on tex_dumpz2");
	printCudaErr(hipBindTexture(NULL,tex_alphaz2, d_alphaz2, channelDescFloat, size_z*sizeof(float)),"hipBindTexture d_alphaz2 on tex_alphaz2");
	printCudaErr(hipBindTexture(NULL,tex_kappaz2, d_kappaz2, channelDescFloat, size_z*sizeof(float)),"hipBindTexture d_kappaz2 on tex_kappaz2");
}
// }}}

// DEVICE FUNCTIONS {{{
__device__ float CPML2 (float vp, float dump, float alpha, float kappa, float phidum, float dx, float dt, float x1, float x2 )
{
  float a, b;

  b = expf ( - ( vp*dump / kappa + alpha ) * dt );
  a = 0.0f;

  if ((vp*dump) > 0.000001f || (vp*dump) < -0.000001f ) 
	a = vp*dump * ( b - 1.0f) / ( kappa * ( vp*dump + kappa * alpha ) );

  return b * phidum + a * ( x2 - x1 ) * (1.f/dx);
}

__device__ float CPML4 (float vp, float dump, float alpha, float kappa, float phidum, float dx, float dt, float x1, float x2, float x3, float x4 )
{
  float a, b;

  b = expf ( - ( vp*dump / kappa + alpha ) * dt );
  a = 0.0f;
  
  if ((vp*dump) > 0.000001f || (vp*dump) < -0.000001f ) 
	a = vp*dump * ( b - 1.0f ) / ( kappa * ( vp*dump + kappa * alpha ) );
	
  return b * phidum + a * ( (9.f/8.f)*( x2 - x1 )/dx - (1.f/24.f)*( x4 - x3 )/dx );
}

__device__ float staggards2 (float lam, float mu, float kappax, float kappay, float kappaz, float dt, float dx, float x1, float x2, float y1, float y2, float z1, float z2 )
{
  return dt*( (lam+2.f*mu)*(x2 - x1)/kappax + lam*(y2 - y1)/kappay + lam*(z2 - z1)/kappaz )/dx;
}

__device__ float staggardt2 (float mu, float kappax, float kappay, float dt, float dx, float x1, float x2, float y1, float y2 )
{
  return dt*mu*( (x2 - x1)/kappax + (y2 - y1)/kappay )/dx;
}

__device__ float staggards4 (float lam, float mu, float kappax, float kappay, float kappaz, float dt, float dx,
	float x1, float x2, float x3, float x4,
	float y1, float y2, float y3, float y4,
	float z1, float z2, float z3, float z4 )
{
  return (9.f*dt/8.f)*( (lam+2.f*mu)*(x2 - x1)/kappax + lam*(y2 - y1)/kappay + lam*(z2 - z1)/kappaz )/dx
        - (dt/24.f)*( (lam+2.f*mu)*(x4 - x3)/kappax + lam*(y4 - y3)/kappay + lam*(z4 - z3)/kappaz )/dx;
}

__device__ float staggardt4 (float mu, float kappax, float kappay, float dt, float dx,
	float x1, float x2, float x3, float x4,
	float y1, float y2, float y3, float y4 )
{
  return (9.f*dt*mu/8.f)*( (x2 - x1)/kappax + (y2 - y1)/kappay )/dx
        - (dt*mu/24.f)*( (x4 - x3)/kappax + (y4 - y3)/kappay )/dx;
}

__device__ float staggardv4 (float b, float kappax, float kappay, float kappaz, float dt, float dx,
	float x1, float x2, float x3, float x4,
	float y1, float y2, float y3, float y4,
	float z1, float z2, float z3, float z4 )
{
  return (9.f*b*dt/8.f)*( (x2 - x1)/kappax + (y2 - y1)/kappay + (z2 - z1)/kappaz )/dx
        - (b*dt/24.f)*( (x4 - x3)/kappax + (y4 - y3)/kappay + (z4 - z3)/kappaz )/dx;
}

__device__ float staggardv2 (float b, float kappax, float kappay, float kappaz, float dt, float dx,
	float x1, float x2,
	float y1, float y2,
	float z1, float z2 )
{
  return b*dt*( (x2 - x1)/kappax + (y2 - y1)/kappay + (z2 - z1)/kappaz )/dx;
}
// }}}

// KERNELS {{{
// COMPUTATIONAL KERNELS {{{
// FOR 1D MODELS {{{
// COMPUTE STRESS {{{
// IMPLEMENTATION {{{
__global__ void compute_stress_1d (	float* d_txx0, float* d_tyy0, float* d_tzz0, float* d_txy0, float* d_txz0, float* d_tyz0,
							float* d_vx0, float* d_vy0, float* d_vz0,
							int* d_npml_tab, float* d_phivxx, float* d_phivxy, float* d_phivxz, float* d_phivyx, float* d_phivyy, float* d_phivyz, float* d_phivzx, float* d_phivzy, float* d_phivzz, 
							int sizex, int sizey, int sizez,
							int pitch_x, int pitch_y, int pitch_z, 
							float ds, float dt, int delta, bool compute_external, int position)
{
	__shared__ float s_vx0[(NPPDX+4)*(NPPDY+4)];
	__shared__ float s_vy0[(NPPDX+4)*(NPPDY+4)];
	__shared__ float s_vz0[(NPPDX+4)*(NPPDY+4)];

	float vx0_m1, vx0_p1, vx0_p2; // vx0 pour k-1, vx0 pour k-2, vx0 pour k+1, vx0 pour k+2
	float vy0_m1, vy0_p1, vy0_p2;
	float vz0_m1, vz0_m2, vz0_p1, vz0_p2;
	
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	int j = blockIdx.y*blockDim.y + threadIdx.y;
	int k = 0;
	int distance_xmin = i;
	int distance_xmax = sizex - i - 1;
	int distance_ymin = j;
	int distance_ymax = sizey - j - 1;
	int distance_zmax = sizez - k - 1;
	int offset, tx;
	bool last_x, last_y;

	// seulement les blocs internes/externes travaillent suivant le parametre compute_external
	// s'il n'y a qu'un point dans le dernier bloc en x ou en y, on en prend deux.
	bool active =	(compute_external && 
				(blockIdx.x == 0 || blockIdx.x >= ((blockDim.x*gridDim.x-sizex == blockDim.x-1)?gridDim.x-2:gridDim.x-1) || blockIdx.y == 0 || blockIdx.y >= ((blockDim.y*gridDim.y-sizey == blockDim.y-1)?gridDim.y-2:gridDim.y-1)))
		||	(!compute_external && 
				!(blockIdx.x == 0 || blockIdx.x >= ((blockDim.x*gridDim.x-sizex == blockDim.x-1)?gridDim.x-2:gridDim.x-1) || blockIdx.y == 0 || blockIdx.y >= ((blockDim.y*gridDim.y-sizey == blockDim.y-1)?gridDim.y-2:gridDim.y-1))); 
	
	//on ne calcule pas les points qui sont en dehors du domaine. Les threads correspondants ne font rien.	
	active = active && ((distance_xmax >=0 && distance_ymax >=0)?true:false);
	last_x = last_y = false;
	if (distance_xmax == 0 || (active && threadIdx.x == (NPPDX-1))) {
		last_x = true;
	}
	if (distance_ymax == 0 || (active && threadIdx.y == (NPPDY-1))) {
		last_y = true;
	}
	// ici distance pour le modele global (on ne s'interesse qu'aux bords, donc si on n'est pas pres du bord, une valeur quelconque >2 suffit)
	distance_xmin = (position & MASK_FIRST_X)?i:DUMMY_VALUE;
	distance_xmax = (position & MASK_LAST_X)?(sizex - i - 1):DUMMY_VALUE;
	distance_ymin = (position & MASK_FIRST_Y)?j:DUMMY_VALUE;
	distance_ymax = (position & MASK_LAST_Y)?(sizey - j - 1):DUMMY_VALUE;

// pour k = 0 -------------------------------------------------------------------------------------------->>>
	// chargement initial des valeurs dans les registres et en memoire partagee
	if (active) {/*{{{*/
		// le tableau est initialis� � z�ro et ces �l�ments ne sont jamais mis � jour
		vx0_m1 = 0.f;
		vy0_m1 = 0.f;
		vz0_m1 = vz0_m2 = 0.f;

		offset = pitch_x*pitch_y + j*pitch_x + i;
		vx0_p1 = d_vx0[offset];
		vy0_p1 = d_vy0[offset];
		vz0_p1 = d_vz0[offset];

		offset += pitch_x*pitch_y;
		vx0_p2 = d_vx0[offset];
		vy0_p2 = d_vy0[offset];
		vz0_p2 = d_vz0[offset];

		// vx0, vy0, vz0
		// chaque thread charge sa valeur en shmem
		tx = (threadIdx.y+2)*(NPPDX+4) + threadIdx.x+2;
		offset = j*pitch_x + i;
		s_vx0[tx] = d_vx0[offset];
		s_vy0[tx] = d_vy0[offset];
		s_vz0[tx] = d_vz0[offset];
		// on charge les deux rang�es i-1 et i-2
		if (threadIdx.x == 0) {
			// i-2
			tx = (threadIdx.y+2)*(NPPDX+4) + 0;
			offset = j*pitch_x + i-2;
			s_vx0[tx] = d_vx0[offset];
			s_vy0[tx] = d_vy0[offset];
			s_vz0[tx] = d_vz0[offset];
			// i-1
			tx = (threadIdx.y+2)*(NPPDX+4) + 1;
			offset = j*pitch_x + i-1;
			s_vx0[tx] = d_vx0[offset];
			s_vy0[tx] = d_vy0[offset];
			s_vz0[tx] = d_vz0[offset];
		}
		// on charge les deux rang�es i+1 et i+2
		if (last_x) {
			// i+1
			tx = (threadIdx.y+2)*(NPPDX+4) + threadIdx.x + 3;
			offset = j*pitch_x + i + 1;
			s_vx0[tx] = d_vx0[offset];
			s_vy0[tx] = d_vy0[offset];
			s_vz0[tx] = d_vz0[offset];
			// i+2
			tx = (threadIdx.y+2)*(NPPDX+4) + threadIdx.x + 4;
			offset = j*pitch_x + i + 2;
			s_vx0[tx] = d_vx0[offset];
			s_vy0[tx] = d_vy0[offset];
			s_vz0[tx] = d_vz0[offset];
		}
		// on charge les rang�es j-1 et j-2
		if (threadIdx.y == 0) {
			// j-2
			tx = threadIdx.x+2;
			offset = (j-2)*pitch_x + i;
			s_vx0[tx] = d_vx0[offset];
			s_vy0[tx] = d_vy0[offset];
			s_vz0[tx] = d_vz0[offset];
			// j-1
			tx = (NPPDX+4) + threadIdx.x+2;
			offset = (j-1)*pitch_x + i;
			s_vx0[tx] = d_vx0[offset];
			s_vy0[tx] = d_vy0[offset];
			s_vz0[tx] = d_vz0[offset];
		}
		// on charge les rang�es j+1 et j+2
		if (last_y) {
			// j+2
			tx = (threadIdx.y + 4)*(NPPDX+4) + threadIdx.x+2;
			offset = (j+2)*pitch_x + i;
			s_vx0[tx] = d_vx0[offset];
			s_vy0[tx] = d_vy0[offset];
			s_vz0[tx] = d_vz0[offset];
			// j+1
			tx = (threadIdx.y + 3)*(NPPDX+4) + threadIdx.x+2;
			offset = (j+1)*pitch_x + i;
			s_vx0[tx] = d_vx0[offset];
			s_vy0[tx] = d_vy0[offset];
			s_vz0[tx] = d_vz0[offset];
		}
	}/*}}}*/
	__syncthreads();
	// calcul
#ifndef NOCPML
	if (active) {/*{{{*/
		// acces coalescent � d_npml_tab
		int npml = d_npml_tab[k*(sizex)*(sizey) + j*(sizex) + i];
		offset = k*pitch_x*pitch_y + j*pitch_x + i;
		/* Calculation of txx, tyy and tzz */
		/* Calculation of txy */
		if ( distance_ymax >= 1 && distance_xmin >= 1 ){
			float muy = MU(k);
			float vpy = VP(k);

			float phixdum =	d_phivyx[npml];
			float phiydum = d_phivxy[npml];
			phixdum = CPML4 (vpy, tex1Dfetch(tex_dumpx, i), tex1Dfetch(tex_alphax, i), tex1Dfetch(tex_kappax, i), phixdum, ds, dt,
			s_vy0[VOFF(-1,0)], s_vy0[VOFF(0,0)],
			s_vy0[VOFF(-2,0)], s_vy0[VOFF(1,0)]);
			phiydum = CPML4 (vpy, tex1Dfetch(tex_dumpy2, j), tex1Dfetch(tex_alphay2, j), tex1Dfetch(tex_kappay2, j), phiydum, ds, dt,
			s_vx0[VOFF(0,0)], s_vx0[VOFF(0,1)],
			s_vx0[VOFF(0,-1)], s_vx0[VOFF(0,2)]);

			d_txy0[offset] += dt*muy*( phixdum + phiydum )
			+ staggardt4 (muy, tex1Dfetch(tex_kappax, i), tex1Dfetch(tex_kappay2, j), dt, ds,
			s_vy0[VOFF(-1,0)], s_vy0[VOFF(0,0)],
			s_vy0[VOFF(-2,0)], s_vy0[VOFF(1,0)],
			s_vx0[VOFF(0,0)], s_vx0[VOFF(0,1)],
			s_vx0[VOFF(0,-1)], s_vx0[VOFF(0,2)]);

			d_phivyx[npml] = phixdum;
			d_phivxy[npml] = phiydum;
		}
		/* Calculation of txz */
		if (distance_xmin >= 1 ){

			float muz = 0.5f*(MU(k) + MU(k+1));
			float vpz = 0.5f*(VP(k) + VP(k+1));
			float phixdum =	d_phivzx[npml];
			float phizdum = d_phivxz[npml];

			phixdum = CPML4 (vpz, tex1Dfetch(tex_dumpx, i), tex1Dfetch(tex_alphax, i), tex1Dfetch(tex_kappax, i), phixdum, ds, dt,
			s_vz0[VOFF(-1,0)], s_vz0[VOFF(0,0)],
			s_vz0[VOFF(-2,0)], s_vz0[VOFF(1,0)]);
			phizdum = CPML4 (vpz, tex1Dfetch(tex_dumpz2, k), tex1Dfetch(tex_alphaz2, k), tex1Dfetch(tex_kappaz2, k), phizdum, ds, dt,
			s_vx0[VOFF(0,0)], vx0_p1,
			vx0_m1, vx0_p2);

			d_txz0[offset] += dt*muz*( phixdum + phizdum )
			+ staggardt4 (muz, tex1Dfetch(tex_kappax, i), tex1Dfetch(tex_kappaz2, k), dt, ds,
			s_vz0[VOFF(-1,0)], s_vz0[VOFF(0,0)],
			s_vz0[VOFF(-2,0)], s_vz0[VOFF(1,0)],
			s_vx0[VOFF(0,0)], vx0_p1,
			vx0_m1, vx0_p2);

			d_phivzx[npml] = phixdum;
			d_phivxz[npml] = phizdum;
		}
		/* Calculation of tyz */
		if (distance_ymax >= 1){
			// (distance_xmax==0)?mu(i,j,k):mu(i+1,j,k);
			float muxyz = 0.125f*(MU(k) + MU(k+1)
						 + MU(k) + MU(k+1)
						 + MU(k) + MU(k+1)
						 + MU(k) + MU(k+1));
			float vpxyz = 0.125f*(VP(k) + VP(k+1)
						 + VP(k) + VP(k+1)
						 + VP(k) + VP(k+1)
						 + VP(k) + VP(k+1));
			float phiydum = d_phivzy[npml];
			float phizdum = d_phivyz[npml];

			phiydum = CPML4 (vpxyz, tex1Dfetch(tex_dumpy2, j), tex1Dfetch(tex_alphay2, j), tex1Dfetch(tex_kappay2, j), phiydum, ds, dt,
			s_vz0[VOFF(0,0)], s_vz0[VOFF(0,1)],
			s_vz0[VOFF(0,-1)], s_vz0[VOFF(0,2)] );
			phizdum = CPML4 (vpxyz, tex1Dfetch(tex_dumpz2, k), tex1Dfetch(tex_alphaz2, k), tex1Dfetch(tex_kappaz2, k), phizdum, ds, dt,
			s_vy0[VOFF(0,0)], vy0_p1,
			vy0_m1, vy0_p2 );

			d_tyz0[offset] += dt*muxyz*( phiydum + phizdum )
			+ staggardt4 (muxyz, tex1Dfetch(tex_kappay2, j), tex1Dfetch(tex_kappaz2, k), dt, ds,
			s_vz0[VOFF(0,0)], s_vz0[VOFF(0,1)],
			s_vz0[VOFF(0,-1)], s_vz0[VOFF(0,2)],
			s_vy0[VOFF(0,0)], vy0_p1,
			vy0_m1, vy0_p2 );
			
			d_phivzy[npml] = phiydum;
			d_phivyz[npml] = phizdum;
		}
	}/*}}}*/
	__syncthreads();
// pour k= 1 a k = delta - 1 (CPML only) ----------------------------------------------------------------->>>
	for (k = 1; k < delta; k++) {/*{{{*/
		// decalage des donnees
		if (active) {/*{{{*/
			// d�calage des donn�es selon l'axe Z
			tx = (threadIdx.y+2)*(NPPDX+4) + threadIdx.x+2;
			vx0_m1 = s_vx0[tx];
			s_vx0[tx] = vx0_p1;
			vx0_p1 = vx0_p2;

			vy0_m1 = s_vy0[tx];
			s_vy0[tx] = vy0_p1;
			vy0_p1 = vy0_p2;

			vz0_m2 = vz0_m1;
			vz0_m1 = s_vz0[tx];
			s_vz0[tx] = vz0_p1;
			vz0_p1 = vz0_p2;

			// remarque : le tableau d�passe de un point ds chaque direction du domaine calcul�, donc si on est au bord du domaine,
			// i+1 est d�fini et la valeur � ce point est nulle (car hors du domaine)
			// i+2 correspond � la valeur pour j+1 et i=-1, cad 0 car hors du domaine => �a reste coh�rent
			// idem pour i - 1 et i - 2 (i-2 toujours d�fini � cause du padding d'alignement)
			// on charge les deux rang�es i-1 et i-2
			if (threadIdx.x == 0) {
				// i-2
				tx = (threadIdx.y+2)*(NPPDX+4) + 0;
				offset = k*pitch_x*pitch_y + j*pitch_x + i-2;
				s_vx0[tx] = d_vx0[offset];
				s_vy0[tx] = d_vy0[offset];
				s_vz0[tx] = d_vz0[offset];
				// i-1
				tx = (threadIdx.y+2)*(NPPDX+4) + 1;
				offset = k*pitch_x*pitch_y + j*pitch_x + i-1;
				s_vx0[tx] = d_vx0[offset];
				s_vy0[tx] = d_vy0[offset];
				s_vz0[tx] = d_vz0[offset];
			}
			// on charge les deux rang�es i+1 et i+2
			if (last_x) {
				// i+1
				tx = (threadIdx.y+2)*(NPPDX+4) + threadIdx.x + 3;
				offset = k*pitch_x*pitch_y + j*pitch_x + i + 1;
				s_vx0[tx] = d_vx0[offset];
				s_vy0[tx] = d_vy0[offset];
				s_vz0[tx] = d_vz0[offset];
				// i+2
				tx = (threadIdx.y+2)*(NPPDX+4) + threadIdx.x + 4;
				offset = k*pitch_x*pitch_y + j*pitch_x + i + 2;
				s_vx0[tx] = d_vx0[offset];
				s_vy0[tx] = d_vy0[offset];
				s_vz0[tx] = d_vz0[offset];
			}
			// on charge les rang�es j-1 et j-2
			if (threadIdx.y == 0) {
				// j-2
				tx = threadIdx.x+2;
				offset = k*pitch_x*pitch_y + (j-2)*pitch_x + i;
				s_vx0[tx] = d_vx0[offset];
				s_vy0[tx] = d_vy0[offset];
				s_vz0[tx] = d_vz0[offset];
				// j-1
				tx = (NPPDX+4) + threadIdx.x+2;
				offset = k*pitch_x*pitch_y + (j-1)*pitch_x + i;
				s_vx0[tx] = d_vx0[offset];
				s_vy0[tx] = d_vy0[offset];
				s_vz0[tx] = d_vz0[offset];
			}
			// on charge les rang�es j+1 et j+2
			if (last_y) {
				// j+2
				tx = (threadIdx.y + 4)*(NPPDX+4) + threadIdx.x+2;
				offset = k*pitch_x*pitch_y + (j+2)*pitch_x + i;
				s_vx0[tx] = d_vx0[offset];
				s_vy0[tx] = d_vy0[offset];
				s_vz0[tx] = d_vz0[offset];
				// j+1
				tx = (threadIdx.y + 3)*(NPPDX+4) + threadIdx.x+2;
				offset = k*pitch_x*pitch_y + (j+1)*pitch_x + i;
				s_vx0[tx] = d_vx0[offset];
				s_vy0[tx] = d_vy0[offset];
				s_vz0[tx] = d_vz0[offset];
			}
			// on charge les points pour k+2 (acces en mem globale)
			if (distance_zmax < 2) {
				vx0_p2 = 0.f;
				vy0_p2 = 0.f;
				vz0_p2 = 0.f;
			} else {
				vx0_p2 = d_vx0[(k+2)*pitch_x*pitch_y + j*pitch_x + i];
				vy0_p2 = d_vy0[(k+2)*pitch_x*pitch_y + j*pitch_x + i];
				vz0_p2 = d_vz0[(k+2)*pitch_x*pitch_y + j*pitch_x + i];
			}
		}/*}}}*/
		__syncthreads();

		offset = k*pitch_x*pitch_y + j*pitch_x + i;
		// calcul
		if (active) {/*{{{*/
			// acces coalescent � d_npml_tab
			int npml = d_npml_tab[k*(sizex)*(sizey) + j*(sizex) + i];
			offset = k*pitch_x*pitch_y + j*pitch_x + i;
			/* Calculation of txx, tyy and tzz */
			if (distance_ymin >= 1 && distance_xmax >= 1 ){
				float mux = MU(k);
				float vpx = VP(k);
				float lamx = LAM(k);
				
				float phixdum =	d_phivxx[npml];
				float phiydum = d_phivyy[npml];
				float phizdum = d_phivzz[npml];

				phixdum = CPML4 (vpx, tex1Dfetch(tex_dumpx2, i), tex1Dfetch(tex_alphax2, i), tex1Dfetch(tex_kappax2, i), phixdum, ds, dt,
				s_vx0[VOFF(0,0)], s_vx0[VOFF(1,0)],
				s_vx0[VOFF(-1,0)], s_vx0[VOFF(2,0)]);
				phiydum = CPML4 (vpx, tex1Dfetch(tex_dumpy, j), tex1Dfetch(tex_alphay, j), tex1Dfetch(tex_kappay, j), phiydum, ds, dt,
				s_vy0[VOFF(0,-1)], s_vy0[VOFF(0,0)],
				s_vy0[VOFF(0,-2)], s_vy0[VOFF(0,1)]);
				phizdum = CPML4 (vpx, tex1Dfetch(tex_dumpz, k), tex1Dfetch(tex_alphaz, k), tex1Dfetch(tex_kappaz, k), phizdum, ds, dt,
				vz0_m1, s_vz0[VOFF(0,0)],
				vz0_m2, vz0_p1);

				d_txx0[offset] += dt*(lamx + 2*mux)*phixdum + dt*lamx*( phiydum + phizdum )
				+ staggards4 (lamx, mux, tex1Dfetch(tex_kappax2, i), tex1Dfetch(tex_kappay, j), tex1Dfetch(tex_kappaz, k), dt, ds,
				s_vx0[VOFF(0,0)], s_vx0[VOFF(1,0)],
				s_vx0[VOFF(-1,0)], s_vx0[VOFF(2,0)],
				s_vy0[VOFF(0,-1)], s_vy0[VOFF(0,0)],
				s_vy0[VOFF(0,-2)], s_vy0[VOFF(0,1)],
				vz0_m1, s_vz0[VOFF(0,0)],
				vz0_m2, vz0_p1);

				d_tyy0[offset] += dt*lamx*( phixdum + phizdum ) + dt*(lamx + 2*mux)*phiydum
				+ staggards4 (lamx, mux, tex1Dfetch(tex_kappay, j), tex1Dfetch(tex_kappax2, i), tex1Dfetch(tex_kappaz, k), dt, ds,
				s_vy0[VOFF(0,-1)], s_vy0[VOFF(0,0)],
				s_vy0[VOFF(0,-2)], s_vy0[VOFF(0,1)],
				s_vx0[VOFF(0,0)], s_vx0[VOFF(1,0)],
				s_vx0[VOFF(-1,0)], s_vx0[VOFF(2,0)],
				vz0_m1, s_vz0[VOFF(0,0)],
				vz0_m2, vz0_p1);

				d_tzz0[offset] += dt*lamx*( phixdum + phiydum ) + dt*(lamx + 2*mux)*phizdum
				+ staggards4 (lamx, mux, tex1Dfetch(tex_kappaz, k), tex1Dfetch(tex_kappax2, i), tex1Dfetch(tex_kappay, j), dt, ds,
				vz0_m1, s_vz0[VOFF(0,0)],
				vz0_m2, vz0_p1,
				s_vx0[VOFF(0,0)], s_vx0[VOFF(1,0)],
				s_vx0[VOFF(-1,0)], s_vx0[VOFF(2,0)],
				s_vy0[VOFF(0,-1)], s_vy0[VOFF(0,0)],
				s_vy0[VOFF(0,-2)], s_vy0[VOFF(0,1)]);

				d_phivxx[npml] = phixdum;
				d_phivyy[npml] = phiydum;
				d_phivzz[npml] = phizdum;
			} // if (distance_ymin >= 1 && distance_xmax <= 1 )
			/* Calculation of txy */
			if ( distance_ymax >= 1 && distance_xmin >= 1 ){
				float muy = MU(k);
				float vpy = VP(k);

				float phixdum =	d_phivyx[npml];
				float phiydum = d_phivxy[npml];

				phixdum = CPML4 (vpy, tex1Dfetch(tex_dumpx, i), tex1Dfetch(tex_alphax, i), tex1Dfetch(tex_kappax, i), phixdum, ds, dt,
				s_vy0[VOFF(-1,0)], s_vy0[VOFF(0,0)],
				s_vy0[VOFF(-2,0)], s_vy0[VOFF(1,0)]);
				phiydum = CPML4 (vpy, tex1Dfetch(tex_dumpy2, j), tex1Dfetch(tex_alphay2, j), tex1Dfetch(tex_kappay2, j), phiydum, ds, dt,
				s_vx0[VOFF(0,0)], s_vx0[VOFF(0,1)],
				s_vx0[VOFF(0,-1)], s_vx0[VOFF(0,2)]);

				d_txy0[offset] += dt*muy*( phixdum + phiydum )
				+ staggardt4 (muy, tex1Dfetch(tex_kappax, i), tex1Dfetch(tex_kappay2, j), dt, ds,
				s_vy0[VOFF(-1,0)], s_vy0[VOFF(0,0)],
				s_vy0[VOFF(-2,0)], s_vy0[VOFF(1,0)],
				s_vx0[VOFF(0,0)], s_vx0[VOFF(0,1)],
				s_vx0[VOFF(0,-1)], s_vx0[VOFF(0,2)]);
				
				d_phivyx[npml] = phixdum;
				d_phivxy[npml] = phiydum;
			}
			/* Calculation of txz */
			if (distance_xmin >= 1 ){

				float muz = 0.5f*(MU(k) + MU(k+1));
				float vpz = 0.5f*(VP(k) + VP(k+1));

				float phixdum =	d_phivzx[npml];
				float phizdum = d_phivxz[npml];

				phixdum = CPML4 (vpz, tex1Dfetch(tex_dumpx, i), tex1Dfetch(tex_alphax, i), tex1Dfetch(tex_kappax, i), phixdum, ds, dt,
				s_vz0[VOFF(-1,0)], s_vz0[VOFF(0,0)],
				s_vz0[VOFF(-2,0)], s_vz0[VOFF(1,0)]);
				phizdum = CPML4 (vpz, tex1Dfetch(tex_dumpz2, k), tex1Dfetch(tex_alphaz2, k), tex1Dfetch(tex_kappaz2, k), phizdum, ds, dt,
				s_vx0[VOFF(0,0)], vx0_p1,
				vx0_m1, vx0_p2);

				d_txz0[offset] += dt*muz*( phixdum + phizdum )
				+ staggardt4 (muz, tex1Dfetch(tex_kappax, i), tex1Dfetch(tex_kappaz2, k), dt, ds,
				s_vz0[VOFF(-1,0)], s_vz0[VOFF(0,0)],
				s_vz0[VOFF(-2,0)], s_vz0[VOFF(1,0)],
				s_vx0[VOFF(0,0)], vx0_p1,
				vx0_m1, vx0_p2);

				d_phivzx[npml] = phixdum;
				d_phivxz[npml] = phizdum;
			}
			/* Calculation of tyz */
			if (distance_ymax >= 1){
				// (distance_xmax==0)?mu(i,j,k):mu(i+1,j,k);
				float muxyz = 0.125f*(MU(k) + MU(k+1)
							 + MU(k) + MU(k+1)
							 + MU(k) + MU(k+1)
							 + MU(k) + MU(k+1));
				float vpxyz = 0.125f*(VP(k) + VP(k+1)
							 + VP(k) + VP(k+1)
							 + VP(k) + VP(k+1)
							 + VP(k) + VP(k+1));
				float phiydum = d_phivzy[npml];
				float phizdum = d_phivyz[npml];

				phiydum = CPML4 (vpxyz, tex1Dfetch(tex_dumpy2, j), tex1Dfetch(tex_alphay2, j), tex1Dfetch(tex_kappay2, j), phiydum, ds, dt,
				s_vz0[VOFF(0,0)], s_vz0[VOFF(0,1)],
				s_vz0[VOFF(0,-1)], s_vz0[VOFF(0,2)] );
				phizdum = CPML4 (vpxyz, tex1Dfetch(tex_dumpz2, k), tex1Dfetch(tex_alphaz2, k), tex1Dfetch(tex_kappaz2, k), phizdum, ds, dt,
				s_vy0[VOFF(0,0)], vy0_p1,
				vy0_m1, vy0_p2 );

				d_tyz0[offset] += dt*muxyz*( phiydum + phizdum )
				+ staggardt4 (muxyz, tex1Dfetch(tex_kappay2, j), tex1Dfetch(tex_kappaz2, k), dt, ds,
				s_vz0[VOFF(0,0)], s_vz0[VOFF(0,1)],
				s_vz0[VOFF(0,-1)], s_vz0[VOFF(0,2)],
				s_vy0[VOFF(0,0)], vy0_p1,
				vy0_m1, vy0_p2 );
				
				d_phivzy[npml] = phiydum;
				d_phivyz[npml] = phizdum;
			}
		}/*}}}*/
		__syncthreads();
	}/*}}}*/
#endif

// pour k= delta a k = sizez - 3 (CPML + ordre 4) ---------------------------------------------------->>>
	for (k = delta ; k < sizez - 2; k++) {/*{{{*/
		// decalage des donnees
		if (active && k>0) {/*{{{*/
			// d�calage des donn�es selon l'axe Z
			tx = (threadIdx.y+2)*(NPPDX+4) + threadIdx.x+2;
			vx0_m1 = s_vx0[tx];
			s_vx0[tx] = vx0_p1;
			vx0_p1 = vx0_p2;

			vy0_m1 = s_vy0[tx];
			s_vy0[tx] = vy0_p1;
			vy0_p1 = vy0_p2;

			vz0_m2 = vz0_m1;
			vz0_m1 = s_vz0[tx];
			s_vz0[tx] = vz0_p1;
			vz0_p1 = vz0_p2;

			// remarque : le tableau d�passe de un point ds chaque direction du domaine calcul�, donc si on est au bord du domaine,
			// i+1 est d�fini et la valeur � ce point est nulle (car hors du domaine)
			// i+2 correspond � la valeur pour j+1 et i=-1, cad 0 car hors du domaine => �a reste coh�rent
			// idem pour i - 1 et i - 2 (i-2 toujours d�fini � cause du padding d'alignement)
			// on charge les deux rang�es i-1 et i-2
			if (threadIdx.x == 0) {
				// i-2
				tx = (threadIdx.y+2)*(NPPDX+4) + 0;
				offset = k*pitch_x*pitch_y + j*pitch_x + i-2;
				s_vx0[tx] = d_vx0[offset];
				s_vy0[tx] = d_vy0[offset];
				s_vz0[tx] = d_vz0[offset];
				// i-1
				tx = (threadIdx.y+2)*(NPPDX+4) + 1;
				offset = k*pitch_x*pitch_y + j*pitch_x + i-1;
				s_vx0[tx] = d_vx0[offset];
				s_vy0[tx] = d_vy0[offset];
				s_vz0[tx] = d_vz0[offset];
			}
			// on charge les deux rang�es i+1 et i+2
			if (last_x) {
				// i+1
				tx = (threadIdx.y+2)*(NPPDX+4) + threadIdx.x + 3;
				offset = k*pitch_x*pitch_y + j*pitch_x + i + 1;
				s_vx0[tx] = d_vx0[offset];
				s_vy0[tx] = d_vy0[offset];
				s_vz0[tx] = d_vz0[offset];
				// i+2
				tx = (threadIdx.y+2)*(NPPDX+4) + threadIdx.x + 4;
				offset = k*pitch_x*pitch_y + j*pitch_x + i + 2;
				s_vx0[tx] = d_vx0[offset];
				s_vy0[tx] = d_vy0[offset];
				s_vz0[tx] = d_vz0[offset];
			}
			// on charge les rang�es j-1 et j-2
			if (threadIdx.y == 0) {
				// j-2
				tx = threadIdx.x+2;
				offset = k*pitch_x*pitch_y + (j-2)*pitch_x + i;
				s_vx0[tx] = d_vx0[offset];
				s_vy0[tx] = d_vy0[offset];
				s_vz0[tx] = d_vz0[offset];
				// j-1
				tx = (NPPDX+4) + threadIdx.x+2;
				offset = k*pitch_x*pitch_y + (j-1)*pitch_x + i;
				s_vx0[tx] = d_vx0[offset];
				s_vy0[tx] = d_vy0[offset];
				s_vz0[tx] = d_vz0[offset];
			}
			// on charge les rang�es j+1 et j+2
			if (last_y) {
				// j+2
				tx = (threadIdx.y + 4)*(NPPDX+4) + threadIdx.x+2;
				offset = k*pitch_x*pitch_y + (j+2)*pitch_x + i;
				s_vx0[tx] = d_vx0[offset];
				s_vy0[tx] = d_vy0[offset];
				s_vz0[tx] = d_vz0[offset];
				// j+1
				tx = (threadIdx.y + 3)*(NPPDX+4) + threadIdx.x+2;
				offset = k*pitch_x*pitch_y + (j+1)*pitch_x + i;
				s_vx0[tx] = d_vx0[offset];
				s_vy0[tx] = d_vy0[offset];
				s_vz0[tx] = d_vz0[offset];
			}
			// on charge les points pour k+2 (acces en mem globale)
			if (distance_zmax < 2) {
				vx0_p2 = 0.f;
				vy0_p2 = 0.f;
				vz0_p2 = 0.f;
			} else {
				vx0_p2 = d_vx0[(k+2)*pitch_x*pitch_y + j*pitch_x + i];
				vy0_p2 = d_vy0[(k+2)*pitch_x*pitch_y + j*pitch_x + i];
				vz0_p2 = d_vz0[(k+2)*pitch_x*pitch_y + j*pitch_x + i];
			}
		}/*}}}*/
		__syncthreads();
		
		offset = k*pitch_x*pitch_y + j*pitch_x + i;
		// calcul
		if (active) {/*{{{*/
			// plus couteux que le test sur les bords -> a revoir un jour
			int npml = d_npml_tab[k*(sizex)*(sizey) + j*(sizex) + i];
			if ( npml >= 0){/*{{{*/
				// acces coalescent � d_npml_tab
				/* Calculation of txx, tyy and tzz */
				if (distance_ymin >= 1 && distance_xmax >= 1 ){
					float mux = 0.5f*(MU(k) + MU(k));
					float vpx = 0.5f*(VP(k) + VP(k));
					float lamx = 0.5f*(LAM(k) + LAM(k));
					float phixdum =	d_phivxx[npml];
					float phiydum = d_phivyy[npml];
					float phizdum = d_phivzz[npml];

					phixdum = CPML4 (vpx, tex1Dfetch(tex_dumpx2, i), tex1Dfetch(tex_alphax2, i), tex1Dfetch(tex_kappax2, i), phixdum, ds, dt,
					s_vx0[VOFF(0,0)], s_vx0[VOFF(1,0)],
					s_vx0[VOFF(-1,0)], s_vx0[VOFF(2,0)]);
					phiydum = CPML4 (vpx, tex1Dfetch(tex_dumpy, j), tex1Dfetch(tex_alphay, j), tex1Dfetch(tex_kappay, j), phiydum, ds, dt,
					s_vy0[VOFF(0,-1)], s_vy0[VOFF(0,0)],
					s_vy0[VOFF(0,-2)], s_vy0[VOFF(0,1)]);
					phizdum = CPML4 (vpx, tex1Dfetch(tex_dumpz, k), tex1Dfetch(tex_alphaz, k), tex1Dfetch(tex_kappaz, k), phizdum, ds, dt,
					vz0_m1, s_vz0[VOFF(0,0)],
					vz0_m2, vz0_p1);

					d_txx0[offset] += dt*(lamx + 2.0f*mux)*phixdum + dt*lamx*( phiydum + phizdum )
					+ staggards4 (lamx, mux, tex1Dfetch(tex_kappax2, i), tex1Dfetch(tex_kappay, j), tex1Dfetch(tex_kappaz, k), dt, ds,
					s_vx0[VOFF(0,0)], s_vx0[VOFF(1,0)],
					s_vx0[VOFF(-1,0)], s_vx0[VOFF(2,0)],
					s_vy0[VOFF(0,-1)], s_vy0[VOFF(0,0)],
					s_vy0[VOFF(0,-2)], s_vy0[VOFF(0,1)],
					vz0_m1, s_vz0[VOFF(0,0)],
					vz0_m2, vz0_p1);

					d_tyy0[offset] += dt*lamx*( phixdum + phizdum ) + dt*(lamx + 2*mux)*phiydum
					+ staggards4 (lamx, mux, tex1Dfetch(tex_kappay, j), tex1Dfetch(tex_kappax2, i), tex1Dfetch(tex_kappaz, k), dt, ds,
					s_vy0[VOFF(0,-1)], s_vy0[VOFF(0,0)],
					s_vy0[VOFF(0,-2)], s_vy0[VOFF(0,1)],
					s_vx0[VOFF(0,0)], s_vx0[VOFF(1,0)],
					s_vx0[VOFF(-1,0)], s_vx0[VOFF(2,0)],
					vz0_m1, s_vz0[VOFF(0,0)],
					vz0_m2, vz0_p1);

					d_tzz0[offset] += dt*lamx*( phixdum + phiydum ) + dt*(lamx + 2*mux)*phizdum
					+ staggards4 (lamx, mux, tex1Dfetch(tex_kappaz, k), tex1Dfetch(tex_kappax2, i), tex1Dfetch(tex_kappay, j), dt, ds,
					vz0_m1, s_vz0[VOFF(0,0)],
					vz0_m2, vz0_p1,
					s_vx0[VOFF(0,0)], s_vx0[VOFF(1,0)],
					s_vx0[VOFF(-1,0)], s_vx0[VOFF(2,0)],
					s_vy0[VOFF(0,-1)], s_vy0[VOFF(0,0)],
					s_vy0[VOFF(0,-2)], s_vy0[VOFF(0,1)]);

					d_phivxx[npml] = phixdum;
					d_phivyy[npml] = phiydum;
					d_phivzz[npml] = phizdum;

				} // if ( distance_zmin >= 1 && distance_ymin >= 1 && distance_xmax <= 1 )
				/* Calculation of txy */
				if ( distance_ymax >= 1 && distance_xmin >= 1 ){
					float muy = MU(k);
					float vpy = VP(k);

					float phixdum =	d_phivyx[npml];
					float phiydum = d_phivxy[npml];

					phixdum = CPML4 (vpy, tex1Dfetch(tex_dumpx, i), tex1Dfetch(tex_alphax, i), tex1Dfetch(tex_kappax, i), phixdum, ds, dt,
					s_vy0[VOFF(-1,0)], s_vy0[VOFF(0,0)],
					s_vy0[VOFF(-2,0)], s_vy0[VOFF(1,0)]);
					phiydum = CPML4 (vpy, tex1Dfetch(tex_dumpy2, j), tex1Dfetch(tex_alphay2, j), tex1Dfetch(tex_kappay2, j), phiydum, ds, dt,
					s_vx0[VOFF(0,0)], s_vx0[VOFF(0,1)],
					s_vx0[VOFF(0,-1)], s_vx0[VOFF(0,2)]);

					d_txy0[offset] += dt*muy*( phixdum + phiydum )
					+ staggardt4 (muy, tex1Dfetch(tex_kappax, i), tex1Dfetch(tex_kappay2, j), dt, ds,
					s_vy0[VOFF(-1,0)], s_vy0[VOFF(0,0)],
					s_vy0[VOFF(-2,0)], s_vy0[VOFF(1,0)],
					s_vx0[VOFF(0,0)], s_vx0[VOFF(0,1)],
					s_vx0[VOFF(0,-1)], s_vx0[VOFF(0,2)]);
					
					d_phivyx[npml] = phixdum;
					d_phivxy[npml] = phiydum;
				}
				/* Calculation of txz */
				if (distance_xmin >= 1 ){

					float muz = 0.5f*(MU(k) + MU(k+1));
					float vpz = 0.5f*(VP(k) + VP(k+1));

					float phixdum =	d_phivzx[npml];
					float phizdum = d_phivxz[npml];

					phixdum = CPML4 (vpz, tex1Dfetch(tex_dumpx, i), tex1Dfetch(tex_alphax, i), tex1Dfetch(tex_kappax, i), phixdum, ds, dt,
					s_vz0[VOFF(-1,0)], s_vz0[VOFF(0,0)],
					s_vz0[VOFF(-2,0)], s_vz0[VOFF(1,0)]);
					phizdum = CPML4 (vpz, tex1Dfetch(tex_dumpz2, k), tex1Dfetch(tex_alphaz2, k), tex1Dfetch(tex_kappaz2, k), phizdum, ds, dt,
					s_vx0[VOFF(0,0)], vx0_p1,
					vx0_m1, vx0_p2);

					d_txz0[offset] += dt*muz*( phixdum + phizdum )
					+ staggardt4 (muz, tex1Dfetch(tex_kappax, i), tex1Dfetch(tex_kappaz2, k), dt, ds,
					s_vz0[VOFF(-1,0)], s_vz0[VOFF(0,0)],
					s_vz0[VOFF(-2,0)], s_vz0[VOFF(1,0)],
					s_vx0[VOFF(0,0)], vx0_p1,
					vx0_m1, vx0_p2);

					d_phivzx[npml] = phixdum;
					d_phivxz[npml] = phizdum;
				}
				/* Calculation of tyz */
				if (distance_ymax >= 1){
					// (distance_xmax==0)?mu(i,j,k):mu(i+1,j,k);
					float muxyz = 0.125f*(MU(k) + MU(k+1)
								 + MU(k) + MU(k+1)
								 + MU(k) + MU(k+1)
								 + MU(k) + MU(k+1));
					float vpxyz = 0.125f*(VP(k) + VP(k+1)
								 + VP(k) + VP(k+1)
								 + VP(k) + VP(k+1)
								 + VP(k) + VP(k+1));
					float phiydum = d_phivzy[npml];
					float phizdum = d_phivyz[npml];

					phiydum = CPML4 (vpxyz, tex1Dfetch(tex_dumpy2, j), tex1Dfetch(tex_alphay2, j), tex1Dfetch(tex_kappay2, j), phiydum, ds, dt,
					s_vz0[VOFF(0,0)], s_vz0[VOFF(0,1)],
					s_vz0[VOFF(0,-1)], s_vz0[VOFF(0,2)] );
					phizdum = CPML4 (vpxyz, tex1Dfetch(tex_dumpz2, k), tex1Dfetch(tex_alphaz2, k), tex1Dfetch(tex_kappaz2, k), phizdum, ds, dt,
					s_vy0[VOFF(0,0)], vy0_p1,
					vy0_m1, vy0_p2 );

					d_tyz0[offset] += dt*muxyz*( phiydum + phizdum )
					+ staggardt4 (muxyz, tex1Dfetch(tex_kappay2, j), tex1Dfetch(tex_kappaz2, k), dt, ds,
					s_vz0[VOFF(0,0)], s_vz0[VOFF(0,1)],
					s_vz0[VOFF(0,-1)], s_vz0[VOFF(0,2)],
					s_vy0[VOFF(0,0)], vy0_p1,
					vy0_m1, vy0_p2 );

					
					d_phivzy[npml] = phiydum;
					d_phivyz[npml] = phizdum;
				}
			} else {/*}}}*/
				float mux = MU(k);
				float lamx = LAM(k);
				float muy = MU(k);
				float muz = 0.5f*(MU(k) + MU(k+1));
				float muxyz = 0.125f*(MU(k) + MU(k+1)
						   + MU(k) + MU(k+1)
						   + MU(k) + MU(k+1)
						   + MU(k) + MU(k+1));

				d_txx0[offset] += (9.f*dt/8.f)*( (lamx+2.f*mux)*(s_vx0[VOFF(1,0)] - s_vx0[VOFF(0,0)]) + lamx*(s_vy0[VOFF(0,0)] - s_vy0[VOFF(0,-1)]) + lamx*(s_vz0[VOFF(0,0)] - vz0_m1) )/ds - (dt/24.f)*( (lamx+2.f*mux)*(s_vx0[VOFF(2,0)] - s_vx0[VOFF(-1,0)]) + lamx*(s_vy0[VOFF(0,1)] - s_vy0[VOFF(0,-2)]) + lamx*(vz0_p1 - vz0_m2) )/ds;
				d_tyy0[offset] += (9.f*dt/8.f)*( (lamx+2.f*mux)*(s_vy0[VOFF(0,0)] - s_vy0[VOFF(0,-1)]) + lamx*(s_vx0[VOFF(1,0)] - s_vx0[VOFF(0,0)]) + lamx*(s_vz0[VOFF(0,0)] - vz0_m1) )/ds - (dt/24.f)*( (lamx+2.f*mux)*(s_vy0[VOFF(0,1)] - s_vy0[VOFF(0,-2)]) + lamx*(s_vx0[VOFF(2,0)] - s_vx0[VOFF(-1,0)]) + lamx*(vz0_p1 - vz0_m2))/ds;
				d_tzz0[offset] += (9.f*dt/8.f)*( (lamx+2.f*mux)*(s_vz0[VOFF(0,0)] - vz0_m1) + lamx*(s_vx0[VOFF(1,0)] - s_vx0[VOFF(0,0)]) + lamx*(s_vy0[VOFF(0,0)] - s_vy0[VOFF(0,-1)]) )/ds - (dt/24.f)*( (lamx+2.f*mux)*(vz0_p1 - vz0_m2) + lamx*(s_vx0[VOFF(2,0)] - s_vx0[VOFF(-1,0)]) + lamx*(s_vy0[VOFF(0,1)] - s_vy0[VOFF(0,-2)]))/ds;
				
				d_txy0[offset] += (9.f*dt*muy/8.f)*((s_vy0[VOFF(0,0)]-s_vy0[VOFF(-1,0)]) + (s_vx0[VOFF(0,1)] - s_vx0[VOFF(0,0)]))/ds - (dt*muy/24.f)*((s_vy0[VOFF(1,0)] - s_vy0[VOFF(-2,0)]) + (s_vx0[VOFF(0,2)]  - s_vx0[VOFF(0,-1)]))/ds;
				d_txz0[offset] += (9.f*dt*muz/8.f)*((s_vz0[VOFF(0,0)]-s_vz0[VOFF(-1,0)]) + (vx0_p1 - s_vx0[VOFF(0,0)]))/ds - (dt*muz/24.f)*((s_vz0[VOFF(1,0)] - s_vz0[VOFF(-2,0)]) + (vx0_p2  - vx0_m1))/ds;
				d_tyz0[offset] += (9.f*dt*muxyz/8.f)*((s_vz0[VOFF(0,1)]-s_vz0[VOFF(0,0)]) + (vy0_p1 - s_vy0[VOFF(0,0)]))/ds - (dt*muxyz/24.f)*((s_vz0[VOFF(0,2)] - s_vz0[VOFF(0,-1)]) + (vy0_p2  - vy0_m1))/ds;
			}
		}/*}}}*/
		// synchro avant de glisser la fen�tre
		__syncthreads();
	}/*}}}*/

// pour k = sizez - 2 (distance_zmax == 1, CPML + ordre 2) ----------------------------------------------->>>
	k = sizez - 2;
	// decalage des donnees
	if (active) {/*{{{*/
		// d�calage des donn�es selon l'axe Z
		tx = (threadIdx.y+2)*(NPPDX+4) + threadIdx.x+2;
		vx0_m1 = s_vx0[tx];
		s_vx0[tx] = vx0_p1;
		vx0_p1 = vx0_p2;

		vy0_m1 = s_vy0[tx];
		s_vy0[tx] = vy0_p1;
		vy0_p1 = vy0_p2;

		vz0_m2 = vz0_m1;
		vz0_m1 = s_vz0[tx];
		s_vz0[tx] = vz0_p1;
		vz0_p1 = vz0_p2;

		// remarque : le tableau d�passe de un point ds chaque direction du domaine calcul�, donc si on est au bord du domaine,
		// i+1 est d�fini et la valeur � ce point est nulle (car hors du domaine)
		// i+2 correspond � la valeur pour j+1 et i=-1, cad 0 car hors du domaine => �a reste coh�rent
		// idem pour i - 1 et i - 2 (i-2 toujours d�fini � cause du padding d'alignement)
		// on charge les deux rang�es i-1 et i-2
		if (threadIdx.x == 0) {
			// i-2
			tx = (threadIdx.y+2)*(NPPDX+4) + 0;
			offset = k*pitch_x*pitch_y + j*pitch_x + i-2;
			s_vx0[tx] = d_vx0[offset];
			s_vy0[tx] = d_vy0[offset];
			s_vz0[tx] = d_vz0[offset];
			// i-1
			tx = (threadIdx.y+2)*(NPPDX+4) + 1;
			offset = k*pitch_x*pitch_y + j*pitch_x + i-1;
			s_vx0[tx] = d_vx0[offset];
			s_vy0[tx] = d_vy0[offset];
			s_vz0[tx] = d_vz0[offset];
		}
		// on charge les deux rang�es i+1 et i+2
		if (last_x) {
			// i+1
			tx = (threadIdx.y+2)*(NPPDX+4) + threadIdx.x + 3;
			offset = k*pitch_x*pitch_y + j*pitch_x + i + 1;
			s_vx0[tx] = d_vx0[offset];
			s_vy0[tx] = d_vy0[offset];
			s_vz0[tx] = d_vz0[offset];
			// i+2
			tx = (threadIdx.y+2)*(NPPDX+4) + threadIdx.x + 4;
			offset = k*pitch_x*pitch_y + j*pitch_x + i + 2;
			s_vx0[tx] = d_vx0[offset];
			s_vy0[tx] = d_vy0[offset];
			s_vz0[tx] = d_vz0[offset];
		}
		// on charge les rang�es j-1 et j-2
		if (threadIdx.y == 0) {
			// j-2
			tx = threadIdx.x+2;
			offset = k*pitch_x*pitch_y + (j-2)*pitch_x + i;
			s_vx0[tx] = d_vx0[offset];
			s_vy0[tx] = d_vy0[offset];
			s_vz0[tx] = d_vz0[offset];
			// j-1
			tx = (NPPDX+4) + threadIdx.x+2;
			offset = k*pitch_x*pitch_y + (j-1)*pitch_x + i;
			s_vx0[tx] = d_vx0[offset];
			s_vy0[tx] = d_vy0[offset];
			s_vz0[tx] = d_vz0[offset];
		}
		// on charge les rang�es j+1 et j+2
		if (last_y) {
			// j+2
			tx = (threadIdx.y + 4)*(NPPDX+4) + threadIdx.x+2;
			offset = k*pitch_x*pitch_y + (j+2)*pitch_x + i;
			s_vx0[tx] = d_vx0[offset];
			s_vy0[tx] = d_vy0[offset];
			s_vz0[tx] = d_vz0[offset];
			// j+1
			tx = (threadIdx.y + 3)*(NPPDX+4) + threadIdx.x+2;
			offset = k*pitch_x*pitch_y + (j+1)*pitch_x + i;
			s_vx0[tx] = d_vx0[offset];
			s_vy0[tx] = d_vy0[offset];
			s_vz0[tx] = d_vz0[offset];
		}
		// on charge les points pour k+2 (acces en mem globale)
		if (distance_zmax < 2) {
			vx0_p2 = 0.f;
			vy0_p2 = 0.f;
			vz0_p2 = 0.f;
		} else {
			vx0_p2 = d_vx0[(k+2)*pitch_x*pitch_y + j*pitch_x + i];
			vy0_p2 = d_vy0[(k+2)*pitch_x*pitch_y + j*pitch_x + i];
			vz0_p2 = d_vz0[(k+2)*pitch_x*pitch_y + j*pitch_x + i];
		}
	}/*}}}*/
	__syncthreads();
	offset = k*pitch_x*pitch_y + j*pitch_x + i;
	// calcul
	if (active) {/*{{{*/
		int npml = d_npml_tab[k*(sizex)*(sizey) + j*(sizex) + i];
		if ( npml >= 0){/*{{{*/
			/* Calculation of txx, tyy and tzz */
			if (distance_ymin >= 1 && distance_xmax >= 1 ){
				float mux = MU(k);
				float vpx = VP(k);
				float lamx = LAM(k);
				
				float phixdum =	d_phivxx[npml];
				float phiydum = d_phivyy[npml];
				float phizdum = d_phivzz[npml];

				phixdum = CPML2 (vpx, tex1Dfetch(tex_dumpx2, i), tex1Dfetch(tex_alphax2, i), tex1Dfetch(tex_kappax2, i), phixdum, ds, dt, s_vx0[VOFF(0,0)], s_vx0[VOFF(1,0)]);
				phiydum = CPML2 (vpx, tex1Dfetch(tex_dumpy, j), tex1Dfetch(tex_alphay, j), tex1Dfetch(tex_kappay, j), phiydum, ds, dt, s_vy0[VOFF(0,-1)], s_vy0[VOFF(0,0)]);
				phizdum = CPML2 (vpx, tex1Dfetch(tex_dumpz, k), tex1Dfetch(tex_alphaz, k), tex1Dfetch(tex_kappaz, k), phizdum, ds, dt, vz0_m1, s_vz0[VOFF(0,0)]);

				d_txx0[offset] += dt*(lamx + 2.0f*mux)*phixdum + dt*lamx*( phiydum + phizdum )
				+ staggards2 (lamx, mux, tex1Dfetch(tex_kappax2, i), tex1Dfetch(tex_kappay, j), tex1Dfetch(tex_kappaz, k), dt, ds,
				s_vx0[VOFF(0,0)], s_vx0[VOFF(1,0)],
				s_vy0[VOFF(0,-1)], s_vy0[VOFF(0,0)], 
				vz0_m1, s_vz0[VOFF(0,0)]);
				
				d_tyy0[offset] += dt*lamx*( phixdum + phizdum ) + dt*(lamx + 2*mux)*phiydum
				+ staggards2 (lamx, mux, tex1Dfetch(tex_kappay, j), tex1Dfetch(tex_kappax2, i), tex1Dfetch(tex_kappaz, k), dt, ds,
				s_vy0[VOFF(0,-1)], s_vy0[VOFF(0,0)],
				s_vx0[VOFF(0,0)], s_vx0[VOFF(1,0)],
				vz0_m1, s_vz0[VOFF(0,0)]);

				d_tzz0[offset] += dt*lamx*( phixdum + phiydum ) + dt*(lamx + 2*mux)*phizdum
				+ staggards2 (lamx, mux, tex1Dfetch(tex_kappaz, k), tex1Dfetch(tex_kappax2, i), tex1Dfetch(tex_kappay, j), dt, ds,
				vz0_m1, s_vz0[VOFF(0,0)],
				s_vx0[VOFF(0,0)], s_vx0[VOFF(1,0)],
				s_vy0[VOFF(0,-1)], s_vy0[VOFF(0,0)]);
				
				d_phivxx[npml] = phixdum;
				d_phivyy[npml] = phiydum;
				d_phivzz[npml] = phizdum;
			} // if ( distance_zmin >= 1 && distance_ymin >= 1 && distance_xmax <= 1 )
			/* Calculation of txy */
			if ( distance_ymax >= 1 && distance_xmin >= 1 ){
				float muy = MU(k);
				float vpy = VP(k);

				float phixdum =	d_phivyx[npml];
				float phiydum = d_phivxy[npml];

				phixdum = CPML2 (vpy, tex1Dfetch(tex_dumpx, i), tex1Dfetch(tex_alphax, i), tex1Dfetch(tex_kappax, i), phixdum, ds, dt,
				s_vy0[VOFF(-1,0)], s_vy0[VOFF(0,0)]);

				phiydum = CPML2 (vpy, tex1Dfetch(tex_dumpy2, j), tex1Dfetch(tex_alphay2, j), tex1Dfetch(tex_kappay2, j), phiydum, ds, dt,
				s_vx0[VOFF(0,0)], s_vx0[VOFF(0,1)]);
				
				d_txy0[offset] += dt*muy*( phixdum + phiydum )
				+ staggardt2 (muy, tex1Dfetch(tex_kappax, i), tex1Dfetch(tex_kappay2, j), dt, ds,
				s_vy0[VOFF(-1,0)], s_vy0[VOFF(0,0)],
				s_vx0[VOFF(0,0)], s_vx0[VOFF(0,1)]);

				d_phivyx[npml] = phixdum;
				d_phivxy[npml] = phiydum;
			}
			/* Calculation of txz */
			if (distance_xmin >= 1 ){

				float muz = 0.5f*(MU(k) + MU(k+1));
				float vpz = 0.5f*(VP(k) + VP(k+1));

				float phixdum =	d_phivzx[npml];
				float phizdum = d_phivxz[npml];

				phixdum = CPML2 (vpz, tex1Dfetch(tex_dumpx, i), tex1Dfetch(tex_alphax, i), tex1Dfetch(tex_kappax, i), phixdum, ds, dt,
				s_vz0[VOFF(-1,0)], s_vz0[VOFF(0,0)]);
				phizdum = CPML2 (vpz, tex1Dfetch(tex_dumpz2, k), tex1Dfetch(tex_alphaz2, k), tex1Dfetch(tex_kappaz2, k), phizdum, ds, dt,
				s_vx0[VOFF(0,0)], vx0_p1 );

				d_txz0[offset] += dt*muz*( phixdum + phizdum )
				+ staggardt2 (muz, tex1Dfetch(tex_kappax, i), tex1Dfetch(tex_kappaz2, k), dt, ds,
				s_vz0[VOFF(-1,0)], s_vz0[VOFF(0,0)],
				s_vx0[VOFF(0,0)], vx0_p1);

				d_phivzx[npml] = phixdum;
				d_phivxz[npml] = phizdum;
			}
			/* Calculation of tyz */
			if (distance_ymax >= 1){
				// (distance_xmax==0)?mu(i,j,k):mu(i+1,j,k);
				float muxyz = 0.125f*(MU(k) + MU(k+1)
							 + MU(k) + MU(k+1)
							 + MU(k) + MU(k+1)
							 + MU(k) + MU(k+1));
				float vpxyz = 0.125f*(VP(k) + VP(k+1)
							 + VP(k) + VP(k+1)
							 + VP(k) + VP(k+1)
							 + VP(k) + VP(k+1));
				float phiydum = d_phivzy[npml];
				float phizdum = d_phivyz[npml];

				phiydum = CPML2 (vpxyz, tex1Dfetch(tex_dumpy2, j), tex1Dfetch(tex_alphay2, j), tex1Dfetch(tex_kappay2, j), phiydum, ds, dt,
				s_vz0[VOFF(0,0)], s_vz0[VOFF(0,1)] );
				phizdum = CPML2 (vpxyz, tex1Dfetch(tex_dumpz2, k), tex1Dfetch(tex_alphaz2, k), tex1Dfetch(tex_kappaz2, k), phizdum, ds, dt,
				s_vy0[VOFF(0,0)], vy0_p1 );

				d_tyz0[offset] += dt*muxyz*( phiydum + phizdum )
				+ staggardt2 (muxyz, tex1Dfetch(tex_kappay2, j), tex1Dfetch(tex_kappaz2, k), dt, ds,
				s_vz0[VOFF(0,0)], s_vz0[VOFF(0,1)],
				s_vy0[VOFF(0,0)], vy0_p1 );
				
				d_phivzy[npml] = phiydum;
				d_phivyz[npml] = phizdum;
			}
		} else {/*}}}*/
			float mux = MU(k);
			float lamx = LAM(k);
			float muy = MU(k);
			float muz = 0.5f*(MU(k) + MU(k+1));
			float muxyz = 0.125f*(MU(k) + MU(k+1)
					   + MU(k) + MU(k+1)
					   + MU(k) + MU(k+1)
					   + MU(k) + MU(k+1));

			d_txx0[offset] += staggards2 (lamx, mux, 1.f, 1.f, 1.f, dt, ds,
			s_vx0[VOFF(0,0)], s_vx0[VOFF(1,0)],
			s_vy0[VOFF(0,-1)], s_vy0[VOFF(0,0)],
			vz0_m1, s_vz0[VOFF(0,0)] );

			d_tyy0[offset] += staggards2 (lamx, mux, 1.f, 1.f, 1.f, dt, ds,
			s_vy0[VOFF(0,-1)], s_vy0[VOFF(0,0)],
			s_vx0[VOFF(0,0)], s_vx0[VOFF(1,0)],
			vz0_m1, s_vz0[VOFF(0,0)] );

			d_tzz0[offset] += staggards2 (lamx, mux, 1.f, 1.f, 1.f, dt, ds,
			vz0_m1, s_vz0[VOFF(0,0)],
			s_vx0[VOFF(0,0)], s_vx0[VOFF(1,0)],
			s_vy0[VOFF(0,-1)], s_vy0[VOFF(0,0)] );

			d_txy0[offset] += staggardt2 (muy, 1.f, 1.f, dt, ds,
			s_vy0[VOFF(-1,0)], s_vy0[VOFF(0,0)],
			s_vx0[VOFF(0,0)], s_vx0[VOFF(0,1)] );

			d_txz0[offset] += staggardt2 (muz, 1.f, 1.f, dt, ds,
			s_vz0[VOFF(-1,0)], s_vz0[VOFF(0,0)],
			s_vx0[VOFF(0,0)], vx0_p1 );

			d_tyz0[offset] += staggardt2 (muxyz, 1.f, 1.f, dt, ds,
			s_vz0[VOFF(0,0)], s_vz0[VOFF(0,1)],
			s_vy0[VOFF(0,0)], vy0_p1 );
		}
	}/*}}}*/
	__syncthreads();

// pour k = sizez-1 (surface libre) ---------------------------------------------------------------------->>>
	k++;
	// decalage des donnees
	if (active) {/*{{{*/
		// d�calage des donn�es selon l'axe Z
		tx = (threadIdx.y+2)*(NPPDX+4) + threadIdx.x+2;
		vx0_m1 = s_vx0[tx];
		s_vx0[tx] = vx0_p1;
		vx0_p1 = vx0_p2;

		vy0_m1 = s_vy0[tx];
		s_vy0[tx] = vy0_p1;
		vy0_p1 = vy0_p2;

		vz0_m2 = vz0_m1;
		vz0_m1 = s_vz0[tx];
		s_vz0[tx] = vz0_p1;
		vz0_p1 = vz0_p2;

		// remarque : le tableau d�passe de un point ds chaque direction du domaine calcul�, donc si on est au bord du domaine,
		// i+1 est d�fini et la valeur � ce point est nulle (car hors du domaine)
		// i+2 correspond � la valeur pour j+1 et i=-1, cad 0 car hors du domaine => �a reste coh�rent
		// idem pour i - 1 et i - 2 (i-2 toujours d�fini � cause du padding d'alignement)
		// on charge les deux rang�es i-1 et i-2
		if (threadIdx.x == 0) {
			// i-2
			tx = (threadIdx.y+2)*(NPPDX+4) + 0;
			offset = k*pitch_x*pitch_y + j*pitch_x + i-2;
			s_vx0[tx] = d_vx0[offset];
			s_vy0[tx] = d_vy0[offset];
			s_vz0[tx] = d_vz0[offset];
			// i-1
			tx = (threadIdx.y+2)*(NPPDX+4) + 1;
			offset = k*pitch_x*pitch_y + j*pitch_x + i-1;
			s_vx0[tx] = d_vx0[offset];
			s_vy0[tx] = d_vy0[offset];
			s_vz0[tx] = d_vz0[offset];
		}
		// on charge les deux rang�es i+1 et i+2
		if (last_x) {
			// i+1
			tx = (threadIdx.y+2)*(NPPDX+4) + threadIdx.x + 3;
			offset = k*pitch_x*pitch_y + j*pitch_x + i + 1;
			s_vx0[tx] = d_vx0[offset];
			s_vy0[tx] = d_vy0[offset];
			s_vz0[tx] = d_vz0[offset];
			// i+2
			tx = (threadIdx.y+2)*(NPPDX+4) + threadIdx.x + 4;
			offset = k*pitch_x*pitch_y + j*pitch_x + i + 2;
			s_vx0[tx] = d_vx0[offset];
			s_vy0[tx] = d_vy0[offset];
			s_vz0[tx] = d_vz0[offset];
		}
		// on charge les rang�es j-1 et j-2
		if (threadIdx.y == 0) {
			// j-2
			tx = threadIdx.x+2;
			offset = k*pitch_x*pitch_y + (j-2)*pitch_x + i;
			s_vx0[tx] = d_vx0[offset];
			s_vy0[tx] = d_vy0[offset];
			s_vz0[tx] = d_vz0[offset];
			// j-1
			tx = (NPPDX+4) + threadIdx.x+2;
			offset = k*pitch_x*pitch_y + (j-1)*pitch_x + i;
			s_vx0[tx] = d_vx0[offset];
			s_vy0[tx] = d_vy0[offset];
			s_vz0[tx] = d_vz0[offset];
		}
		// on charge les rang�es j+1 et j+2
		if (last_y) {
			// j+2
			tx = (threadIdx.y + 4)*(NPPDX+4) + threadIdx.x+2;
			offset = k*pitch_x*pitch_y + (j+2)*pitch_x + i;
			s_vx0[tx] = d_vx0[offset];
			s_vy0[tx] = d_vy0[offset];
			s_vz0[tx] = d_vz0[offset];
			// j+1
			tx = (threadIdx.y + 3)*(NPPDX+4) + threadIdx.x+2;
			offset = k*pitch_x*pitch_y + (j+1)*pitch_x + i;
			s_vx0[tx] = d_vx0[offset];
			s_vy0[tx] = d_vy0[offset];
			s_vz0[tx] = d_vz0[offset];
		}
		// on charge les points pour k+2 (acces en mem globale)
		if (distance_zmax < 2) {
			vx0_p2 = 0.f;
			vy0_p2 = 0.f;
			vz0_p2 = 0.f;
		} else {
			vx0_p2 = d_vx0[(k+2)*pitch_x*pitch_y + j*pitch_x + i];
			vy0_p2 = d_vy0[(k+2)*pitch_x*pitch_y + j*pitch_x + i];
			vz0_p2 = d_vz0[(k+2)*pitch_x*pitch_y + j*pitch_x + i];
		}
	}/*}}}*/
	__syncthreads();
	offset = k*pitch_x*pitch_y + j*pitch_x + i;
	// calcul
	if (active) {/*{{{*/
		int npml = d_npml_tab[k*(sizex)*(sizey) + j*(sizex) + i];
		if ( npml >= 0){/*{{{*/
			/* Calculation of txx, tyy and tzz */
			if (distance_ymin >= 1 && distance_xmax >= 1 ){
				float mux = MU(k);
				float vpx = VP(k);
				float lamx = LAM(k);
				
				float b1 = 4.f * mux * (lamx + mux) / (lamx + 2.f*mux);
				float b2 = 2.f * mux * lamx / (lamx + 2.f*mux);
				float phixdum =	d_phivxx[npml];
				float phiydum = d_phivyy[npml];

				phixdum = CPML2 (vpx, tex1Dfetch(tex_dumpx2, i), tex1Dfetch(tex_alphax2, i), tex1Dfetch(tex_kappax2, i), phixdum, ds, dt, s_vx0[VOFF(0,0)], s_vx0[VOFF(1,0)]);
				phiydum = CPML2 (vpx, tex1Dfetch(tex_dumpy, j), tex1Dfetch(tex_alphay, j), tex1Dfetch(tex_kappay, j), phiydum, ds, dt, s_vy0[VOFF(0,-1)], s_vy0[VOFF(0,0)]);

				d_txx0[offset] += b1*dt*phixdum + b2*dt*phiydum
				+ b1*dt*(s_vx0[VOFF(1,0)] - s_vx0[VOFF(0,0)])/(tex1Dfetch(tex_kappax2, i)*ds)
				+ b2*dt*(s_vy0[VOFF(0,0)] - s_vy0[VOFF(0,-1)])/(tex1Dfetch(tex_kappay, j)*ds);

				d_tyy0[offset] += b1*dt*phiydum + b2*dt*phixdum
				+ b1*dt*(s_vy0[VOFF(0,0)] - s_vy0[VOFF(0,-1)])/(tex1Dfetch(tex_kappay, j)*ds)
				+ b2*dt*(s_vx0[VOFF(1,0)] - s_vx0[VOFF(0,0)])/(tex1Dfetch(tex_kappax2, i)*ds);
			
				d_tzz0[offset] = 0.f;
				
				d_phivxx[npml] = phixdum;
				d_phivyy[npml] = phiydum;
			} // if ( distance_zmin >= 1 && distance_ymin >= 1 && distance_xmax <= 1 )
			/* Calculation of txy */
			if ( distance_ymax >= 1 && distance_xmin >= 1 ){
				float muy = MU(k);
				float vpy = VP(k);

				float phixdum =	d_phivyx[npml];
				float phiydum = d_phivxy[npml];

				phixdum = CPML2 (vpy, tex1Dfetch(tex_dumpx, i), tex1Dfetch(tex_alphax, i), tex1Dfetch(tex_kappax, i), phixdum, ds, dt,
				s_vy0[VOFF(-1,0)], s_vy0[VOFF(0,0)]);

				phiydum = CPML2 (vpy, tex1Dfetch(tex_dumpy2, j), tex1Dfetch(tex_alphay2, j), tex1Dfetch(tex_kappay2, j), phiydum, ds, dt,
				s_vx0[VOFF(0,0)], s_vx0[VOFF(0,1)]);
				
				d_txy0[offset] += dt*muy*( phixdum + phiydum )
				+ staggardt2 (muy, tex1Dfetch(tex_kappax, i), tex1Dfetch(tex_kappay2, j), dt, ds,
				s_vy0[VOFF(-1,0)], s_vy0[VOFF(0,0)],
				s_vx0[VOFF(0,0)], s_vx0[VOFF(0,1)]);

				d_phivyx[npml] = phixdum;
				d_phivxy[npml] = phiydum;
			}
			d_txz0[offset] = - d_txz0[(k-1)*pitch_x*pitch_y + j*pitch_x + i]; // calcul� � l'it�ration pr�c�dente
			d_tyz0[offset] = - d_txz0[(k-1)*pitch_x*pitch_y + j*pitch_x + i]; // calcul� � l'it�ration pr�c�dente
		} else {/*}}}*/
			float mux = MU(k);
			float lamx = LAM(k);
			float muy = MU(k);
			float b1 = 4.f * mux * (lamx + mux) / (lamx + 2.f*mux);
			float b2 = 2.f * mux * lamx / (lamx + 2.f*mux);

			d_txx0[offset] += b1*dt*(s_vx0[VOFF(1,0)]-s_vx0[VOFF(0,0)])/ds + b2*dt*(s_vy0[VOFF(0,0)] - s_vy0[VOFF(0,-1)])/ds;
			d_tyy0[offset] += b1*dt*(s_vy0[VOFF(0,0)]-s_vy0[VOFF(0,-1)])/ds	+ b2*dt*(s_vx0[VOFF(1,0)]-s_vx0[VOFF(0,0)])/ds;
			d_tzz0[offset] = 0.f;

			d_txy0[offset] += staggardt2 (muy, 1.f, 1.f, dt, ds, s_vy0[VOFF(-1,0)], s_vy0[VOFF(0,0)], s_vx0[VOFF(0,0)], s_vx0[VOFF(0,1)]);
		}
	}/*}}}*/
	return;
}
// }}}

// WRAPPER {{{
void computeStress1D (	float* d_txx0, float* d_tyy0, float* d_tzz0, float* d_txy0, float* d_txz0, float* d_tyz0,
			float* d_vx0, float* d_vy0, float* d_vz0,
			int* d_npml_tab, float* d_phivxx, float* d_phivxy, float* d_phivxz, float* d_phivyx, float* d_phivyy, float* d_phivyz, float* d_phivzx, float* d_phivzy, float* d_phivzz, 
			int sizex, int sizey, int sizez,
			int pitch_x, int pitch_y, int pitch_z, 
			float ds, float dt, int delta, int compute_external,
			int grid_x, int grid_y, int grid_z, int block_x, int block_y, int block_z, int position) {

#ifdef ENABLE_VERY_SLOW_ERROR_CHECKING
	hipDeviceSynchronize();
	printCudaErr(hipGetLastError(), "before computeStress kernel");
#endif
	dim3 grid_dim(grid_x,grid_y,grid_z);
	dim3 block_dim(block_x, block_y, block_z);
	compute_stress_1d <<< grid_dim, block_dim, 0 >>> (	d_txx0, d_tyy0, d_tzz0, d_txy0, d_txz0, d_tyz0,
							d_vx0, d_vy0, d_vz0,
							d_npml_tab, d_phivxx, d_phivxy, d_phivxz, d_phivyx, d_phivyy, d_phivyz, d_phivzx, d_phivzy, d_phivzz, 
							sizex, sizey, sizez,
							pitch_x, pitch_y, pitch_z, 
							ds, dt, delta, (compute_external)?true:false, position);
#ifdef ENABLE_VERY_SLOW_ERROR_CHECKING
	hipDeviceSynchronize();
	printCudaErr(hipGetLastError(), "after computeStress kernel");
#endif
}
// }}}
// }}}

// COMPUTE VELOCITY {{{
// IMPLEMENTATION {{{
__global__ void compute_veloc_1d (	float* d_txx0, float* d_tyy0, float* d_tzz0, float* d_txy0, float* d_txz0, float* d_tyz0,
							float* d_vx0, float* d_vy0, float* d_vz0,
							float* d_fx, float* d_fy, float* d_fz, 
							int* d_npml_tab, float* d_phitxxx, float* d_phitxyy, float* d_phitxzz, float *d_phitxyx, float *d_phityyy, float *d_phityzz, float *d_phitxzx, float *d_phityzy, float *d_phitzzz,
							int sizex, int sizey, int sizez,
							int pitch_x, int pitch_y, int pitch_z, 
							float ds, float dt, int delta, bool compute_external, int position)
{
	__shared__ float s_txx0[(NPPDX_K2+4)*(NPPDY_K2+4)];
	__shared__ float s_tyy0[(NPPDX_K2+4)*(NPPDY_K2+4)];
	__shared__ float s_tzz0[(NPPDX_K2+4)*(NPPDY_K2+4)];
	__shared__ float s_txy0[(NPPDX_K2+4)*(NPPDY_K2+4)];
	__shared__ float s_txz0[(NPPDX_K2+4)*(NPPDY_K2+4)];
	__shared__ float s_tyz0[(NPPDX_K2+4)*(NPPDY_K2+4)];
	
	// m1 pour k-1, m2 pour k-2, p1 pour k+1, p2 pour k+2
	float tzz0_m1, tzz0_p1, tzz0_p2;
	float txz0_m1, txz0_m2, txz0_p1, txz0_ip1_km1;
	float tyz0_m1, tyz0_m2, tyz0_p1, tyz0_jm1_km1;
	
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	int j = blockIdx.y*blockDim.y + threadIdx.y;
	int distance_xmin = i;
	int distance_xmax = sizex - i - 1;
	int distance_ymin = j;
	int distance_ymax = sizey - j - 1;
	int offset, offset_source, tx;
	bool last_x, last_y;

	// seulement les blocs internes/externes travaillent suivant le parametre compute_external
	// s'il n'y a qu'un point dans le dernier bloc en x ou en y, on en prend deux.
	bool active =	(compute_external && 
				(blockIdx.x == 0 || blockIdx.x >= ((blockDim.x*gridDim.x-sizex == blockDim.x-1)?gridDim.x-2:gridDim.x-1) || blockIdx.y == 0 || blockIdx.y >= ((blockDim.y*gridDim.y-sizey == blockDim.y-1)?gridDim.y-2:gridDim.y-1)))
		||	(!compute_external && 
				!(blockIdx.x == 0 || blockIdx.x >= ((blockDim.x*gridDim.x-sizex == blockDim.x-1)?gridDim.x-2:gridDim.x-1) || blockIdx.y == 0 || blockIdx.y >= ((blockDim.y*gridDim.y-sizey == blockDim.y-1)?gridDim.y-2:gridDim.y-1))); 

	//on ne calcule pas les points qui sont en dehors du domaine. Les threads correspondants ne font rien.	
	active = active && ((distance_xmax >=0 && distance_ymax >=0)?true:false);
	last_x = last_y = false;
	if (distance_xmax == 0 || (active && threadIdx.x == (NPPDX-1))) {
		last_x = true;
	}
	if (distance_ymax == 0 || (active && threadIdx.y == (NPPDY-1))) {
		last_y = true;
	}

	// ici distance pour le modele global (on ne s'interesse qu'aux bords, donc si on n'est pas pres du bord, une valeur quelconque >2 suffit)
	distance_xmin = (position & MASK_FIRST_X)?i:DUMMY_VALUE;
	distance_xmax = (position & MASK_LAST_X)?(sizex - i - 1):DUMMY_VALUE;
	distance_ymin = (position & MASK_FIRST_Y)?j:DUMMY_VALUE;
	distance_ymax = (position & MASK_LAST_Y)?(sizey - j - 1):DUMMY_VALUE;


	
	// chargement initial des valeurs dans les registres et en memoire partagee
	if (active) {/*{{{*/
		// le tableau est initialis� � z�ro et ces �l�ments ne sont jamais mis � jour
		tzz0_m1 = 0.f;
		txz0_m1 = txz0_m2 = 0.f;
		tyz0_m1 = tyz0_m2 = 0.f;
		txz0_ip1_km1 = 0.f;
		tyz0_jm1_km1 = 0.f;

		offset = pitch_x*pitch_y + j*pitch_x + i;
		tzz0_p1 = d_tzz0[offset];
		txz0_p1 = d_txz0[offset];
		tyz0_p1 = d_tyz0[offset];

		offset = 2*pitch_x*pitch_y + j*pitch_x + i;
		tzz0_p2 = d_tzz0[offset];

		// vx0, vy0, vz0
		// chaque thread charge sa valeur en shmem
		tx = (threadIdx.y+2)*(NPPDX_K2+4) + threadIdx.x+2;
		offset = j*pitch_x + i;
		s_txx0[tx] = d_txx0[offset];
		s_tyy0[tx] = d_tyy0[offset];
		s_tzz0[tx] = d_tzz0[offset];
		s_txy0[tx] = d_txy0[offset];
		s_txz0[tx] = d_txz0[offset];
		s_tyz0[tx] = d_tyz0[offset];
		// on charge les deux rang�es i-1 et i-2
		if (threadIdx.x == 0) {
			// i-2
			tx = (threadIdx.y+2)*(NPPDX_K2+4) + 0;
			offset = j*pitch_x + i-2;
			s_txx0[tx] = d_txx0[offset];
			s_tyy0[tx] = d_tyy0[offset];
			s_tzz0[tx] = d_tzz0[offset];
			s_txy0[tx] = d_txy0[offset];
			s_txz0[tx] = d_txz0[offset];
			s_tyz0[tx] = d_tyz0[offset];
			// i-1
			tx = (threadIdx.y+2)*(NPPDX_K2+4) + 1;
			offset = j*pitch_x + i-1;
			s_txx0[tx] = d_txx0[offset];
			s_tyy0[tx] = d_tyy0[offset];
			s_tzz0[tx] = d_tzz0[offset];
			s_txy0[tx] = d_txy0[offset];
			s_txz0[tx] = d_txz0[offset];
			s_tyz0[tx] = d_tyz0[offset];
		}
		// on charge les deux rang�es i+1 et i+2
		if (last_x) {
			// i+1
			tx = (threadIdx.y+2)*(NPPDX_K2+4) + threadIdx.x + 3;
			offset = j*pitch_x + i + 1;
			s_txx0[tx] = d_txx0[offset];
			s_tyy0[tx] = d_tyy0[offset];
			s_tzz0[tx] = d_tzz0[offset];
			s_txy0[tx] = d_txy0[offset];
			s_txz0[tx] = d_txz0[offset];
			s_tyz0[tx] = d_tyz0[offset];
			// i+2
			tx = (threadIdx.y+2)*(NPPDX_K2+4) + threadIdx.x + 4;
			offset = j*pitch_x + i + 2;
			s_txx0[tx] = d_txx0[offset];
			s_tyy0[tx] = d_tyy0[offset];
			s_tzz0[tx] = d_tzz0[offset];
			s_txy0[tx] = d_txy0[offset];
			s_txz0[tx] = d_txz0[offset];
			s_tyz0[tx] = d_tyz0[offset];
		}
		// on charge les rang�es j-1 et j-2
		if (threadIdx.y == 0) {
			// j-2
			tx = threadIdx.x+2;
			offset = (j-2)*pitch_x + i;
			s_txx0[tx] = d_txx0[offset];
			s_tyy0[tx] = d_tyy0[offset];
			s_tzz0[tx] = d_tzz0[offset];
			s_txy0[tx] = d_txy0[offset];
			s_txz0[tx] = d_txz0[offset];
			s_tyz0[tx] = d_tyz0[offset];
			// j-1
			tx = (NPPDX_K2+4) + threadIdx.x+2;
			offset = (j-1)*pitch_x + i;
			s_txx0[tx] = d_txx0[offset];
			s_tyy0[tx] = d_tyy0[offset];
			s_tzz0[tx] = d_tzz0[offset];
			s_txy0[tx] = d_txy0[offset];
			s_txz0[tx] = d_txz0[offset];
			s_tyz0[tx] = d_tyz0[offset];
		}
		// on charge les rang�es j+1 et j+2
		if (last_y) {
			// j+2
			tx = (threadIdx.y + 4)*(NPPDX_K2+4) + threadIdx.x+2;
			offset = (j+2)*pitch_x + i;
			s_txx0[tx] = d_txx0[offset];
			s_tyy0[tx] = d_tyy0[offset];
			s_tzz0[tx] = d_tzz0[offset];
			s_txy0[tx] = d_txy0[offset];
			s_txz0[tx] = d_txz0[offset];
			s_tyz0[tx] = d_tyz0[offset];
			// j+1
			tx = (threadIdx.y + 3)*(NPPDX_K2+4) + threadIdx.x+2;
			offset = (j+1)*pitch_x + i;
			s_txx0[tx] = d_txx0[offset];
			s_tyy0[tx] = d_tyy0[offset];
			s_tzz0[tx] = d_tzz0[offset];
			s_txy0[tx] = d_txy0[offset];
			s_txz0[tx] = d_txz0[offset];
			s_tyz0[tx] = d_tyz0[offset];
		}
	}/*}}}*/
	int npml=-2;
	float rhoxy, rhoxz;
	// boucle sur z
	for (int k = 0; k < sizez; k++) {/*{{{*/
		int distance_zmin = k;
		int distance_zmax = sizez - k - 1;
		
		// apres decalage de la fenetre, on decalle les valeurs selon l'axe des Z.
		if (active) {/*{{{*/
			if (k>0) {
				// d�calage des donn�es selon l'axe Z
				txz0_ip1_km1 = s_txz0[(threadIdx.y+2)*(NPPDX_K2+4) + threadIdx.x+2+1];
				tyz0_jm1_km1 = s_tyz0[(threadIdx.y+2-1)*(NPPDX_K2+4) + threadIdx.x+2];
			}
		}/*}}}*/
		// synchro avant d'ecraser s_txz0 et s_tyz0
		__syncthreads();
		if (active) {/*{{{*/
			if (k>0) {/*{{{*/
				tx = (threadIdx.y+2)*(NPPDX_K2+4) + threadIdx.x+2;
				tzz0_m1 = s_tzz0[tx];
				s_tzz0[tx] = tzz0_p1;
				tzz0_p1 = tzz0_p2;
				
				txz0_m2 = txz0_m1;
				txz0_m1 = s_txz0[tx];
				s_txz0[tx] = txz0_p1;
				
				tyz0_m2 = tyz0_m1;
				tyz0_m1 = s_tyz0[tx];
				s_tyz0[tx] = tyz0_p1;

				// remarque : le tableau d�passe de un point ds chaque direction du domaine calcul�, donc si on est au bord du domaine,
				// i+1 est d�fini et la valeur � ce point est nulle (car hors du domaine)
				// i+2 correspond � la valeur pour j+1 et i=-1, cad 0 car hors du domaine => �a reste coh�rent
				// idem pour i - 1 et i - 2 (i-2 toujours d�fini � cause du padding d'alignement)
				tx = (threadIdx.y+2)*(NPPDX_K2+4) + threadIdx.x+2;
				offset = k*pitch_x*pitch_y + j*pitch_x + i;
				s_txx0[tx] = d_txx0[offset];
				s_tyy0[tx] = d_tyy0[offset];
				// ! ici !!! : txy a la place de tyz
				s_txy0[tx] = d_txy0[offset];
				// on charge les deux rang�es i-1 et i-2
				if (threadIdx.x == 0) {
					// i-2
					tx = (threadIdx.y+2)*(NPPDX_K2+4) + 0;
					offset = k*pitch_x*pitch_y + j*pitch_x + i-2;
					s_txx0[tx] = d_txx0[offset];
					s_tyy0[tx] = d_tyy0[offset];
					s_tzz0[tx] = d_tzz0[offset];
					s_txy0[tx] = d_txy0[offset];
					s_txz0[tx] = d_txz0[offset];
					s_tyz0[tx] = d_tyz0[offset];
					// i-1
					tx = (threadIdx.y+2)*(NPPDX_K2+4) + 1;
					offset = k*pitch_x*pitch_y + j*pitch_x + i-1;
					s_txx0[tx] = d_txx0[offset];
					s_tyy0[tx] = d_tyy0[offset];
					s_tzz0[tx] = d_tzz0[offset];
					s_txy0[tx] = d_txy0[offset];
					s_txz0[tx] = d_txz0[offset];
					s_tyz0[tx] = d_tyz0[offset];
				}
				// on charge les deux rang�es i+1 et i+2
				if (last_x) {
					// i+1
					tx = (threadIdx.y+2)*(NPPDX_K2+4) + threadIdx.x + 3;
					offset = k*pitch_x*pitch_y + j*pitch_x + i + 1;
					s_txx0[tx] = d_txx0[offset];
					s_tyy0[tx] = d_tyy0[offset];
					s_tzz0[tx] = d_tzz0[offset];
					s_txy0[tx] = d_txy0[offset];
					s_txz0[tx] = d_txz0[offset];
					s_tyz0[tx] = d_tyz0[offset];
					// i+2
					tx = (threadIdx.y+2)*(NPPDX_K2+4) + threadIdx.x + 4;
					offset = k*pitch_x*pitch_y + j*pitch_x + i + 2;
					s_txx0[tx] = d_txx0[offset];
					s_tyy0[tx] = d_tyy0[offset];
					s_tzz0[tx] = d_tzz0[offset];
					s_txy0[tx] = d_txy0[offset];
					s_txz0[tx] = d_txz0[offset];
					s_tyz0[tx] = d_tyz0[offset];
				}
				// on charge les rang�es j-1 et j-2
				if (threadIdx.y == 0) {
					// j-2
					tx = threadIdx.x+2;
					offset = k*pitch_x*pitch_y + (j-2)*pitch_x + i;
					s_txx0[tx] = d_txx0[offset];
					s_tyy0[tx] = d_tyy0[offset];
					s_tzz0[tx] = d_tzz0[offset];
					s_txy0[tx] = d_txy0[offset];
					s_txz0[tx] = d_txz0[offset];
					s_tyz0[tx] = d_tyz0[offset];
					// j-1
					tx = (NPPDX_K2+4) + threadIdx.x+2;
					offset = k*pitch_x*pitch_y + (j-1)*pitch_x + i;
					s_txx0[tx] = d_txx0[offset];
					s_tyy0[tx] = d_tyy0[offset];
					s_tzz0[tx] = d_tzz0[offset];
					s_txy0[tx] = d_txy0[offset];
					s_txz0[tx] = d_txz0[offset];
					s_tyz0[tx] = d_tyz0[offset];
				}
				// on charge les rang�es j+1 et j+2
				if (last_y) {
					// j+2
					tx = (threadIdx.y + 4)*(NPPDX_K2+4) + threadIdx.x+2;
					offset = k*pitch_x*pitch_y + (j+2)*pitch_x + i;
					s_txx0[tx] = d_txx0[offset];
					s_tyy0[tx] = d_tyy0[offset];
					s_tzz0[tx] = d_tzz0[offset];
					s_txy0[tx] = d_txy0[offset];
					s_txz0[tx] = d_txz0[offset];
					s_tyz0[tx] = d_tyz0[offset];
					// j+1
					tx = (threadIdx.y + 3)*(NPPDX_K2+4) + threadIdx.x+2;
					offset = k*pitch_x*pitch_y + (j+1)*pitch_x + i;
					s_txx0[tx] = d_txx0[offset];
					s_tyy0[tx] = d_tyy0[offset];
					s_tzz0[tx] = d_tzz0[offset];
					s_txy0[tx] = d_txy0[offset];
					s_txz0[tx] = d_txz0[offset];
					s_tyz0[tx] = d_tyz0[offset];
				}
				// on charge les points pour k+2 (acces en mem globale)
				if (distance_zmax < 2) {
					tzz0_p2 = 0.f;
				} else {
					tzz0_p2 = d_tzz0[(k+2)*pitch_x*pitch_y + j*pitch_x + i];
				}
				if (distance_zmax < 1) {
					txz0_p1 = 0.f;
					tyz0_p1 = 0.f;
				} else {
					txz0_p1 = d_txz0[(k+1)*pitch_x*pitch_y + j*pitch_x + i];
					tyz0_p1 = d_tyz0[(k+1)*pitch_x*pitch_y + j*pitch_x + i];
				}
			}/*}}}*/
		}/*}}}*/
		
		// synchro : on attends que tous les threads aient fini d'ecrire dans la memoire partagee
		__syncthreads();
		
		offset = k*pitch_x*pitch_y + j*pitch_x + i;
		offset_source = k*pitch_x*pitch_y + j*pitch_x + i;
	
		if (active) {/*{{{*/
			npml = d_npml_tab[k*(sizex)*(sizey) + j*(sizex) + i];
		 	if (npml >= 0) {/*{{{*/
				//CPML
				// ICI !!!!!!!!
				if (distance_zmin >= 1 && distance_ymin >= 1 && distance_xmin >= 1) {/* VX {{{*/
					/* Calculation of vx */
					if ( distance_zmax == 0 ){ /* free surface */
						float phixdum = d_phitxxx[npml];
						float phiydum = d_phitxyy[npml];
						float phizdum = d_phitxzz[npml];

						phixdum = CPML2 (VP(k), tex1Dfetch(tex_dumpx, i), tex1Dfetch(tex_alphax, i), tex1Dfetch(tex_kappax, i), phixdum, ds, dt, s_txx0[TOFF(-1,0)], s_txx0[TOFF(0,0)] );
						phiydum = CPML2 (VP(k), tex1Dfetch(tex_dumpy, j), tex1Dfetch(tex_alphay, j), tex1Dfetch(tex_kappay, j), phiydum, ds, dt, s_txy0[TOFF(0,-1)], s_txy0[TOFF(0,0)] );
						phizdum = CPML2 (VP(k), tex1Dfetch(tex_dumpz, k), tex1Dfetch(tex_alphaz, k), tex1Dfetch(tex_kappaz, k), phizdum, ds, dt, txz0_m1, - txz0_m1 );

						if (distance_xmax == 0 || distance_ymax ==0) { // boundary condition
							d_vx0[offset] = 0.0f;
						} else {
							d_vx0[offset] += (dt/RHO(k))*( phixdum + phiydum + phizdum )
							+ staggardv2 (1.f/RHO(k), tex1Dfetch(tex_kappax, i), tex1Dfetch(tex_kappay, j), tex1Dfetch(tex_kappaz, k), dt, ds,
							s_txx0[TOFF(-1,0)], s_txx0[TOFF(0,0)],
							s_txy0[TOFF(0,-1)], s_txy0[TOFF(0,0)],
							txz0_m1, - txz0_m1 );
						}
						
						d_phitxxx[npml] = phixdum;
						d_phitxyy[npml] = phiydum;
						d_phitxzz[npml] = phizdum;
					} else if ( distance_zmax == 1 ){ /* in the first cell, 2nd order finite-difference instead of 4th order finite-difference */
						float phixdum = d_phitxxx[npml];
						float phiydum = d_phitxyy[npml];
						float phizdum = d_phitxzz[npml];

						phixdum = CPML2 (VP(k), tex1Dfetch(tex_dumpx, i), tex1Dfetch(tex_alphax, i), tex1Dfetch(tex_kappax, i), phixdum, ds, dt,
						s_txx0[TOFF(-1,0)], s_txx0[TOFF(0,0)] );
						phiydum = CPML2 (VP(k), tex1Dfetch(tex_dumpy, j), tex1Dfetch(tex_alphay, j), tex1Dfetch(tex_kappay, j), phiydum, ds, dt,
						s_txy0[TOFF(0,-1)], s_txy0[TOFF(0,0)] );
						phizdum = CPML2 (VP(k), tex1Dfetch(tex_dumpz, k), tex1Dfetch(tex_alphaz, k), tex1Dfetch(tex_kappaz, k), phizdum, ds, dt,
						txz0_m1, s_txz0[TOFF(0,0)] );
						
						if (distance_xmax == 0 || distance_ymax ==0) { // boundary condition
							d_vx0[offset] = 0.0f;
						} else {
							d_vx0[offset] += (dt/RHO(k))*( phixdum + phiydum + phizdum )
							+ staggardv2 (1.f/RHO(k), tex1Dfetch(tex_kappax, i), tex1Dfetch(tex_kappay, j), tex1Dfetch(tex_kappaz, k), dt, ds,
							s_txx0[TOFF(-1,0)], s_txx0[TOFF(0,0)],
							s_txy0[TOFF(0,-1)], s_txy0[TOFF(0,0)],
							txz0_m1, s_txz0[TOFF(0,0)] );
						}
						
						d_phitxxx[npml] = phixdum;
						d_phitxyy[npml] = phiydum;
						d_phitxzz[npml] = phizdum;
					} else { /* regular domain */
						// ICI !!!!!!!!
						float phixdum = d_phitxxx[npml];
						float phiydum = d_phitxyy[npml];
						float phizdum = d_phitxzz[npml];

						phixdum = CPML4 (VP(k), tex1Dfetch(tex_dumpx, i), tex1Dfetch(tex_alphax, i), tex1Dfetch(tex_kappax, i), phixdum, ds, dt,
						s_txx0[TOFF(-1,0)], s_txx0[TOFF(0,0)],
						s_txx0[TOFF(-2,0)], s_txx0[TOFF(1,0)] );
						phiydum = CPML4 (VP(k), tex1Dfetch(tex_dumpy, j), tex1Dfetch(tex_alphay, j), tex1Dfetch(tex_kappay, j), phiydum, ds, dt,
						s_txy0[TOFF(0,-1)], s_txy0[TOFF(0,0)],
						s_txy0[TOFF(0,-2)], s_txy0[TOFF(0,1)] );
						phizdum = CPML4 (VP(k), tex1Dfetch(tex_dumpz, k), tex1Dfetch(tex_alphaz, k), tex1Dfetch(tex_kappaz, k), phizdum, ds, dt,
						txz0_m1, s_txz0[TOFF(0,0)],
						txz0_m2, txz0_p1 );

						if (distance_xmax == 0 || distance_ymax ==0) { // boundary condition
							d_vx0[offset] = 0.0f;
						} else {
							d_vx0[offset] += (dt/RHO(k))*( phixdum + phiydum + phizdum )
							+ staggardv4 (1.f/RHO(k), tex1Dfetch(tex_kappax, i), tex1Dfetch(tex_kappay, j), tex1Dfetch(tex_kappaz, k), dt, ds,
							s_txx0[TOFF(-1,0)], s_txx0[TOFF(0,0)],
							s_txx0[TOFF(-2,0)], s_txx0[TOFF(1,0)],
							s_txy0[TOFF(0,-1)], s_txy0[TOFF(0,0)],
							s_txy0[TOFF(0,-2)], s_txy0[TOFF(0,1)],
							txz0_m1, s_txz0[TOFF(0,0)],
							txz0_m2, txz0_p1 );
						}
							
						d_phitxxx[npml] = phixdum;
						d_phitxyy[npml] = phiydum;
						d_phitxzz[npml] = phizdum;
					} /* end of if "free surface" */
				}/*}}}*/
				/* Calculation of vy */
				if ( distance_zmin >= 1 && distance_ymax >= 1 && distance_xmax >= 1 ){/*VY{{{*/
					rhoxy = RHO(k);
					float vpxy = VP(k);
					if ( distance_zmax == 0 ){ /* free surface */
						float phixdum = d_phitxyx[npml];
						float phiydum = d_phityyy[npml];
						float phizdum = d_phityzz[npml];

						phixdum = CPML2 (vpxy, tex1Dfetch(tex_dumpx2, i), tex1Dfetch(tex_alphax2, i), tex1Dfetch(tex_kappax2, i), phixdum, ds, dt,
						s_txy0[TOFF(0,0)], s_txy0[TOFF(1,0)] );
						phiydum = CPML2 (vpxy, tex1Dfetch(tex_dumpy2, j), tex1Dfetch(tex_alphay2, j), tex1Dfetch(tex_kappay2, j), phiydum, ds, dt,
						s_tyy0[TOFF(0,0)], s_tyy0[TOFF(0,1)] );
						phizdum = CPML2 (vpxy, tex1Dfetch(tex_dumpz, k), tex1Dfetch(tex_alphaz, k), tex1Dfetch(tex_kappaz, k), phizdum, ds, dt,
						tyz0_m1, - tyz0_m1 );
						
						if (distance_xmin == 0 || distance_ymin == 0) { // boundary condition
							d_vy0[offset] = 0.0f;
						} else {
							d_vy0[offset] += (dt/rhoxy)*( phixdum + phiydum + phizdum )
							+ staggardv2 (1.f/rhoxy, tex1Dfetch(tex_kappax2, i), tex1Dfetch(tex_kappay2, j), tex1Dfetch(tex_kappaz, k), dt, ds,
							s_txy0[TOFF(0,0)], s_txy0[TOFF(1,0)],
							s_tyy0[TOFF(0,0)], s_tyy0[TOFF(0,1)],
							tyz0_m1, - tyz0_m1 );
						}
						
						d_phitxyx[npml] = phixdum;
						d_phityyy[npml] = phiydum;
						d_phityzz[npml] = phizdum;
					} else if ( distance_zmax == 1 ){ /* in the first cell, 2nd order finite-difference instead of 4th order finite-difference */
						float phixdum = d_phitxyx[npml];
						float phiydum = d_phityyy[npml];
						float phizdum = d_phityzz[npml];

						phixdum = CPML2 (vpxy, tex1Dfetch(tex_dumpx2, i), tex1Dfetch(tex_alphax2, i), tex1Dfetch(tex_kappax2, i), phixdum, ds, dt,
						s_txy0[TOFF(0,0)], s_txy0[TOFF(1,0)] );
						phiydum = CPML2 (vpxy, tex1Dfetch(tex_dumpy2, j), tex1Dfetch(tex_alphay2, j), tex1Dfetch(tex_kappay2, j), phiydum, ds, dt,
						s_tyy0[TOFF(0,0)], s_tyy0[TOFF(0,1)] );
						phizdum = CPML2 (vpxy, tex1Dfetch(tex_dumpz, k), tex1Dfetch(tex_alphaz, k), tex1Dfetch(tex_kappaz, k), phizdum, ds, dt,
						tyz0_m1, s_tyz0[TOFF(0,0)] );

						if (distance_xmin == 0 || distance_ymin == 0) { // boundary condition
							d_vy0[offset] = 0.0f;
						} else {
							d_vy0[offset] += (dt/rhoxy)*( phixdum + phiydum + phizdum )
							+ staggardv2 (1.f/rhoxy, tex1Dfetch(tex_kappax2, i), tex1Dfetch(tex_kappay2, j), tex1Dfetch(tex_kappaz, k), dt, ds,
							s_txy0[TOFF(0,0)], s_txy0[TOFF(1,0)],
							s_tyy0[TOFF(0,0)], s_tyy0[TOFF(0,1)],
							tyz0_m1, s_tyz0[TOFF(0,0)] );
						}

						d_phitxyx[npml] = phixdum;
						d_phityyy[npml] = phiydum;
						d_phityzz[npml] = phizdum;
					} else { /* regular domain */
						float phixdum = d_phitxyx[npml];
						float phiydum = d_phityyy[npml];
						float phizdum = d_phityzz[npml];
						
						phixdum = CPML4 (vpxy, tex1Dfetch(tex_dumpx2, i), tex1Dfetch(tex_alphax2, i), tex1Dfetch(tex_kappax2, i), phixdum, ds, dt,
						s_txy0[TOFF(0,0)], s_txy0[TOFF(1,0)],
						s_txy0[TOFF(-1,0)], s_txy0[TOFF(2,0)] );
						phiydum = CPML4 (vpxy, tex1Dfetch(tex_dumpy2, j), tex1Dfetch(tex_alphay2, j), tex1Dfetch(tex_kappay2, j), phiydum, ds, dt,
						s_tyy0[TOFF(0,0)], s_tyy0[TOFF(0,1)],
						s_tyy0[TOFF(0,-1)], s_tyy0[TOFF(0,2)] );
						phizdum = CPML4 (vpxy, tex1Dfetch(tex_dumpz, k), tex1Dfetch(tex_alphaz, k), tex1Dfetch(tex_kappaz, k), phizdum, ds, dt,
						tyz0_m1, s_tyz0[TOFF(0,0)],
						tyz0_m2, tyz0_p1 );

						if (distance_xmin == 0 || distance_ymin == 0) { // boundary condition
							d_vy0[offset] = 0.0f;
						} else {
							d_vy0[offset] += (dt/rhoxy)*( phixdum + phiydum + phizdum )
							+ staggardv4 (1.f/rhoxy, tex1Dfetch(tex_kappax2, i), tex1Dfetch(tex_kappay2, j), tex1Dfetch(tex_kappaz, k), dt, ds,
							s_txy0[TOFF(0,0)], s_txy0[TOFF(1,0)],
							s_txy0[TOFF(-1,0)], s_txy0[TOFF(2,0)],
							s_tyy0[TOFF(0,0)], s_tyy0[TOFF(0,1)],
							s_tyy0[TOFF(0,-1)], s_tyy0[TOFF(0,2)],
							tyz0_m1, s_tyz0[TOFF(0,0)],
							tyz0_m2, tyz0_p1 );
						}
						
						d_phitxyx[npml] = phixdum;
						d_phityyy[npml] = phiydum;
						d_phityzz[npml] = phizdum;
					} /* end of if "free surface" */
				}/*}}}*/
				/* Calculation of vz */
				if ( distance_ymin >= 1 && distance_xmax >= 1 ){/*VZ{{{*/
					rhoxz = 0.25f*(RHO(k) + RHO(k+1)
								+ RHO(k) + RHO(k+1));
					float vpxz = 0.25f*(VP(k) + VP(k+1)
							   + VP(k) + VP(k+1));
					if ( distance_zmax == 0 ){ /* free surface */
						float phixdum = d_phitxzx[npml];
						float phiydum = d_phityzy[npml];
						float phizdum = d_phitzzz[npml];

						phixdum = CPML2 (vpxz, tex1Dfetch(tex_dumpx2, i), tex1Dfetch(tex_alphax2, i), tex1Dfetch(tex_kappax2, i), phixdum, ds, dt,
						- txz0_m1, - txz0_ip1_km1 );
						phiydum = CPML2 (vpxz, tex1Dfetch(tex_dumpy, j), tex1Dfetch(tex_alphay, j), tex1Dfetch(tex_kappay, j), phiydum, ds, dt,
						- tyz0_jm1_km1, - tyz0_m1 );
						phizdum = CPML2 (vpxz, tex1Dfetch(tex_dumpz2, k), tex1Dfetch(tex_alphaz2, k), tex1Dfetch(tex_kappaz2, k), phizdum, ds, dt,
						s_tzz0[TOFF(0,0)], - tzz0_m1 );

						if (distance_xmin == 0 || distance_ymax ==0) { // boundary condition
							d_vz0[offset] = 0.0f;
						} else {
							d_vz0[offset] += (dt/rhoxz)*( phixdum + phiydum + phizdum )
							+ staggardv2 (1.f/rhoxz, tex1Dfetch(tex_kappax2, i), tex1Dfetch(tex_kappay, j), tex1Dfetch(tex_kappaz2, k), dt, ds,
							- txz0_m1, - txz0_ip1_km1,
							- tyz0_jm1_km1, - tyz0_m1,
							s_tzz0[TOFF(0,0)], - tzz0_m1 );
						}
											
						d_phitxzx[npml] = phixdum;
						d_phityzy[npml] = phiydum;
						d_phitzzz[npml] = phizdum;
					} else if ( distance_zmax == 1 ){ /* in the first cell, 2nd order finite-difference instead of 4th order finite-difference */
						float phixdum = d_phitxzx[npml];
						float phiydum = d_phityzy[npml];
						float phizdum = d_phitzzz[npml];

						phixdum = CPML2 (vpxz, tex1Dfetch(tex_dumpx2, i), tex1Dfetch(tex_alphax2, i), tex1Dfetch(tex_kappax2, i), phixdum, ds, dt,
						s_txz0[TOFF(0,0)], s_txz0[TOFF(1,0)] );
						phiydum = CPML2 (vpxz, tex1Dfetch(tex_dumpy, j), tex1Dfetch(tex_alphay, j), tex1Dfetch(tex_kappay, j), phiydum, ds, dt,
						s_tyz0[TOFF(0,-1)], s_tyz0[TOFF(0,0)] );
						phizdum = CPML2 (vpxz, tex1Dfetch(tex_dumpz2, k), tex1Dfetch(tex_alphaz2, k), tex1Dfetch(tex_kappaz2, k), phizdum, ds, dt,
						s_tzz0[TOFF(0,0)], tzz0_p1 );

						
						if (distance_xmin == 0 || distance_ymax ==0) { // boundary condition
							d_vz0[offset] = 0.0f;
						} else {
							d_vz0[offset] += (dt/rhoxz)*( phixdum + phiydum + phizdum )
							+ staggardv2 (1.f/rhoxz, tex1Dfetch(tex_kappax2, i), tex1Dfetch(tex_kappay, j), tex1Dfetch(tex_kappaz2, k), dt, ds,
							s_txz0[TOFF(0,0)], s_txz0[TOFF(1,0)],
							s_tyz0[TOFF(0,-1)], s_tyz0[TOFF(0,0)],
							s_tzz0[TOFF(0,0)], tzz0_p1 );
						}
						
						d_phitxzx[npml] = phixdum;
						d_phityzy[npml] = phiydum;
						d_phitzzz[npml] = phizdum;
					} else { /* regular domain */
						float phixdum = d_phitxzx[npml];
						float phiydum = d_phityzy[npml];
						float phizdum = d_phitzzz[npml];

						phixdum = CPML4 (vpxz, tex1Dfetch(tex_dumpx2, i), tex1Dfetch(tex_alphax2, i), tex1Dfetch(tex_kappax2, i), phixdum, ds, dt,
						s_txz0[TOFF(0,0)], s_txz0[TOFF(1,0)],
						s_txz0[TOFF(-1,0)], s_txz0[TOFF(2,0)] );
						phiydum = CPML4 (vpxz, tex1Dfetch(tex_dumpy, j), tex1Dfetch(tex_alphay, j), tex1Dfetch(tex_kappay, j), phiydum, ds, dt,
						s_tyz0[TOFF(0,-1)], s_tyz0[TOFF(0,0)],
						s_tyz0[TOFF(0,-2)], s_tyz0[TOFF(0,1)] );
						phizdum = CPML4 (vpxz, tex1Dfetch(tex_dumpz2, k), tex1Dfetch(tex_alphaz2, k), tex1Dfetch(tex_kappaz2, k), phizdum, ds, dt,
						s_tzz0[TOFF(0,0)], tzz0_p1,
						tzz0_m1, tzz0_p2 );

						
						if (distance_xmin == 0 || distance_ymax ==0 || distance_zmin == 0) { // boundary condition
							d_vz0[offset] = 0.0f;
						} else {
							d_vz0[offset] += (dt/rhoxz)*( phixdum + phiydum + phizdum )
							+ staggardv4 (1.f/rhoxz, tex1Dfetch(tex_kappax2, i), tex1Dfetch(tex_kappay, j), tex1Dfetch(tex_kappaz2, k), dt, ds,
							s_txz0[TOFF(0,0)], s_txz0[TOFF(1,0)],
							s_txz0[TOFF(-1,0)], s_txz0[TOFF(2,0)],
							s_tyz0[TOFF(0,-1)], s_tyz0[TOFF(0,0)],
							s_tyz0[TOFF(0,-2)], s_tyz0[TOFF(0,1)],
							s_tzz0[TOFF(0,0)], tzz0_p1,
							tzz0_m1, tzz0_p2 );
						}
						
						d_phitxzx[npml] = phixdum;
						d_phityzy[npml] = phiydum;
						d_phitzzz[npml] = phizdum;
					} /* end of if "free surface" */
				}/*}}}*/
			/* Normal mode }}}*/
			} else {/*{{{*/
				rhoxy = RHO(k);
				rhoxz = 0.25f*(RHO(k) + RHO(k+1)
						  + RHO(k) + RHO(k+1));
				if (distance_xmin == 0 || distance_xmax == 0 || distance_ymin == 0 || distance_ymax == 0 || distance_zmin == 0) {
                                       d_vx0[offset] = 0.f;
                                       d_vy0[offset] = 0.f;
                                       d_vz0[offset] = 0.f;
				} else if ( distance_zmax == 0 ){ /* free surface */
					d_vx0[offset] += (1.f/RHO(k))*d_fx[offset_source]*dt/ds
					+ staggardv2 (1.f/RHO(k), 1.f, 1.f, 1.f, dt, ds,
					s_txx0[TOFF(-1,0)], s_txx0[TOFF(0,0)],
					s_txy0[TOFF(0,-1)], s_txy0[TOFF(0,0)],
					txz0_m1, - txz0_m1 );

					d_vy0[offset] += (1.f/rhoxy)*d_fy[offset_source]*dt/ds
					+ staggardv2 (1.f/rhoxy, 1.f, 1.f, 1.f, dt, ds,
					s_txy0[TOFF(0,0)], s_txy0[TOFF(1,0)],
					s_tyy0[TOFF(0,0)], s_tyy0[TOFF(0,1)],
					tyz0_m1, - tyz0_m1 );

					d_vz0[offset] += (1.f/rhoxz)*d_fz[offset_source]*dt/ds
					+ staggardv2 (1.f/rhoxz, 1.f, 1.f, 1.f, dt, ds,
					- txz0_m1, - txz0_ip1_km1,
					- tyz0_jm1_km1, - tyz0_m1,
					s_tzz0[TOFF(0,0)], - tzz0_m1 );
				} else if ( distance_zmax == 1 ){ /* in the first cell, 2nd order finite-difference instead of 4th order finite-difference */
					d_vx0[offset] += (1.f/RHO(k))*d_fx[offset_source]*dt/ds
					+ staggardv2 (1.f/RHO(k), 1.f, 1.f, 1.f, dt, ds,
					s_txx0[TOFF(-1,0)], s_txx0[TOFF(0,0)],
					s_txy0[TOFF(0,-1)], s_txy0[TOFF(0,0)],
					txz0_m1, s_txz0[TOFF(0,0)] );

					d_vy0[offset] += (1.f/rhoxy)*d_fy[offset_source]*dt/ds
					+ staggardv2 (1.f/rhoxy, 1.f, 1.f, 1.f, dt, ds,
					s_txy0[TOFF(0,0)], s_txy0[TOFF(1,0)],
					s_tyy0[TOFF(0,0)], s_tyy0[TOFF(0,1)],
					tyz0_m1, s_tyz0[TOFF(0,0)] );

					d_vz0[offset] += (1.f/rhoxz)*d_fz[offset_source]*dt/ds
					+ staggardv2 (1.f/rhoxz, 1.f, 1.f, 1.f, dt, ds,
					s_txz0[TOFF(0,0)], s_txz0[TOFF(1,0)],
					s_tyz0[TOFF(0,-1)], s_tyz0[TOFF(0,0)],
					s_tzz0[TOFF(0,0)], tzz0_p1 );
				} else { /* regular domain */

					d_vx0[offset] += (1.f/RHO(k))*d_fx[offset_source]*dt/ds
					+ staggardv4 (1.f/RHO(k), 1.f, 1.f, 1.f, dt, ds,
					s_txx0[TOFF(-1,0)], s_txx0[TOFF(0,0)],
					s_txx0[TOFF(-2,0)], s_txx0[TOFF(1,0)],
					s_txy0[TOFF(0,-1)], s_txy0[TOFF(0,0)],
					s_txy0[TOFF(0,-2)], s_txy0[TOFF(0,1)],
					txz0_m1, s_txz0[TOFF(0,0)],
					txz0_m2, txz0_p1 );

					d_vy0[offset] += (1.f/rhoxy)*d_fy[offset_source]*dt/ds
					+ staggardv4 (1.f/rhoxy, 1.f, 1.f, 1.f, dt, ds,
					s_txy0[TOFF(0,0)], s_txy0[TOFF(1,0)],
					s_txy0[TOFF(-1,0)], s_txy0[TOFF(2,0)],
					s_tyy0[TOFF(0,0)], s_tyy0[TOFF(0,1)],
					s_tyy0[TOFF(0,-1)], s_tyy0[TOFF(0,2)],
					tyz0_m1, s_tyz0[TOFF(0,0)],
					tyz0_m2, tyz0_p1 );

					d_vz0[offset] += (1.f/rhoxz)*d_fz[offset_source]*dt/ds
					+ staggardv4 (1.f/rhoxz, 1.f, 1.f, 1.f, dt, ds,
					s_txz0[TOFF(0,0)], s_txz0[TOFF(1,0)],
					s_txz0[TOFF(-1,0)], s_txz0[TOFF(2,0)],
					s_tyz0[TOFF(0,-1)], s_tyz0[TOFF(0,0)],
					s_tyz0[TOFF(0,-2)], s_tyz0[TOFF(0,1)],
					s_tzz0[TOFF(0,0)], tzz0_p1,
					tzz0_m1, tzz0_p2 );
				} /* end of if "free surface" */
			}/*}}}*/ /* end of normal mode */
		} // end of active/*}}}*/
		__syncthreads();
	}/*}}}*/
}
// }}}

// WRAPPER {{{
void computeVeloc1D (	float* d_txx0, float* d_tyy0, float* d_tzz0, float* d_txy0, float* d_txz0, float* d_tyz0,
			float* d_vx0, float* d_vy0, float* d_vz0,
			float* d_fx, float* d_fy, float* d_fz, 
			int* d_npml_tab, float* d_phitxxx, float* d_phitxyy, float* d_phitxzz, float *d_phitxyx, float *d_phityyy, float *d_phityzz, float *d_phitxzx, float *d_phityzy, float *d_phitzzz,
			int sizex, int sizey, int sizez,
			int pitch_x, int pitch_y, int pitch_z, 
			float ds, float dt, int delta, int compute_external,
			int grid_x, int grid_y, int grid_z, int block_x, int block_y, int block_z, int position) {

#ifdef ENABLE_VERY_SLOW_ERROR_CHECKING
	hipDeviceSynchronize();
	printCudaErr(hipGetLastError(), "before computeVeloc kernel");
#endif
	dim3 grid_dim(grid_x,grid_y,grid_z);
	dim3 block_dim(block_x, block_y, block_z);
	compute_veloc_1d <<< grid_dim, block_dim, 0 >>> (	d_txx0, d_tyy0, d_tzz0, d_txy0, d_txz0, d_tyz0,
							d_vx0, d_vy0, d_vz0,
							d_fx, d_fy, d_fz, 
							d_npml_tab, d_phitxxx, d_phitxyy, d_phitxzz, d_phitxyx, d_phityyy, d_phityzz, d_phitxzx, d_phityzy, d_phitzzz,
							sizex, sizey, sizez,
							pitch_x, pitch_y, pitch_z, 
							ds, dt, delta, (compute_external)?true:false, position);

#ifdef ENABLE_VERY_SLOW_ERROR_CHECKING
	hipDeviceSynchronize();
	printCudaErr(hipGetLastError(), "after computeVeloc kernel");
#endif

}
// }}}
// }}}
// }}}

// FOR 3D MODELS {{{
// COMPUTE STRESS {{{
// IMPLEMENTATION {{{
__global__ void compute_stress_3d (	float* d_txx0, float* d_tyy0, float* d_tzz0, float* d_txy0, float* d_txz0, float* d_tyz0,
							float* d_vx0, float* d_vy0, float* d_vz0,
							int* d_npml_tab, float* d_phivxx, float* d_phivxy, float* d_phivxz, float* d_phivyx, float* d_phivyy, float* d_phivyz, float* d_phivzx, float* d_phivzy, float* d_phivzz, 
							float* d_mu, float* d_lam, float* d_vp, 
							int sizex, int sizey, int sizez,
							int pitch_x, int pitch_y, int pitch_z, 
							float ds, float dt, int delta, bool compute_external, int position)
{
	__shared__ float s_mu[(NPPDX+1)*(NPPDY+1)][2];
	__shared__ float s_vp[(NPPDX+1)*(NPPDY+1)][2];
	__shared__ float s_lam[(NPPDX+1)*NPPDY]; // on peut s'en passer au prix d'une lecture supplementaire en mem glob (a voir qd je connaitrai la limitation : registres ou shmem)
	__shared__ float s_vx0[(NPPDX+4)*(NPPDY+4)];
	__shared__ float s_vy0[(NPPDX+4)*(NPPDY+4)];
	__shared__ float s_vz0[(NPPDX+4)*(NPPDY+4)];

	float vx0_m1, vx0_p1, vx0_p2; // vx0 pour k-1, vx0 pour k-2, vx0 pour k+1, vx0 pour k+2
	float vy0_m1, vy0_p1, vy0_p2;
	float vz0_m1, vz0_m2, vz0_p1, vz0_p2;
	
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	int j = blockIdx.y*blockDim.y + threadIdx.y;
	int k = 0;
	int distance_xmin = i;
	int distance_xmax = sizex - i - 1;
	int distance_ymin = j;
	int distance_ymax = sizey - j - 1;
	int distance_zmax = sizez - k - 1;
	int offset, tx;
	bool last_x, last_y;

	// seulement les blocs internes/externes travaillent suivant le parametre compute_external
	// s'il n'y a qu'un point dans le dernier bloc en x ou en y, on en prend deux.
	bool active =	(compute_external && 
				(blockIdx.x == 0 || blockIdx.x >= ((blockDim.x*gridDim.x-sizex == blockDim.x-1)?gridDim.x-2:gridDim.x-1) || blockIdx.y == 0 || blockIdx.y >= ((blockDim.y*gridDim.y-sizey == blockDim.y-1)?gridDim.y-2:gridDim.y-1)))
		||	(!compute_external && 
				!(blockIdx.x == 0 || blockIdx.x >= ((blockDim.x*gridDim.x-sizex == blockDim.x-1)?gridDim.x-2:gridDim.x-1) || blockIdx.y == 0 || blockIdx.y >= ((blockDim.y*gridDim.y-sizey == blockDim.y-1)?gridDim.y-2:gridDim.y-1))); 
	
	//on ne calcule pas les points qui sont en dehors du domaine. Les threads correspondants ne font rien.	
	active = active && ((distance_xmax >=0 && distance_ymax >=0)?true:false);
	last_x = last_y = false;
	if (distance_xmax == 0 || (active && threadIdx.x == (NPPDX-1))) {
		last_x = true;
	}
	if (distance_ymax == 0 || (active && threadIdx.y == (NPPDY-1))) {
		last_y = true;
	}
	// ici distance pour le modele global (on ne s'interesse qu'aux bords, donc si on n'est pas pres du bord, une valeur quelconque >2 suffit)
	distance_xmin = (position & MASK_FIRST_X)?i:DUMMY_VALUE;
	distance_xmax = (position & MASK_LAST_X)?(sizex - i - 1):DUMMY_VALUE;
	distance_ymin = (position & MASK_FIRST_Y)?j:DUMMY_VALUE;
	distance_ymax = (position & MASK_LAST_Y)?(sizey - j - 1):DUMMY_VALUE;

// pour k = 0 -------------------------------------------------------------------------------------------->>>
	// chargement initial des valeurs dans les registres et en memoire partagee
	if (active) {/*{{{*/
		// le tableau est initialis� � z�ro et ces �l�ments ne sont jamais mis � jour
		vx0_m1 = 0.f;
		vy0_m1 = 0.f;
		vz0_m1 = vz0_m2 = 0.f;

		offset = pitch_x*pitch_y + j*pitch_x + i;
		vx0_p1 = d_vx0[offset];
		vy0_p1 = d_vy0[offset];
		vz0_p1 = d_vz0[offset];

		offset += pitch_x*pitch_y;
		vx0_p2 = d_vx0[offset];
		vy0_p2 = d_vy0[offset];
		vz0_p2 = d_vz0[offset];

		// chaque thread charge une donn�e du domaine pour k=0
		tx = threadIdx.y*(NPPDX+1) + threadIdx.x;
		offset = j*pitch_x + i;
		s_mu[tx][0] = d_mu[offset];
		s_vp[tx][0] = d_vp[offset];
		s_lam[tx] = d_lam[offset];
		// maintenant, chaque thread charge une donn�e pour k=1 pour mu et vp
		offset = pitch_x*(pitch_y) + j*pitch_x + i;
		s_mu[tx][1] = d_mu[offset];
		s_vp[tx][1] = d_vp[offset];
		// maintenant, on charge les donn�es pour i+1(hors du block)
		// la derni�re rang�e charge les donn�es du halo
		if (last_x) {
			tx = threadIdx.y*(NPPDX+1) +threadIdx.x+1;
			offset = j*pitch_x + i+1;
			s_lam[tx] = d_lam[offset];
			s_mu[tx][0] = d_mu[offset];
			s_vp[tx][0] = d_vp[offset];
			// i+1, k+1
			offset = pitch_x*(pitch_y) + j*pitch_x + i+1;
			s_mu[tx][1] = d_mu[offset];
			s_vp[tx][1] = d_vp[offset];
		}
		// maintenant, on charge les donn�es pour j+1(hors du block) : uniquement vp et mu
		if (last_y) {
			tx = (threadIdx.y+1)*(NPPDX+1) + threadIdx.x;
			offset = (j+1)*pitch_x + i;
			s_mu[tx][0] = d_mu[offset];
			s_vp[tx][0] = d_vp[offset];
			offset = pitch_x*(pitch_y) + (j+1)*pitch_x + i;
			s_mu[tx][1] = d_mu[offset];
			s_vp[tx][1] = d_vp[offset];
		}
		// maintenant, on charge la donn�e pour j+1 & i+1(hors du block) : uniquement vp et mu
		if (last_y && last_x) {
			tx = (threadIdx.y+1)*(NPPDX+1) + threadIdx.x+1;
			offset = (j+1)*pitch_x + i+1;
			s_mu[tx][0] = d_mu[offset];
			s_vp[tx][0] = d_vp[offset];
			offset = pitch_x*(pitch_y) + (j+1)*pitch_x + i+1;
			s_mu[tx][1] = d_mu[offset];
			s_vp[tx][1] = d_vp[offset];
		}
		// vx0, vy0, vz0
		// chaque thread charge sa valeur en shmem
		tx = (threadIdx.y+2)*(NPPDX+4) + threadIdx.x+2;
		offset = j*pitch_x + i;
		s_vx0[tx] = d_vx0[offset];
		s_vy0[tx] = d_vy0[offset];
		s_vz0[tx] = d_vz0[offset];
		// on charge les deux rang�es i-1 et i-2
		if (threadIdx.x == 0) {
			// i-2
			tx = (threadIdx.y+2)*(NPPDX+4) + 0;
			offset = j*pitch_x + i-2;
			s_vx0[tx] = d_vx0[offset];
			s_vy0[tx] = d_vy0[offset];
			s_vz0[tx] = d_vz0[offset];
			// i-1
			tx = (threadIdx.y+2)*(NPPDX+4) + 1;
			offset = j*pitch_x + i-1;
			s_vx0[tx] = d_vx0[offset];
			s_vy0[tx] = d_vy0[offset];
			s_vz0[tx] = d_vz0[offset];
		}
		// on charge les deux rang�es i+1 et i+2
		if (last_x) {
			// i+1
			tx = (threadIdx.y+2)*(NPPDX+4) + threadIdx.x + 3;
			offset = j*pitch_x + i + 1;
			s_vx0[tx] = d_vx0[offset];
			s_vy0[tx] = d_vy0[offset];
			s_vz0[tx] = d_vz0[offset];
			// i+2
			tx = (threadIdx.y+2)*(NPPDX+4) + threadIdx.x + 4;
			offset = j*pitch_x + i + 2;
			s_vx0[tx] = d_vx0[offset];
			s_vy0[tx] = d_vy0[offset];
			s_vz0[tx] = d_vz0[offset];
		}
		// on charge les rang�es j-1 et j-2
		if (threadIdx.y == 0) {
			// j-2
			tx = threadIdx.x+2;
			offset = (j-2)*pitch_x + i;
			s_vx0[tx] = d_vx0[offset];
			s_vy0[tx] = d_vy0[offset];
			s_vz0[tx] = d_vz0[offset];
			// j-1
			tx = (NPPDX+4) + threadIdx.x+2;
			offset = (j-1)*pitch_x + i;
			s_vx0[tx] = d_vx0[offset];
			s_vy0[tx] = d_vy0[offset];
			s_vz0[tx] = d_vz0[offset];
		}
		// on charge les rang�es j+1 et j+2
		if (last_y) {
			// j+2
			tx = (threadIdx.y + 4)*(NPPDX+4) + threadIdx.x+2;
			offset = (j+2)*pitch_x + i;
			s_vx0[tx] = d_vx0[offset];
			s_vy0[tx] = d_vy0[offset];
			s_vz0[tx] = d_vz0[offset];
			// j+1
			tx = (threadIdx.y + 3)*(NPPDX+4) + threadIdx.x+2;
			offset = (j+1)*pitch_x + i;
			s_vx0[tx] = d_vx0[offset];
			s_vy0[tx] = d_vy0[offset];
			s_vz0[tx] = d_vz0[offset];
		}
	}/*}}}*/
	__syncthreads();
	// calcul
#ifndef NOCPML
	if (active) {/*{{{*/
		// acces coalescent � d_npml_tab
		int npml = d_npml_tab[k*(sizex)*(sizey) + j*(sizex) + i];
		offset = k*pitch_x*pitch_y + j*pitch_x + i;
		/* Calculation of txx, tyy and tzz */
		/* Calculation of txy */
		if ( distance_ymax >= 1 && distance_xmin >= 1 ){
			float muy = 0.5f*(s_mu[MUOFF(0,0)][0] + s_mu[MUOFF(0,1)][0]);
			float vpy = 0.5f*(s_vp[VPOFF(0,0)][0] + s_vp[VPOFF(0,1)][0]);

			float phixdum =	d_phivyx[npml];
			float phiydum = d_phivxy[npml];
			phixdum = CPML4 (vpy, tex1Dfetch(tex_dumpx, i), tex1Dfetch(tex_alphax, i), tex1Dfetch(tex_kappax, i), phixdum, ds, dt,
			s_vy0[VOFF(-1,0)], s_vy0[VOFF(0,0)],
			s_vy0[VOFF(-2,0)], s_vy0[VOFF(1,0)]);
			phiydum = CPML4 (vpy, tex1Dfetch(tex_dumpy2, j), tex1Dfetch(tex_alphay2, j), tex1Dfetch(tex_kappay2, j), phiydum, ds, dt,
			s_vx0[VOFF(0,0)], s_vx0[VOFF(0,1)],
			s_vx0[VOFF(0,-1)], s_vx0[VOFF(0,2)]);

			d_txy0[offset] += dt*muy*( phixdum + phiydum )
			+ staggardt4 (muy, tex1Dfetch(tex_kappax, i), tex1Dfetch(tex_kappay2, j), dt, ds,
			s_vy0[VOFF(-1,0)], s_vy0[VOFF(0,0)],
			s_vy0[VOFF(-2,0)], s_vy0[VOFF(1,0)],
			s_vx0[VOFF(0,0)], s_vx0[VOFF(0,1)],
			s_vx0[VOFF(0,-1)], s_vx0[VOFF(0,2)]);

			d_phivyx[npml] = phixdum;
			d_phivxy[npml] = phiydum;
		}
		/* Calculation of txz */
		if (distance_xmin >= 1 ){

			float muz = 0.5f*(s_mu[MUOFF(0,0)][0] + s_mu[MUOFF(0,0)][1]);
			float vpz = 0.5f*(s_vp[VPOFF(0,0)][0] + s_vp[VPOFF(0,0)][1]);
			float phixdum =	d_phivzx[npml];
			float phizdum = d_phivxz[npml];

			phixdum = CPML4 (vpz, tex1Dfetch(tex_dumpx, i), tex1Dfetch(tex_alphax, i), tex1Dfetch(tex_kappax, i), phixdum, ds, dt,
			s_vz0[VOFF(-1,0)], s_vz0[VOFF(0,0)],
			s_vz0[VOFF(-2,0)], s_vz0[VOFF(1,0)]);
			phizdum = CPML4 (vpz, tex1Dfetch(tex_dumpz2, k), tex1Dfetch(tex_alphaz2, k), tex1Dfetch(tex_kappaz2, k), phizdum, ds, dt,
			s_vx0[VOFF(0,0)], vx0_p1,
			vx0_m1, vx0_p2);

			d_txz0[offset] += dt*muz*( phixdum + phizdum )
			+ staggardt4 (muz, tex1Dfetch(tex_kappax, i), tex1Dfetch(tex_kappaz2, k), dt, ds,
			s_vz0[VOFF(-1,0)], s_vz0[VOFF(0,0)],
			s_vz0[VOFF(-2,0)], s_vz0[VOFF(1,0)],
			s_vx0[VOFF(0,0)], vx0_p1,
			vx0_m1, vx0_p2);

			d_phivzx[npml] = phixdum;
			d_phivxz[npml] = phizdum;
		}
		/* Calculation of tyz */
		if (distance_ymax >= 1){
			// (distance_xmax==0)?mu(i,j,k):mu(i+1,j,k);
			float muxyz = 0.125f*(s_mu[MUOFF(0,0)][0] + s_mu[MUOFF(0,0)][1]
						 + s_mu[MUOFF(0,1)][0] + s_mu[MUOFF(0,1)][1]
						 + s_mu[MUOFF(1,0)][0] + s_mu[MUOFF(1,0)][1]
						 + s_mu[MUOFF(1,1)][0] + s_mu[MUOFF(1,1)][1]);
			float vpxyz = 0.125f*(s_vp[VPOFF(0,0)][0] + s_vp[VPOFF(0,0)][1]
						 + s_vp[VPOFF(0,1)][0] + s_vp[VPOFF(0,1)][1]
						 + s_vp[VPOFF(1,0)][0] + s_vp[VPOFF(1,0)][1]
						 + s_vp[VPOFF(1,1)][0] + s_vp[VPOFF(1,1)][1]);
			float phiydum = d_phivzy[npml];
			float phizdum = d_phivyz[npml];

			phiydum = CPML4 (vpxyz, tex1Dfetch(tex_dumpy2, j), tex1Dfetch(tex_alphay2, j), tex1Dfetch(tex_kappay2, j), phiydum, ds, dt,
			s_vz0[VOFF(0,0)], s_vz0[VOFF(0,1)],
			s_vz0[VOFF(0,-1)], s_vz0[VOFF(0,2)] );
			phizdum = CPML4 (vpxyz, tex1Dfetch(tex_dumpz2, k), tex1Dfetch(tex_alphaz2, k), tex1Dfetch(tex_kappaz2, k), phizdum, ds, dt,
			s_vy0[VOFF(0,0)], vy0_p1,
			vy0_m1, vy0_p2 );

			d_tyz0[offset] += dt*muxyz*( phiydum + phizdum )
			+ staggardt4 (muxyz, tex1Dfetch(tex_kappay2, j), tex1Dfetch(tex_kappaz2, k), dt, ds,
			s_vz0[VOFF(0,0)], s_vz0[VOFF(0,1)],
			s_vz0[VOFF(0,-1)], s_vz0[VOFF(0,2)],
			s_vy0[VOFF(0,0)], vy0_p1,
			vy0_m1, vy0_p2 );
			
			d_phivzy[npml] = phiydum;
			d_phivyz[npml] = phizdum;
		}
	}/*}}}*/
	__syncthreads();
// pour k= 1 a k = delta - 1 (CPML only) ----------------------------------------------------------------->>>
	for (k = 1; k < delta; k++) {/*{{{*/
		// decalage des donnees
		if (active) {/*{{{*/
			// chaque thread charge une donn�e du domaine
			tx = threadIdx.y*(NPPDX+1) + threadIdx.x;
			// on a d�j� lu les donn�es pour k � l'it�ration pr�c�dente pour mu et vp
			s_mu[tx][0] = s_mu[tx][1];
			s_vp[tx][0] = s_vp[tx][1];
			offset = k*pitch_x*(pitch_y) + j*pitch_x + i;
			s_lam[tx] = d_lam[offset];
			// maintenant, chaque thread charge une donn�e pour k+1 pour mu et vp
			offset = (k+1)*pitch_x*(pitch_y) + j*pitch_x + i;
			s_mu[tx][1] = d_mu[offset];
			s_vp[tx][1] = d_vp[offset];
			// maintenant, on charge les donn�es pour i+1(hors du block)
			if (last_x) {
				// i+1
				tx = threadIdx.y*(NPPDX+1) +threadIdx.x+1;
				s_mu[tx][0] = s_mu[tx][1];
				s_vp[tx][0] = s_vp[tx][1];
				offset = k*pitch_x*(pitch_y)+j*pitch_x + i+1;
				s_lam[tx] = d_lam[offset];
				// i+1, k+1
				offset = (k+1)*pitch_x*(pitch_y)+j*pitch_x + i+1;
				s_mu[tx][1] = d_mu[offset];
				s_vp[tx][1] = d_vp[offset];
			}
			// maintenant, on charge les donn�es pour j+1(hors du block) : uniquement vp et mu
			if (last_y) {
				tx = (threadIdx.y+1)*(NPPDX+1) +threadIdx.x;
				s_mu[tx][0] = s_mu[tx][1];
				s_vp[tx][0] = s_vp[tx][1];
				offset = (k+1)*pitch_x*(pitch_y) + (j+1)*pitch_x + i;
				s_mu[tx][1] = d_mu[offset];
				s_vp[tx][1] = d_vp[offset];
			}
			// maintenant, on charge la donn�e pour j+1 & i+1(hors du block) : uniquement vp et mu
			if (last_y && last_x) {
				tx = (threadIdx.y+1)*(NPPDX+1) + threadIdx.x+1;
				s_mu[tx][0] = s_mu[tx][1];
				s_vp[tx][0] = s_vp[tx][1];
				offset = (k+1)*pitch_x*(pitch_y) + (j+1)*pitch_x + i+1;
				s_mu[tx][1] = d_mu[offset];
				s_vp[tx][1] = d_vp[offset];
			}
			// d�calage des donn�es selon l'axe Z
			tx = (threadIdx.y+2)*(NPPDX+4) + threadIdx.x+2;
			vx0_m1 = s_vx0[tx];
			s_vx0[tx] = vx0_p1;
			vx0_p1 = vx0_p2;

			vy0_m1 = s_vy0[tx];
			s_vy0[tx] = vy0_p1;
			vy0_p1 = vy0_p2;

			vz0_m2 = vz0_m1;
			vz0_m1 = s_vz0[tx];
			s_vz0[tx] = vz0_p1;
			vz0_p1 = vz0_p2;

			// remarque : le tableau d�passe de un point ds chaque direction du domaine calcul�, donc si on est au bord du domaine,
			// i+1 est d�fini et la valeur � ce point est nulle (car hors du domaine)
			// i+2 correspond � la valeur pour j+1 et i=-1, cad 0 car hors du domaine => �a reste coh�rent
			// idem pour i - 1 et i - 2 (i-2 toujours d�fini � cause du padding d'alignement)
			// on charge les deux rang�es i-1 et i-2
			if (threadIdx.x == 0) {
				// i-2
				tx = (threadIdx.y+2)*(NPPDX+4) + 0;
				offset = k*pitch_x*pitch_y + j*pitch_x + i-2;
				s_vx0[tx] = d_vx0[offset];
				s_vy0[tx] = d_vy0[offset];
				s_vz0[tx] = d_vz0[offset];
				// i-1
				tx = (threadIdx.y+2)*(NPPDX+4) + 1;
				offset = k*pitch_x*pitch_y + j*pitch_x + i-1;
				s_vx0[tx] = d_vx0[offset];
				s_vy0[tx] = d_vy0[offset];
				s_vz0[tx] = d_vz0[offset];
			}
			// on charge les deux rang�es i+1 et i+2
			if (last_x) {
				// i+1
				tx = (threadIdx.y+2)*(NPPDX+4) + threadIdx.x + 3;
				offset = k*pitch_x*pitch_y + j*pitch_x + i + 1;
				s_vx0[tx] = d_vx0[offset];
				s_vy0[tx] = d_vy0[offset];
				s_vz0[tx] = d_vz0[offset];
				// i+2
				tx = (threadIdx.y+2)*(NPPDX+4) + threadIdx.x + 4;
				offset = k*pitch_x*pitch_y + j*pitch_x + i + 2;
				s_vx0[tx] = d_vx0[offset];
				s_vy0[tx] = d_vy0[offset];
				s_vz0[tx] = d_vz0[offset];
			}
			// on charge les rang�es j-1 et j-2
			if (threadIdx.y == 0) {
				// j-2
				tx = threadIdx.x+2;
				offset = k*pitch_x*pitch_y + (j-2)*pitch_x + i;
				s_vx0[tx] = d_vx0[offset];
				s_vy0[tx] = d_vy0[offset];
				s_vz0[tx] = d_vz0[offset];
				// j-1
				tx = (NPPDX+4) + threadIdx.x+2;
				offset = k*pitch_x*pitch_y + (j-1)*pitch_x + i;
				s_vx0[tx] = d_vx0[offset];
				s_vy0[tx] = d_vy0[offset];
				s_vz0[tx] = d_vz0[offset];
			}
			// on charge les rang�es j+1 et j+2
			if (last_y) {
				// j+2
				tx = (threadIdx.y + 4)*(NPPDX+4) + threadIdx.x+2;
				offset = k*pitch_x*pitch_y + (j+2)*pitch_x + i;
				s_vx0[tx] = d_vx0[offset];
				s_vy0[tx] = d_vy0[offset];
				s_vz0[tx] = d_vz0[offset];
				// j+1
				tx = (threadIdx.y + 3)*(NPPDX+4) + threadIdx.x+2;
				offset = k*pitch_x*pitch_y + (j+1)*pitch_x + i;
				s_vx0[tx] = d_vx0[offset];
				s_vy0[tx] = d_vy0[offset];
				s_vz0[tx] = d_vz0[offset];
			}
			// on charge les points pour k+2 (acces en mem globale)
			if (distance_zmax < 2) {
				vx0_p2 = 0.f;
				vy0_p2 = 0.f;
				vz0_p2 = 0.f;
			} else {
				vx0_p2 = d_vx0[(k+2)*pitch_x*pitch_y + j*pitch_x + i];
				vy0_p2 = d_vy0[(k+2)*pitch_x*pitch_y + j*pitch_x + i];
				vz0_p2 = d_vz0[(k+2)*pitch_x*pitch_y + j*pitch_x + i];
			}
		}/*}}}*/
		__syncthreads();

		offset = k*pitch_x*pitch_y + j*pitch_x + i;
		// calcul
		if (active) {/*{{{*/
			// acces coalescent � d_npml_tab
			int npml = d_npml_tab[k*(sizex)*(sizey) + j*(sizex) + i];
			offset = k*pitch_x*pitch_y + j*pitch_x + i;
			/* Calculation of txx, tyy and tzz */
			if (distance_ymin >= 1 && distance_xmax >= 1 ){
				float mux = 0.5f*(s_mu[MUOFF(0,0)][0] + s_mu[MUOFF(1,0)][0]);
				float vpx = 0.5f*(s_vp[VPOFF(0,0)][0] + s_vp[VPOFF(1,0)][0]);
				float lamx = 0.5f*(s_lam[LAMOFF(0,0)] + s_lam[LAMOFF(1,0)]);
				
				float phixdum =	d_phivxx[npml];
				float phiydum = d_phivyy[npml];
				float phizdum = d_phivzz[npml];

				phixdum = CPML4 (vpx, tex1Dfetch(tex_dumpx2, i), tex1Dfetch(tex_alphax2, i), tex1Dfetch(tex_kappax2, i), phixdum, ds, dt,
				s_vx0[VOFF(0,0)], s_vx0[VOFF(1,0)],
				s_vx0[VOFF(-1,0)], s_vx0[VOFF(2,0)]);
				phiydum = CPML4 (vpx, tex1Dfetch(tex_dumpy, j), tex1Dfetch(tex_alphay, j), tex1Dfetch(tex_kappay, j), phiydum, ds, dt,
				s_vy0[VOFF(0,-1)], s_vy0[VOFF(0,0)],
				s_vy0[VOFF(0,-2)], s_vy0[VOFF(0,1)]);
				phizdum = CPML4 (vpx, tex1Dfetch(tex_dumpz, k), tex1Dfetch(tex_alphaz, k), tex1Dfetch(tex_kappaz, k), phizdum, ds, dt,
				vz0_m1, s_vz0[VOFF(0,0)],
				vz0_m2, vz0_p1);

				d_txx0[offset] += dt*(lamx + 2*mux)*phixdum + dt*lamx*( phiydum + phizdum )
				+ staggards4 (lamx, mux, tex1Dfetch(tex_kappax2, i), tex1Dfetch(tex_kappay, j), tex1Dfetch(tex_kappaz, k), dt, ds,
				s_vx0[VOFF(0,0)], s_vx0[VOFF(1,0)],
				s_vx0[VOFF(-1,0)], s_vx0[VOFF(2,0)],
				s_vy0[VOFF(0,-1)], s_vy0[VOFF(0,0)],
				s_vy0[VOFF(0,-2)], s_vy0[VOFF(0,1)],
				vz0_m1, s_vz0[VOFF(0,0)],
				vz0_m2, vz0_p1);

				d_tyy0[offset] += dt*lamx*( phixdum + phizdum ) + dt*(lamx + 2*mux)*phiydum
				+ staggards4 (lamx, mux, tex1Dfetch(tex_kappay, j), tex1Dfetch(tex_kappax2, i), tex1Dfetch(tex_kappaz, k), dt, ds,
				s_vy0[VOFF(0,-1)], s_vy0[VOFF(0,0)],
				s_vy0[VOFF(0,-2)], s_vy0[VOFF(0,1)],
				s_vx0[VOFF(0,0)], s_vx0[VOFF(1,0)],
				s_vx0[VOFF(-1,0)], s_vx0[VOFF(2,0)],
				vz0_m1, s_vz0[VOFF(0,0)],
				vz0_m2, vz0_p1);

				d_tzz0[offset] += dt*lamx*( phixdum + phiydum ) + dt*(lamx + 2*mux)*phizdum
				+ staggards4 (lamx, mux, tex1Dfetch(tex_kappaz, k), tex1Dfetch(tex_kappax2, i), tex1Dfetch(tex_kappay, j), dt, ds,
				vz0_m1, s_vz0[VOFF(0,0)],
				vz0_m2, vz0_p1,
				s_vx0[VOFF(0,0)], s_vx0[VOFF(1,0)],
				s_vx0[VOFF(-1,0)], s_vx0[VOFF(2,0)],
				s_vy0[VOFF(0,-1)], s_vy0[VOFF(0,0)],
				s_vy0[VOFF(0,-2)], s_vy0[VOFF(0,1)]);

				d_phivxx[npml] = phixdum;
				d_phivyy[npml] = phiydum;
				d_phivzz[npml] = phizdum;
			} // if (distance_ymin >= 1 && distance_xmax <= 1 )
			/* Calculation of txy */
			if ( distance_ymax >= 1 && distance_xmin >= 1 ){
				float muy = 0.5f*(s_mu[MUOFF(0,0)][0] + s_mu[MUOFF(0,1)][0]);
				float vpy = 0.5f*(s_vp[VPOFF(0,0)][0] + s_vp[VPOFF(0,1)][0]);

				float phixdum =	d_phivyx[npml];
				float phiydum = d_phivxy[npml];

				phixdum = CPML4 (vpy, tex1Dfetch(tex_dumpx, i), tex1Dfetch(tex_alphax, i), tex1Dfetch(tex_kappax, i), phixdum, ds, dt,
				s_vy0[VOFF(-1,0)], s_vy0[VOFF(0,0)],
				s_vy0[VOFF(-2,0)], s_vy0[VOFF(1,0)]);
				phiydum = CPML4 (vpy, tex1Dfetch(tex_dumpy2, j), tex1Dfetch(tex_alphay2, j), tex1Dfetch(tex_kappay2, j), phiydum, ds, dt,
				s_vx0[VOFF(0,0)], s_vx0[VOFF(0,1)],
				s_vx0[VOFF(0,-1)], s_vx0[VOFF(0,2)]);

				d_txy0[offset] += dt*muy*( phixdum + phiydum )
				+ staggardt4 (muy, tex1Dfetch(tex_kappax, i), tex1Dfetch(tex_kappay2, j), dt, ds,
				s_vy0[VOFF(-1,0)], s_vy0[VOFF(0,0)],
				s_vy0[VOFF(-2,0)], s_vy0[VOFF(1,0)],
				s_vx0[VOFF(0,0)], s_vx0[VOFF(0,1)],
				s_vx0[VOFF(0,-1)], s_vx0[VOFF(0,2)]);
				
				d_phivyx[npml] = phixdum;
				d_phivxy[npml] = phiydum;
			}
			/* Calculation of txz */
			if (distance_xmin >= 1 ){

				float muz = 0.5f*(s_mu[MUOFF(0,0)][0] + s_mu[MUOFF(0,0)][1]);
				float vpz = 0.5f*(s_vp[VPOFF(0,0)][0] + s_vp[VPOFF(0,0)][1]);

				float phixdum =	d_phivzx[npml];
				float phizdum = d_phivxz[npml];

				phixdum = CPML4 (vpz, tex1Dfetch(tex_dumpx, i), tex1Dfetch(tex_alphax, i), tex1Dfetch(tex_kappax, i), phixdum, ds, dt,
				s_vz0[VOFF(-1,0)], s_vz0[VOFF(0,0)],
				s_vz0[VOFF(-2,0)], s_vz0[VOFF(1,0)]);
				phizdum = CPML4 (vpz, tex1Dfetch(tex_dumpz2, k), tex1Dfetch(tex_alphaz2, k), tex1Dfetch(tex_kappaz2, k), phizdum, ds, dt,
				s_vx0[VOFF(0,0)], vx0_p1,
				vx0_m1, vx0_p2);

				d_txz0[offset] += dt*muz*( phixdum + phizdum )
				+ staggardt4 (muz, tex1Dfetch(tex_kappax, i), tex1Dfetch(tex_kappaz2, k), dt, ds,
				s_vz0[VOFF(-1,0)], s_vz0[VOFF(0,0)],
				s_vz0[VOFF(-2,0)], s_vz0[VOFF(1,0)],
				s_vx0[VOFF(0,0)], vx0_p1,
				vx0_m1, vx0_p2);

				d_phivzx[npml] = phixdum;
				d_phivxz[npml] = phizdum;
			}
			/* Calculation of tyz */
			if (distance_ymax >= 1){
				// (distance_xmax==0)?mu(i,j,k):mu(i+1,j,k);
				float muxyz = 0.125f*(s_mu[MUOFF(0,0)][0] + s_mu[MUOFF(0,0)][1]
							 + s_mu[MUOFF(0,1)][0] + s_mu[MUOFF(0,1)][1]
							 + s_mu[MUOFF(1,0)][0] + s_mu[MUOFF(1,0)][1]
							 + s_mu[MUOFF(1,1)][0] + s_mu[MUOFF(1,1)][1]);
				float vpxyz = 0.125f*(s_vp[VPOFF(0,0)][0] + s_vp[VPOFF(0,0)][1]
							 + s_vp[VPOFF(0,1)][0] + s_vp[VPOFF(0,1)][1]
							 + s_vp[VPOFF(1,0)][0] + s_vp[VPOFF(1,0)][1]
							 + s_vp[VPOFF(1,1)][0] + s_vp[VPOFF(1,1)][1]);
				float phiydum = d_phivzy[npml];
				float phizdum = d_phivyz[npml];

				phiydum = CPML4 (vpxyz, tex1Dfetch(tex_dumpy2, j), tex1Dfetch(tex_alphay2, j), tex1Dfetch(tex_kappay2, j), phiydum, ds, dt,
				s_vz0[VOFF(0,0)], s_vz0[VOFF(0,1)],
				s_vz0[VOFF(0,-1)], s_vz0[VOFF(0,2)] );
				phizdum = CPML4 (vpxyz, tex1Dfetch(tex_dumpz2, k), tex1Dfetch(tex_alphaz2, k), tex1Dfetch(tex_kappaz2, k), phizdum, ds, dt,
				s_vy0[VOFF(0,0)], vy0_p1,
				vy0_m1, vy0_p2 );

				d_tyz0[offset] += dt*muxyz*( phiydum + phizdum )
				+ staggardt4 (muxyz, tex1Dfetch(tex_kappay2, j), tex1Dfetch(tex_kappaz2, k), dt, ds,
				s_vz0[VOFF(0,0)], s_vz0[VOFF(0,1)],
				s_vz0[VOFF(0,-1)], s_vz0[VOFF(0,2)],
				s_vy0[VOFF(0,0)], vy0_p1,
				vy0_m1, vy0_p2 );
				
				d_phivzy[npml] = phiydum;
				d_phivyz[npml] = phizdum;
			}
		}/*}}}*/
		__syncthreads();
	}/*}}}*/
#endif

// pour k= delta a k = sizez - 3 (CPML + ordre 4) ---------------------------------------------------->>>
	for (k = delta ; k < sizez - 2; k++) {/*{{{*/
		// decalage des donnees
		if (active && k>0) {/*{{{*/
			// chaque thread charge une donn�e du domaine
			tx = threadIdx.y*(NPPDX+1) + threadIdx.x;
			// on a d�j� lu les donn�es pour k � l'it�ration pr�c�dente pour mu et vp
			s_mu[tx][0] = s_mu[tx][1];
			s_vp[tx][0] = s_vp[tx][1];
			offset = k*pitch_x*(pitch_y) + j*pitch_x + i;
			s_lam[tx] = d_lam[offset];
			// maintenant, chaque thread charge une donn�e pour k+1 pour mu et vp
			offset = (k+1)*pitch_x*(pitch_y) + j*pitch_x + i;
			s_mu[tx][1] = d_mu[offset];
			s_vp[tx][1] = d_vp[offset];
			// maintenant, on charge les donn�es pour i+1(hors du block)
			if (last_x) {
				// i+1
				tx = threadIdx.y*(NPPDX+1) +threadIdx.x+1;
				s_mu[tx][0] = s_mu[tx][1];
				s_vp[tx][0] = s_vp[tx][1];
				offset = k*pitch_x*(pitch_y)+j*pitch_x + i+1;
				s_lam[tx] = d_lam[offset];
				// i+1, k+1
				offset = (k+1)*pitch_x*(pitch_y)+j*pitch_x + i+1;
				s_mu[tx][1] = d_mu[offset];
				s_vp[tx][1] = d_vp[offset];
			}
			// maintenant, on charge les donn�es pour j+1(hors du block) : uniquement vp et mu
			if (last_y) {
				tx = (threadIdx.y+1)*(NPPDX+1) +threadIdx.x;
				s_mu[tx][0] = s_mu[tx][1];
				s_vp[tx][0] = s_vp[tx][1];
				offset = (k+1)*pitch_x*(pitch_y) + (j+1)*pitch_x + i;
				s_mu[tx][1] = d_mu[offset];
				s_vp[tx][1] = d_vp[offset];
			}
			// maintenant, on charge la donn�e pour j+1 & i+1(hors du block) : uniquement vp et mu
			if (last_y && last_x) {
				tx = (threadIdx.y+1)*(NPPDX+1) + threadIdx.x+1;
				s_mu[tx][0] = s_mu[tx][1];
				s_vp[tx][0] = s_vp[tx][1];
				offset = (k+1)*pitch_x*(pitch_y) + (j+1)*pitch_x + i+1;
				s_mu[tx][1] = d_mu[offset];
				s_vp[tx][1] = d_vp[offset];
			}
			// d�calage des donn�es selon l'axe Z
			tx = (threadIdx.y+2)*(NPPDX+4) + threadIdx.x+2;
			vx0_m1 = s_vx0[tx];
			s_vx0[tx] = vx0_p1;
			vx0_p1 = vx0_p2;

			vy0_m1 = s_vy0[tx];
			s_vy0[tx] = vy0_p1;
			vy0_p1 = vy0_p2;

			vz0_m2 = vz0_m1;
			vz0_m1 = s_vz0[tx];
			s_vz0[tx] = vz0_p1;
			vz0_p1 = vz0_p2;

			// remarque : le tableau d�passe de un point ds chaque direction du domaine calcul�, donc si on est au bord du domaine,
			// i+1 est d�fini et la valeur � ce point est nulle (car hors du domaine)
			// i+2 correspond � la valeur pour j+1 et i=-1, cad 0 car hors du domaine => �a reste coh�rent
			// idem pour i - 1 et i - 2 (i-2 toujours d�fini � cause du padding d'alignement)
			// on charge les deux rang�es i-1 et i-2
			if (threadIdx.x == 0) {
				// i-2
				tx = (threadIdx.y+2)*(NPPDX+4) + 0;
				offset = k*pitch_x*pitch_y + j*pitch_x + i-2;
				s_vx0[tx] = d_vx0[offset];
				s_vy0[tx] = d_vy0[offset];
				s_vz0[tx] = d_vz0[offset];
				// i-1
				tx = (threadIdx.y+2)*(NPPDX+4) + 1;
				offset = k*pitch_x*pitch_y + j*pitch_x + i-1;
				s_vx0[tx] = d_vx0[offset];
				s_vy0[tx] = d_vy0[offset];
				s_vz0[tx] = d_vz0[offset];
			}
			// on charge les deux rang�es i+1 et i+2
			if (last_x) {
				// i+1
				tx = (threadIdx.y+2)*(NPPDX+4) + threadIdx.x + 3;
				offset = k*pitch_x*pitch_y + j*pitch_x + i + 1;
				s_vx0[tx] = d_vx0[offset];
				s_vy0[tx] = d_vy0[offset];
				s_vz0[tx] = d_vz0[offset];
				// i+2
				tx = (threadIdx.y+2)*(NPPDX+4) + threadIdx.x + 4;
				offset = k*pitch_x*pitch_y + j*pitch_x + i + 2;
				s_vx0[tx] = d_vx0[offset];
				s_vy0[tx] = d_vy0[offset];
				s_vz0[tx] = d_vz0[offset];
			}
			// on charge les rang�es j-1 et j-2
			if (threadIdx.y == 0) {
				// j-2
				tx = threadIdx.x+2;
				offset = k*pitch_x*pitch_y + (j-2)*pitch_x + i;
				s_vx0[tx] = d_vx0[offset];
				s_vy0[tx] = d_vy0[offset];
				s_vz0[tx] = d_vz0[offset];
				// j-1
				tx = (NPPDX+4) + threadIdx.x+2;
				offset = k*pitch_x*pitch_y + (j-1)*pitch_x + i;
				s_vx0[tx] = d_vx0[offset];
				s_vy0[tx] = d_vy0[offset];
				s_vz0[tx] = d_vz0[offset];
			}
			// on charge les rang�es j+1 et j+2
			if (last_y) {
				// j+2
				tx = (threadIdx.y + 4)*(NPPDX+4) + threadIdx.x+2;
				offset = k*pitch_x*pitch_y + (j+2)*pitch_x + i;
				s_vx0[tx] = d_vx0[offset];
				s_vy0[tx] = d_vy0[offset];
				s_vz0[tx] = d_vz0[offset];
				// j+1
				tx = (threadIdx.y + 3)*(NPPDX+4) + threadIdx.x+2;
				offset = k*pitch_x*pitch_y + (j+1)*pitch_x + i;
				s_vx0[tx] = d_vx0[offset];
				s_vy0[tx] = d_vy0[offset];
				s_vz0[tx] = d_vz0[offset];
			}
			// on charge les points pour k+2 (acces en mem globale)
			if (distance_zmax < 2) {
				vx0_p2 = 0.f;
				vy0_p2 = 0.f;
				vz0_p2 = 0.f;
			} else {
				vx0_p2 = d_vx0[(k+2)*pitch_x*pitch_y + j*pitch_x + i];
				vy0_p2 = d_vy0[(k+2)*pitch_x*pitch_y + j*pitch_x + i];
				vz0_p2 = d_vz0[(k+2)*pitch_x*pitch_y + j*pitch_x + i];
			}
		}/*}}}*/
		__syncthreads();
		
		offset = k*pitch_x*pitch_y + j*pitch_x + i;
		// calcul
		if (active) {/*{{{*/
			// plus couteux que le test sur les bords -> a revoir un jour
			int npml = d_npml_tab[k*(sizex)*(sizey) + j*(sizex) + i];
			if ( npml >= 0){/*{{{*/
				// acces coalescent � d_npml_tab
				/* Calculation of txx, tyy and tzz */
				if (distance_ymin >= 1 && distance_xmax >= 1 ){
					float mux = 0.5f*(s_mu[MUOFF(0,0)][0] + s_mu[MUOFF(1,0)][0]);
					float vpx = 0.5f*(s_vp[VPOFF(0,0)][0] + s_vp[VPOFF(1,0)][0]);
					float lamx = 0.5f*(s_lam[LAMOFF(0,0)] + s_lam[LAMOFF(1,0)]);
					float phixdum =	d_phivxx[npml];
					float phiydum = d_phivyy[npml];
					float phizdum = d_phivzz[npml];

					phixdum = CPML4 (vpx, tex1Dfetch(tex_dumpx2, i), tex1Dfetch(tex_alphax2, i), tex1Dfetch(tex_kappax2, i), phixdum, ds, dt,
					s_vx0[VOFF(0,0)], s_vx0[VOFF(1,0)],
					s_vx0[VOFF(-1,0)], s_vx0[VOFF(2,0)]);
					phiydum = CPML4 (vpx, tex1Dfetch(tex_dumpy, j), tex1Dfetch(tex_alphay, j), tex1Dfetch(tex_kappay, j), phiydum, ds, dt,
					s_vy0[VOFF(0,-1)], s_vy0[VOFF(0,0)],
					s_vy0[VOFF(0,-2)], s_vy0[VOFF(0,1)]);
					phizdum = CPML4 (vpx, tex1Dfetch(tex_dumpz, k), tex1Dfetch(tex_alphaz, k), tex1Dfetch(tex_kappaz, k), phizdum, ds, dt,
					vz0_m1, s_vz0[VOFF(0,0)],
					vz0_m2, vz0_p1);

					d_txx0[offset] += dt*(lamx + 2.0f*mux)*phixdum + dt*lamx*( phiydum + phizdum )
					+ staggards4 (lamx, mux, tex1Dfetch(tex_kappax2, i), tex1Dfetch(tex_kappay, j), tex1Dfetch(tex_kappaz, k), dt, ds,
					s_vx0[VOFF(0,0)], s_vx0[VOFF(1,0)],
					s_vx0[VOFF(-1,0)], s_vx0[VOFF(2,0)],
					s_vy0[VOFF(0,-1)], s_vy0[VOFF(0,0)],
					s_vy0[VOFF(0,-2)], s_vy0[VOFF(0,1)],
					vz0_m1, s_vz0[VOFF(0,0)],
					vz0_m2, vz0_p1);

					d_tyy0[offset] += dt*lamx*( phixdum + phizdum ) + dt*(lamx + 2*mux)*phiydum
					+ staggards4 (lamx, mux, tex1Dfetch(tex_kappay, j), tex1Dfetch(tex_kappax2, i), tex1Dfetch(tex_kappaz, k), dt, ds,
					s_vy0[VOFF(0,-1)], s_vy0[VOFF(0,0)],
					s_vy0[VOFF(0,-2)], s_vy0[VOFF(0,1)],
					s_vx0[VOFF(0,0)], s_vx0[VOFF(1,0)],
					s_vx0[VOFF(-1,0)], s_vx0[VOFF(2,0)],
					vz0_m1, s_vz0[VOFF(0,0)],
					vz0_m2, vz0_p1);

					d_tzz0[offset] += dt*lamx*( phixdum + phiydum ) + dt*(lamx + 2*mux)*phizdum
					+ staggards4 (lamx, mux, tex1Dfetch(tex_kappaz, k), tex1Dfetch(tex_kappax2, i), tex1Dfetch(tex_kappay, j), dt, ds,
					vz0_m1, s_vz0[VOFF(0,0)],
					vz0_m2, vz0_p1,
					s_vx0[VOFF(0,0)], s_vx0[VOFF(1,0)],
					s_vx0[VOFF(-1,0)], s_vx0[VOFF(2,0)],
					s_vy0[VOFF(0,-1)], s_vy0[VOFF(0,0)],
					s_vy0[VOFF(0,-2)], s_vy0[VOFF(0,1)]);

					d_phivxx[npml] = phixdum;
					d_phivyy[npml] = phiydum;
					d_phivzz[npml] = phizdum;

				} // if ( distance_zmin >= 1 && distance_ymin >= 1 && distance_xmax <= 1 )
				/* Calculation of txy */
				if ( distance_ymax >= 1 && distance_xmin >= 1 ){
					float muy = 0.5f*(s_mu[MUOFF(0,0)][0] + s_mu[MUOFF(0,1)][0]);
					float vpy = 0.5f*(s_vp[VPOFF(0,0)][0] + s_vp[VPOFF(0,1)][0]);

					float phixdum =	d_phivyx[npml];
					float phiydum = d_phivxy[npml];

					phixdum = CPML4 (vpy, tex1Dfetch(tex_dumpx, i), tex1Dfetch(tex_alphax, i), tex1Dfetch(tex_kappax, i), phixdum, ds, dt,
					s_vy0[VOFF(-1,0)], s_vy0[VOFF(0,0)],
					s_vy0[VOFF(-2,0)], s_vy0[VOFF(1,0)]);
					phiydum = CPML4 (vpy, tex1Dfetch(tex_dumpy2, j), tex1Dfetch(tex_alphay2, j), tex1Dfetch(tex_kappay2, j), phiydum, ds, dt,
					s_vx0[VOFF(0,0)], s_vx0[VOFF(0,1)],
					s_vx0[VOFF(0,-1)], s_vx0[VOFF(0,2)]);

					d_txy0[offset] += dt*muy*( phixdum + phiydum )
					+ staggardt4 (muy, tex1Dfetch(tex_kappax, i), tex1Dfetch(tex_kappay2, j), dt, ds,
					s_vy0[VOFF(-1,0)], s_vy0[VOFF(0,0)],
					s_vy0[VOFF(-2,0)], s_vy0[VOFF(1,0)],
					s_vx0[VOFF(0,0)], s_vx0[VOFF(0,1)],
					s_vx0[VOFF(0,-1)], s_vx0[VOFF(0,2)]);
					
					d_phivyx[npml] = phixdum;
					d_phivxy[npml] = phiydum;
				}
				/* Calculation of txz */
				if (distance_xmin >= 1 ){

					float muz = 0.5f*(s_mu[MUOFF(0,0)][0] + s_mu[MUOFF(0,0)][1]);
					float vpz = 0.5f*(s_vp[VPOFF(0,0)][0] + s_vp[VPOFF(0,0)][1]);

					float phixdum =	d_phivzx[npml];
					float phizdum = d_phivxz[npml];

					phixdum = CPML4 (vpz, tex1Dfetch(tex_dumpx, i), tex1Dfetch(tex_alphax, i), tex1Dfetch(tex_kappax, i), phixdum, ds, dt,
					s_vz0[VOFF(-1,0)], s_vz0[VOFF(0,0)],
					s_vz0[VOFF(-2,0)], s_vz0[VOFF(1,0)]);
					phizdum = CPML4 (vpz, tex1Dfetch(tex_dumpz2, k), tex1Dfetch(tex_alphaz2, k), tex1Dfetch(tex_kappaz2, k), phizdum, ds, dt,
					s_vx0[VOFF(0,0)], vx0_p1,
					vx0_m1, vx0_p2);

					d_txz0[offset] += dt*muz*( phixdum + phizdum )
					+ staggardt4 (muz, tex1Dfetch(tex_kappax, i), tex1Dfetch(tex_kappaz2, k), dt, ds,
					s_vz0[VOFF(-1,0)], s_vz0[VOFF(0,0)],
					s_vz0[VOFF(-2,0)], s_vz0[VOFF(1,0)],
					s_vx0[VOFF(0,0)], vx0_p1,
					vx0_m1, vx0_p2);

					d_phivzx[npml] = phixdum;
					d_phivxz[npml] = phizdum;
				}
				/* Calculation of tyz */
				if (distance_ymax >= 1){
					// (distance_xmax==0)?mu(i,j,k):mu(i+1,j,k);
					float muxyz = 0.125f*(s_mu[MUOFF(0,0)][0] + s_mu[MUOFF(0,0)][1]
								 + s_mu[MUOFF(0,1)][0] + s_mu[MUOFF(0,1)][1]
								 + s_mu[MUOFF(1,0)][0] + s_mu[MUOFF(1,0)][1]
								 + s_mu[MUOFF(1,1)][0] + s_mu[MUOFF(1,1)][1]);
					float vpxyz = 0.125f*(s_vp[VPOFF(0,0)][0] + s_vp[VPOFF(0,0)][1]
								 + s_vp[VPOFF(0,1)][0] + s_vp[VPOFF(0,1)][1]
								 + s_vp[VPOFF(1,0)][0] + s_vp[VPOFF(1,0)][1]
								 + s_vp[VPOFF(1,1)][0] + s_vp[VPOFF(1,1)][1]);
					float phiydum = d_phivzy[npml];
					float phizdum = d_phivyz[npml];

					phiydum = CPML4 (vpxyz, tex1Dfetch(tex_dumpy2, j), tex1Dfetch(tex_alphay2, j), tex1Dfetch(tex_kappay2, j), phiydum, ds, dt,
					s_vz0[VOFF(0,0)], s_vz0[VOFF(0,1)],
					s_vz0[VOFF(0,-1)], s_vz0[VOFF(0,2)] );
					phizdum = CPML4 (vpxyz, tex1Dfetch(tex_dumpz2, k), tex1Dfetch(tex_alphaz2, k), tex1Dfetch(tex_kappaz2, k), phizdum, ds, dt,
					s_vy0[VOFF(0,0)], vy0_p1,
					vy0_m1, vy0_p2 );

					d_tyz0[offset] += dt*muxyz*( phiydum + phizdum )
					+ staggardt4 (muxyz, tex1Dfetch(tex_kappay2, j), tex1Dfetch(tex_kappaz2, k), dt, ds,
					s_vz0[VOFF(0,0)], s_vz0[VOFF(0,1)],
					s_vz0[VOFF(0,-1)], s_vz0[VOFF(0,2)],
					s_vy0[VOFF(0,0)], vy0_p1,
					vy0_m1, vy0_p2 );

					
					d_phivzy[npml] = phiydum;
					d_phivyz[npml] = phizdum;
				}
			} else {/*}}}*/
				float mux = 0.5f*(s_mu[MUOFF(0,0)][0] + s_mu[MUOFF(1,0)][0]);
				float lamx = 0.5f*(s_lam[LAMOFF(0,0)] + s_lam[LAMOFF(1,0)]);
				float muy = 0.5f*(s_mu[MUOFF(0,0)][0] + s_mu[MUOFF(0,1)][0]);
				float muz = 0.5f*(s_mu[MUOFF(0,0)][0] + s_mu[MUOFF(0,0)][1]);
				float muxyz = 0.125f*(s_mu[MUOFF(0,0)][0] + s_mu[MUOFF(0,0)][1]
						   + s_mu[MUOFF(0,1)][0] + s_mu[MUOFF(0,1)][1]
						   + s_mu[MUOFF(1,0)][0] + s_mu[MUOFF(1,0)][1]
						   + s_mu[MUOFF(1,1)][0] + s_mu[MUOFF(1,1)][1]);

				d_txx0[offset] += (9.f*dt/8.f)*( (lamx+2.f*mux)*(s_vx0[VOFF(1,0)] - s_vx0[VOFF(0,0)]) + lamx*(s_vy0[VOFF(0,0)] - s_vy0[VOFF(0,-1)]) + lamx*(s_vz0[VOFF(0,0)] - vz0_m1) )/ds - (dt/24.f)*( (lamx+2.f*mux)*(s_vx0[VOFF(2,0)] - s_vx0[VOFF(-1,0)]) + lamx*(s_vy0[VOFF(0,1)] - s_vy0[VOFF(0,-2)]) + lamx*(vz0_p1 - vz0_m2) )/ds;
				d_tyy0[offset] += (9.f*dt/8.f)*( (lamx+2.f*mux)*(s_vy0[VOFF(0,0)] - s_vy0[VOFF(0,-1)]) + lamx*(s_vx0[VOFF(1,0)] - s_vx0[VOFF(0,0)]) + lamx*(s_vz0[VOFF(0,0)] - vz0_m1) )/ds - (dt/24.f)*( (lamx+2.f*mux)*(s_vy0[VOFF(0,1)] - s_vy0[VOFF(0,-2)]) + lamx*(s_vx0[VOFF(2,0)] - s_vx0[VOFF(-1,0)]) + lamx*(vz0_p1 - vz0_m2))/ds;
				d_tzz0[offset] += (9.f*dt/8.f)*( (lamx+2.f*mux)*(s_vz0[VOFF(0,0)] - vz0_m1) + lamx*(s_vx0[VOFF(1,0)] - s_vx0[VOFF(0,0)]) + lamx*(s_vy0[VOFF(0,0)] - s_vy0[VOFF(0,-1)]) )/ds - (dt/24.f)*( (lamx+2.f*mux)*(vz0_p1 - vz0_m2) + lamx*(s_vx0[VOFF(2,0)] - s_vx0[VOFF(-1,0)]) + lamx*(s_vy0[VOFF(0,1)] - s_vy0[VOFF(0,-2)]))/ds;
				
				d_txy0[offset] += (9.f*dt*muy/8.f)*((s_vy0[VOFF(0,0)]-s_vy0[VOFF(-1,0)]) + (s_vx0[VOFF(0,1)] - s_vx0[VOFF(0,0)]))/ds - (dt*muy/24.f)*((s_vy0[VOFF(1,0)] - s_vy0[VOFF(-2,0)]) + (s_vx0[VOFF(0,2)]  - s_vx0[VOFF(0,-1)]))/ds;
				d_txz0[offset] += (9.f*dt*muz/8.f)*((s_vz0[VOFF(0,0)]-s_vz0[VOFF(-1,0)]) + (vx0_p1 - s_vx0[VOFF(0,0)]))/ds - (dt*muz/24.f)*((s_vz0[VOFF(1,0)] - s_vz0[VOFF(-2,0)]) + (vx0_p2  - vx0_m1))/ds;
				d_tyz0[offset] += (9.f*dt*muxyz/8.f)*((s_vz0[VOFF(0,1)]-s_vz0[VOFF(0,0)]) + (vy0_p1 - s_vy0[VOFF(0,0)]))/ds - (dt*muxyz/24.f)*((s_vz0[VOFF(0,2)] - s_vz0[VOFF(0,-1)]) + (vy0_p2  - vy0_m1))/ds;
			}
		}/*}}}*/
		// synchro avant de glisser la fen�tre
		__syncthreads();
	}/*}}}*/

// pour k = sizez - 2 (distance_zmax == 1, CPML + ordre 2) ----------------------------------------------->>>
	k = sizez - 2;
	// decalage des donnees
	if (active) {/*{{{*/
		// chaque thread charge une donn�e du domaine
		tx = threadIdx.y*(NPPDX+1) + threadIdx.x;
		// on a d�j� lu les donn�es pour k � l'it�ration pr�c�dente pour mu et vp
		s_mu[tx][0] = s_mu[tx][1];
		s_vp[tx][0] = s_vp[tx][1];
		offset = k*pitch_x*(pitch_y) + j*pitch_x + i;
		s_lam[tx] = d_lam[offset];
		// maintenant, chaque thread charge une donn�e pour k+1 pour mu et vp
		offset = (k+1)*pitch_x*(pitch_y) + j*pitch_x + i;
		s_mu[tx][1] = d_mu[offset];
		s_vp[tx][1] = d_vp[offset];
		// maintenant, on charge les donn�es pour i+1(hors du block)
		if (last_x) {
			// i+1
			tx = threadIdx.y*(NPPDX+1) +threadIdx.x+1;
			s_mu[tx][0] = s_mu[tx][1];
			s_vp[tx][0] = s_vp[tx][1];
			offset = k*pitch_x*(pitch_y)+j*pitch_x + i+1;
			s_lam[tx] = d_lam[offset];
			// i+1, k+1
			offset = (k+1)*pitch_x*(pitch_y)+j*pitch_x + i+1;
			s_mu[tx][1] = d_mu[offset];
			s_vp[tx][1] = d_vp[offset];
		}
		// maintenant, on charge les donn�es pour j+1(hors du block) : uniquement vp et mu
		if (last_y) {
			tx = (threadIdx.y+1)*(NPPDX+1) +threadIdx.x;
			s_mu[tx][0] = s_mu[tx][1];
			s_vp[tx][0] = s_vp[tx][1];
			offset = (k+1)*pitch_x*(pitch_y) + (j+1)*pitch_x + i;
			s_mu[tx][1] = d_mu[offset];
			s_vp[tx][1] = d_vp[offset];
		}
		// maintenant, on charge la donn�e pour j+1 & i+1(hors du block) : uniquement vp et mu
		if (last_y && last_x) {
			tx = (threadIdx.y+1)*(NPPDX+1) + threadIdx.x+1;
			s_mu[tx][0] = s_mu[tx][1];
			s_vp[tx][0] = s_vp[tx][1];
			offset = (k+1)*pitch_x*(pitch_y) + (j+1)*pitch_x + i+1;
			s_mu[tx][1] = d_mu[offset];
			s_vp[tx][1] = d_vp[offset];
		}
		// d�calage des donn�es selon l'axe Z
		tx = (threadIdx.y+2)*(NPPDX+4) + threadIdx.x+2;
		vx0_m1 = s_vx0[tx];
		s_vx0[tx] = vx0_p1;
		vx0_p1 = vx0_p2;

		vy0_m1 = s_vy0[tx];
		s_vy0[tx] = vy0_p1;
		vy0_p1 = vy0_p2;

		vz0_m2 = vz0_m1;
		vz0_m1 = s_vz0[tx];
		s_vz0[tx] = vz0_p1;
		vz0_p1 = vz0_p2;

		// remarque : le tableau d�passe de un point ds chaque direction du domaine calcul�, donc si on est au bord du domaine,
		// i+1 est d�fini et la valeur � ce point est nulle (car hors du domaine)
		// i+2 correspond � la valeur pour j+1 et i=-1, cad 0 car hors du domaine => �a reste coh�rent
		// idem pour i - 1 et i - 2 (i-2 toujours d�fini � cause du padding d'alignement)
		// on charge les deux rang�es i-1 et i-2
		if (threadIdx.x == 0) {
			// i-2
			tx = (threadIdx.y+2)*(NPPDX+4) + 0;
			offset = k*pitch_x*pitch_y + j*pitch_x + i-2;
			s_vx0[tx] = d_vx0[offset];
			s_vy0[tx] = d_vy0[offset];
			s_vz0[tx] = d_vz0[offset];
			// i-1
			tx = (threadIdx.y+2)*(NPPDX+4) + 1;
			offset = k*pitch_x*pitch_y + j*pitch_x + i-1;
			s_vx0[tx] = d_vx0[offset];
			s_vy0[tx] = d_vy0[offset];
			s_vz0[tx] = d_vz0[offset];
		}
		// on charge les deux rang�es i+1 et i+2
		if (last_x) {
			// i+1
			tx = (threadIdx.y+2)*(NPPDX+4) + threadIdx.x + 3;
			offset = k*pitch_x*pitch_y + j*pitch_x + i + 1;
			s_vx0[tx] = d_vx0[offset];
			s_vy0[tx] = d_vy0[offset];
			s_vz0[tx] = d_vz0[offset];
			// i+2
			tx = (threadIdx.y+2)*(NPPDX+4) + threadIdx.x + 4;
			offset = k*pitch_x*pitch_y + j*pitch_x + i + 2;
			s_vx0[tx] = d_vx0[offset];
			s_vy0[tx] = d_vy0[offset];
			s_vz0[tx] = d_vz0[offset];
		}
		// on charge les rang�es j-1 et j-2
		if (threadIdx.y == 0) {
			// j-2
			tx = threadIdx.x+2;
			offset = k*pitch_x*pitch_y + (j-2)*pitch_x + i;
			s_vx0[tx] = d_vx0[offset];
			s_vy0[tx] = d_vy0[offset];
			s_vz0[tx] = d_vz0[offset];
			// j-1
			tx = (NPPDX+4) + threadIdx.x+2;
			offset = k*pitch_x*pitch_y + (j-1)*pitch_x + i;
			s_vx0[tx] = d_vx0[offset];
			s_vy0[tx] = d_vy0[offset];
			s_vz0[tx] = d_vz0[offset];
		}
		// on charge les rang�es j+1 et j+2
		if (last_y) {
			// j+2
			tx = (threadIdx.y + 4)*(NPPDX+4) + threadIdx.x+2;
			offset = k*pitch_x*pitch_y + (j+2)*pitch_x + i;
			s_vx0[tx] = d_vx0[offset];
			s_vy0[tx] = d_vy0[offset];
			s_vz0[tx] = d_vz0[offset];
			// j+1
			tx = (threadIdx.y + 3)*(NPPDX+4) + threadIdx.x+2;
			offset = k*pitch_x*pitch_y + (j+1)*pitch_x + i;
			s_vx0[tx] = d_vx0[offset];
			s_vy0[tx] = d_vy0[offset];
			s_vz0[tx] = d_vz0[offset];
		}
		// on charge les points pour k+2 (acces en mem globale)
		if (distance_zmax < 2) {
			vx0_p2 = 0.f;
			vy0_p2 = 0.f;
			vz0_p2 = 0.f;
		} else {
			vx0_p2 = d_vx0[(k+2)*pitch_x*pitch_y + j*pitch_x + i];
			vy0_p2 = d_vy0[(k+2)*pitch_x*pitch_y + j*pitch_x + i];
			vz0_p2 = d_vz0[(k+2)*pitch_x*pitch_y + j*pitch_x + i];
		}
	}/*}}}*/
	__syncthreads();
	offset = k*pitch_x*pitch_y + j*pitch_x + i;
	// calcul
	if (active) {/*{{{*/
		int npml = d_npml_tab[k*(sizex)*(sizey) + j*(sizex) + i];
		if ( npml >= 0){/*{{{*/
			/* Calculation of txx, tyy and tzz */
			if (distance_ymin >= 1 && distance_xmax >= 1 ){
				float mux = 0.5f*(s_mu[MUOFF(0,0)][0] + s_mu[MUOFF(1,0)][0]);
				float vpx = 0.5f*(s_vp[VPOFF(0,0)][0] + s_vp[VPOFF(1,0)][0]);
				float lamx = 0.5f*(s_lam[LAMOFF(0,0)] + s_lam[LAMOFF(1,0)]);
				
				float phixdum =	d_phivxx[npml];
				float phiydum = d_phivyy[npml];
				float phizdum = d_phivzz[npml];

				phixdum = CPML2 (vpx, tex1Dfetch(tex_dumpx2, i), tex1Dfetch(tex_alphax2, i), tex1Dfetch(tex_kappax2, i), phixdum, ds, dt, s_vx0[VOFF(0,0)], s_vx0[VOFF(1,0)]);
				phiydum = CPML2 (vpx, tex1Dfetch(tex_dumpy, j), tex1Dfetch(tex_alphay, j), tex1Dfetch(tex_kappay, j), phiydum, ds, dt, s_vy0[VOFF(0,-1)], s_vy0[VOFF(0,0)]);
				phizdum = CPML2 (vpx, tex1Dfetch(tex_dumpz, k), tex1Dfetch(tex_alphaz, k), tex1Dfetch(tex_kappaz, k), phizdum, ds, dt, vz0_m1, s_vz0[VOFF(0,0)]);

				d_txx0[offset] += dt*(lamx + 2.0f*mux)*phixdum + dt*lamx*( phiydum + phizdum )
				+ staggards2 (lamx, mux, tex1Dfetch(tex_kappax2, i), tex1Dfetch(tex_kappay, j), tex1Dfetch(tex_kappaz, k), dt, ds,
				s_vx0[VOFF(0,0)], s_vx0[VOFF(1,0)],
				s_vy0[VOFF(0,-1)], s_vy0[VOFF(0,0)], 
				vz0_m1, s_vz0[VOFF(0,0)]);
				
				d_tyy0[offset] += dt*lamx*( phixdum + phizdum ) + dt*(lamx + 2*mux)*phiydum
				+ staggards2 (lamx, mux, tex1Dfetch(tex_kappay, j), tex1Dfetch(tex_kappax2, i), tex1Dfetch(tex_kappaz, k), dt, ds,
				s_vy0[VOFF(0,-1)], s_vy0[VOFF(0,0)],
				s_vx0[VOFF(0,0)], s_vx0[VOFF(1,0)],
				vz0_m1, s_vz0[VOFF(0,0)]);

				d_tzz0[offset] += dt*lamx*( phixdum + phiydum ) + dt*(lamx + 2*mux)*phizdum
				+ staggards2 (lamx, mux, tex1Dfetch(tex_kappaz, k), tex1Dfetch(tex_kappax2, i), tex1Dfetch(tex_kappay, j), dt, ds,
				vz0_m1, s_vz0[VOFF(0,0)],
				s_vx0[VOFF(0,0)], s_vx0[VOFF(1,0)],
				s_vy0[VOFF(0,-1)], s_vy0[VOFF(0,0)]);
				
				d_phivxx[npml] = phixdum;
				d_phivyy[npml] = phiydum;
				d_phivzz[npml] = phizdum;
			} // if ( distance_zmin >= 1 && distance_ymin >= 1 && distance_xmax <= 1 )
			/* Calculation of txy */
			if ( distance_ymax >= 1 && distance_xmin >= 1 ){
				float muy = 0.5f*(s_mu[MUOFF(0,0)][0] + s_mu[MUOFF(0,1)][0]);
				float vpy = 0.5f*(s_vp[VPOFF(0,0)][0] + s_vp[VPOFF(0,1)][0]);

				float phixdum =	d_phivyx[npml];
				float phiydum = d_phivxy[npml];

				phixdum = CPML2 (vpy, tex1Dfetch(tex_dumpx, i), tex1Dfetch(tex_alphax, i), tex1Dfetch(tex_kappax, i), phixdum, ds, dt,
				s_vy0[VOFF(-1,0)], s_vy0[VOFF(0,0)]);

				phiydum = CPML2 (vpy, tex1Dfetch(tex_dumpy2, j), tex1Dfetch(tex_alphay2, j), tex1Dfetch(tex_kappay2, j), phiydum, ds, dt,
				s_vx0[VOFF(0,0)], s_vx0[VOFF(0,1)]);
				
				d_txy0[offset] += dt*muy*( phixdum + phiydum )
				+ staggardt2 (muy, tex1Dfetch(tex_kappax, i), tex1Dfetch(tex_kappay2, j), dt, ds,
				s_vy0[VOFF(-1,0)], s_vy0[VOFF(0,0)],
				s_vx0[VOFF(0,0)], s_vx0[VOFF(0,1)]);

				d_phivyx[npml] = phixdum;
				d_phivxy[npml] = phiydum;
			}
			/* Calculation of txz */
			if (distance_xmin >= 1 ){

				float muz = 0.5f*(s_mu[MUOFF(0,0)][0] + s_mu[MUOFF(0,0)][1]);
				float vpz = 0.5f*(s_vp[VPOFF(0,0)][0] + s_vp[VPOFF(0,0)][1]);

				float phixdum =	d_phivzx[npml];
				float phizdum = d_phivxz[npml];

				phixdum = CPML2 (vpz, tex1Dfetch(tex_dumpx, i), tex1Dfetch(tex_alphax, i), tex1Dfetch(tex_kappax, i), phixdum, ds, dt,
				s_vz0[VOFF(-1,0)], s_vz0[VOFF(0,0)]);
				phizdum = CPML2 (vpz, tex1Dfetch(tex_dumpz2, k), tex1Dfetch(tex_alphaz2, k), tex1Dfetch(tex_kappaz2, k), phizdum, ds, dt,
				s_vx0[VOFF(0,0)], vx0_p1 );

				d_txz0[offset] += dt*muz*( phixdum + phizdum )
				+ staggardt2 (muz, tex1Dfetch(tex_kappax, i), tex1Dfetch(tex_kappaz2, k), dt, ds,
				s_vz0[VOFF(-1,0)], s_vz0[VOFF(0,0)],
				s_vx0[VOFF(0,0)], vx0_p1);

				d_phivzx[npml] = phixdum;
				d_phivxz[npml] = phizdum;
			}
			/* Calculation of tyz */
			if (distance_ymax >= 1){
				// (distance_xmax==0)?mu(i,j,k):mu(i+1,j,k);
				float muxyz = 0.125f*(s_mu[MUOFF(0,0)][0] + s_mu[MUOFF(0,0)][1]
							 + s_mu[MUOFF(0,1)][0] + s_mu[MUOFF(0,1)][1]
							 + s_mu[MUOFF(1,0)][0] + s_mu[MUOFF(1,0)][1]
							 + s_mu[MUOFF(1,1)][0] + s_mu[MUOFF(1,1)][1]);
				float vpxyz = 0.125f*(s_vp[VPOFF(0,0)][0] + s_vp[VPOFF(0,0)][1]
							 + s_vp[VPOFF(0,1)][0] + s_vp[VPOFF(0,1)][1]
							 + s_vp[VPOFF(1,0)][0] + s_vp[VPOFF(1,0)][1]
							 + s_vp[VPOFF(1,1)][0] + s_vp[VPOFF(1,1)][1]);
				float phiydum = d_phivzy[npml];
				float phizdum = d_phivyz[npml];

				phiydum = CPML2 (vpxyz, tex1Dfetch(tex_dumpy2, j), tex1Dfetch(tex_alphay2, j), tex1Dfetch(tex_kappay2, j), phiydum, ds, dt,
				s_vz0[VOFF(0,0)], s_vz0[VOFF(0,1)] );
				phizdum = CPML2 (vpxyz, tex1Dfetch(tex_dumpz2, k), tex1Dfetch(tex_alphaz2, k), tex1Dfetch(tex_kappaz2, k), phizdum, ds, dt,
				s_vy0[VOFF(0,0)], vy0_p1 );

				d_tyz0[offset] += dt*muxyz*( phiydum + phizdum )
				+ staggardt2 (muxyz, tex1Dfetch(tex_kappay2, j), tex1Dfetch(tex_kappaz2, k), dt, ds,
				s_vz0[VOFF(0,0)], s_vz0[VOFF(0,1)],
				s_vy0[VOFF(0,0)], vy0_p1 );
				
				d_phivzy[npml] = phiydum;
				d_phivyz[npml] = phizdum;
			}
		} else {/*}}}*/
			float mux = 0.5f*(s_mu[MUOFF(0,0)][0] + s_mu[MUOFF(1,0)][0]);
			float lamx = 0.5f*(s_lam[LAMOFF(0,0)] + s_lam[LAMOFF(1,0)]);
			float muy = 0.5f*(s_mu[MUOFF(0,0)][0] + s_mu[MUOFF(0,1)][0]);
			float muz = 0.5f*(s_mu[MUOFF(0,0)][0] + s_mu[MUOFF(0,0)][1]);
			float muxyz = 0.125f*(s_mu[MUOFF(0,0)][0] + s_mu[MUOFF(0,0)][1]
					   + s_mu[MUOFF(0,1)][0] + s_mu[MUOFF(0,1)][1]
					   + s_mu[MUOFF(1,0)][0] + s_mu[MUOFF(1,0)][1]
					   + s_mu[MUOFF(1,1)][0] + s_mu[MUOFF(1,1)][1]);

			d_txx0[offset] += staggards2 (lamx, mux, 1.f, 1.f, 1.f, dt, ds,
			s_vx0[VOFF(0,0)], s_vx0[VOFF(1,0)],
			s_vy0[VOFF(0,-1)], s_vy0[VOFF(0,0)],
			vz0_m1, s_vz0[VOFF(0,0)] );

			d_tyy0[offset] += staggards2 (lamx, mux, 1.f, 1.f, 1.f, dt, ds,
			s_vy0[VOFF(0,-1)], s_vy0[VOFF(0,0)],
			s_vx0[VOFF(0,0)], s_vx0[VOFF(1,0)],
			vz0_m1, s_vz0[VOFF(0,0)] );

			d_tzz0[offset] += staggards2 (lamx, mux, 1.f, 1.f, 1.f, dt, ds,
			vz0_m1, s_vz0[VOFF(0,0)],
			s_vx0[VOFF(0,0)], s_vx0[VOFF(1,0)],
			s_vy0[VOFF(0,-1)], s_vy0[VOFF(0,0)] );

			d_txy0[offset] += staggardt2 (muy, 1.f, 1.f, dt, ds,
			s_vy0[VOFF(-1,0)], s_vy0[VOFF(0,0)],
			s_vx0[VOFF(0,0)], s_vx0[VOFF(0,1)] );

			d_txz0[offset] += staggardt2 (muz, 1.f, 1.f, dt, ds,
			s_vz0[VOFF(-1,0)], s_vz0[VOFF(0,0)],
			s_vx0[VOFF(0,0)], vx0_p1 );

			d_tyz0[offset] += staggardt2 (muxyz, 1.f, 1.f, dt, ds,
			s_vz0[VOFF(0,0)], s_vz0[VOFF(0,1)],
			s_vy0[VOFF(0,0)], vy0_p1 );
		}
	}/*}}}*/
	__syncthreads();

// pour k = sizez-1 (surface libre) ---------------------------------------------------------------------->>>
	k++;
	// decalage des donnees
	if (active) {/*{{{*/
		// chaque thread charge une donn�e du domaine
		tx = threadIdx.y*(NPPDX+1) + threadIdx.x;
		// on a d�j� lu les donn�es pour k � l'it�ration pr�c�dente pour mu et vp
		s_mu[tx][0] = s_mu[tx][1];
		s_vp[tx][0] = s_vp[tx][1];
		offset = k*pitch_x*(pitch_y) + j*pitch_x + i;
		s_lam[tx] = d_lam[offset];
		// maintenant, chaque thread charge une donn�e pour k+1 pour mu et vp
		offset = (k+1)*pitch_x*(pitch_y) + j*pitch_x + i;
		s_mu[tx][1] = d_mu[offset];
		s_vp[tx][1] = d_vp[offset];
		// maintenant, on charge les donn�es pour i+1(hors du block)
		if (last_x) {
			// i+1
			tx = threadIdx.y*(NPPDX+1) +threadIdx.x+1;
			s_mu[tx][0] = s_mu[tx][1];
			s_vp[tx][0] = s_vp[tx][1];
			offset = k*pitch_x*(pitch_y)+j*pitch_x + i+1;
			s_lam[tx] = d_lam[offset];
			// i+1, k+1
			offset = (k+1)*pitch_x*(pitch_y)+j*pitch_x + i+1;
			s_mu[tx][1] = d_mu[offset];
			s_vp[tx][1] = d_vp[offset];
		}
		// maintenant, on charge les donn�es pour j+1(hors du block) : uniquement vp et mu
		if (last_y) {
			tx = (threadIdx.y+1)*(NPPDX+1) +threadIdx.x;
			s_mu[tx][0] = s_mu[tx][1];
			s_vp[tx][0] = s_vp[tx][1];
			offset = (k+1)*pitch_x*(pitch_y) + (j+1)*pitch_x + i;
			s_mu[tx][1] = d_mu[offset];
			s_vp[tx][1] = d_vp[offset];
		}
		// maintenant, on charge la donn�e pour j+1 & i+1(hors du block) : uniquement vp et mu
		if (last_y && last_x) {
			tx = (threadIdx.y+1)*(NPPDX+1) + threadIdx.x+1;
			s_mu[tx][0] = s_mu[tx][1];
			s_vp[tx][0] = s_vp[tx][1];
			offset = (k+1)*pitch_x*(pitch_y) + (j+1)*pitch_x + i+1;
			s_mu[tx][1] = d_mu[offset];
			s_vp[tx][1] = d_vp[offset];
		}
		// d�calage des donn�es selon l'axe Z
		tx = (threadIdx.y+2)*(NPPDX+4) + threadIdx.x+2;
		vx0_m1 = s_vx0[tx];
		s_vx0[tx] = vx0_p1;
		vx0_p1 = vx0_p2;

		vy0_m1 = s_vy0[tx];
		s_vy0[tx] = vy0_p1;
		vy0_p1 = vy0_p2;

		vz0_m2 = vz0_m1;
		vz0_m1 = s_vz0[tx];
		s_vz0[tx] = vz0_p1;
		vz0_p1 = vz0_p2;

		// remarque : le tableau d�passe de un point ds chaque direction du domaine calcul�, donc si on est au bord du domaine,
		// i+1 est d�fini et la valeur � ce point est nulle (car hors du domaine)
		// i+2 correspond � la valeur pour j+1 et i=-1, cad 0 car hors du domaine => �a reste coh�rent
		// idem pour i - 1 et i - 2 (i-2 toujours d�fini � cause du padding d'alignement)
		// on charge les deux rang�es i-1 et i-2
		if (threadIdx.x == 0) {
			// i-2
			tx = (threadIdx.y+2)*(NPPDX+4) + 0;
			offset = k*pitch_x*pitch_y + j*pitch_x + i-2;
			s_vx0[tx] = d_vx0[offset];
			s_vy0[tx] = d_vy0[offset];
			s_vz0[tx] = d_vz0[offset];
			// i-1
			tx = (threadIdx.y+2)*(NPPDX+4) + 1;
			offset = k*pitch_x*pitch_y + j*pitch_x + i-1;
			s_vx0[tx] = d_vx0[offset];
			s_vy0[tx] = d_vy0[offset];
			s_vz0[tx] = d_vz0[offset];
		}
		// on charge les deux rang�es i+1 et i+2
		if (last_x) {
			// i+1
			tx = (threadIdx.y+2)*(NPPDX+4) + threadIdx.x + 3;
			offset = k*pitch_x*pitch_y + j*pitch_x + i + 1;
			s_vx0[tx] = d_vx0[offset];
			s_vy0[tx] = d_vy0[offset];
			s_vz0[tx] = d_vz0[offset];
			// i+2
			tx = (threadIdx.y+2)*(NPPDX+4) + threadIdx.x + 4;
			offset = k*pitch_x*pitch_y + j*pitch_x + i + 2;
			s_vx0[tx] = d_vx0[offset];
			s_vy0[tx] = d_vy0[offset];
			s_vz0[tx] = d_vz0[offset];
		}
		// on charge les rang�es j-1 et j-2
		if (threadIdx.y == 0) {
			// j-2
			tx = threadIdx.x+2;
			offset = k*pitch_x*pitch_y + (j-2)*pitch_x + i;
			s_vx0[tx] = d_vx0[offset];
			s_vy0[tx] = d_vy0[offset];
			s_vz0[tx] = d_vz0[offset];
			// j-1
			tx = (NPPDX+4) + threadIdx.x+2;
			offset = k*pitch_x*pitch_y + (j-1)*pitch_x + i;
			s_vx0[tx] = d_vx0[offset];
			s_vy0[tx] = d_vy0[offset];
			s_vz0[tx] = d_vz0[offset];
		}
		// on charge les rang�es j+1 et j+2
		if (last_y) {
			// j+2
			tx = (threadIdx.y + 4)*(NPPDX+4) + threadIdx.x+2;
			offset = k*pitch_x*pitch_y + (j+2)*pitch_x + i;
			s_vx0[tx] = d_vx0[offset];
			s_vy0[tx] = d_vy0[offset];
			s_vz0[tx] = d_vz0[offset];
			// j+1
			tx = (threadIdx.y + 3)*(NPPDX+4) + threadIdx.x+2;
			offset = k*pitch_x*pitch_y + (j+1)*pitch_x + i;
			s_vx0[tx] = d_vx0[offset];
			s_vy0[tx] = d_vy0[offset];
			s_vz0[tx] = d_vz0[offset];
		}
		// on charge les points pour k+2 (acces en mem globale)
		if (distance_zmax < 2) {
			vx0_p2 = 0.f;
			vy0_p2 = 0.f;
			vz0_p2 = 0.f;
		} else {
			vx0_p2 = d_vx0[(k+2)*pitch_x*pitch_y + j*pitch_x + i];
			vy0_p2 = d_vy0[(k+2)*pitch_x*pitch_y + j*pitch_x + i];
			vz0_p2 = d_vz0[(k+2)*pitch_x*pitch_y + j*pitch_x + i];
		}
	}/*}}}*/
	__syncthreads();
	offset = k*pitch_x*pitch_y + j*pitch_x + i;
	// calcul
	if (active) {/*{{{*/
		int npml = d_npml_tab[k*(sizex)*(sizey) + j*(sizex) + i];
		if ( npml >= 0){/*{{{*/
			/* Calculation of txx, tyy and tzz */
			if (distance_ymin >= 1 && distance_xmax >= 1 ){
				float mux = 0.5f*(s_mu[MUOFF(0,0)][0] + s_mu[MUOFF(1,0)][0]);
				float vpx = 0.5f*(s_vp[VPOFF(0,0)][0] + s_vp[VPOFF(1,0)][0]);
				float lamx = 0.5f*(s_lam[LAMOFF(0,0)] + s_lam[LAMOFF(1,0)]);
				
				float b1 = 4.f * mux * (lamx + mux) / (lamx + 2.f*mux);
				float b2 = 2.f * mux * lamx / (lamx + 2.f*mux);
				float phixdum =	d_phivxx[npml];
				float phiydum = d_phivyy[npml];

				phixdum = CPML2 (vpx, tex1Dfetch(tex_dumpx2, i), tex1Dfetch(tex_alphax2, i), tex1Dfetch(tex_kappax2, i), phixdum, ds, dt, s_vx0[VOFF(0,0)], s_vx0[VOFF(1,0)]);
				phiydum = CPML2 (vpx, tex1Dfetch(tex_dumpy, j), tex1Dfetch(tex_alphay, j), tex1Dfetch(tex_kappay, j), phiydum, ds, dt, s_vy0[VOFF(0,-1)], s_vy0[VOFF(0,0)]);

				d_txx0[offset] += b1*dt*phixdum + b2*dt*phiydum
				+ b1*dt*(s_vx0[VOFF(1,0)] - s_vx0[VOFF(0,0)])/(tex1Dfetch(tex_kappax2, i)*ds)
				+ b2*dt*(s_vy0[VOFF(0,0)] - s_vy0[VOFF(0,-1)])/(tex1Dfetch(tex_kappay, j)*ds);

				d_tyy0[offset] += b1*dt*phiydum + b2*dt*phixdum
				+ b1*dt*(s_vy0[VOFF(0,0)] - s_vy0[VOFF(0,-1)])/(tex1Dfetch(tex_kappay, j)*ds)
				+ b2*dt*(s_vx0[VOFF(1,0)] - s_vx0[VOFF(0,0)])/(tex1Dfetch(tex_kappax2, i)*ds);
			
				d_tzz0[offset] = 0.f;
				
				d_phivxx[npml] = phixdum;
				d_phivyy[npml] = phiydum;
			} // if ( distance_zmin >= 1 && distance_ymin >= 1 && distance_xmax <= 1 )
			/* Calculation of txy */
			if ( distance_ymax >= 1 && distance_xmin >= 1 ){
				float muy = 0.5f*(s_mu[MUOFF(0,0)][0] + s_mu[MUOFF(0,1)][0]);
				float vpy = 0.5f*(s_vp[VPOFF(0,0)][0] + s_vp[VPOFF(0,1)][0]);

				float phixdum =	d_phivyx[npml];
				float phiydum = d_phivxy[npml];

				phixdum = CPML2 (vpy, tex1Dfetch(tex_dumpx, i), tex1Dfetch(tex_alphax, i), tex1Dfetch(tex_kappax, i), phixdum, ds, dt,
				s_vy0[VOFF(-1,0)], s_vy0[VOFF(0,0)]);

				phiydum = CPML2 (vpy, tex1Dfetch(tex_dumpy2, j), tex1Dfetch(tex_alphay2, j), tex1Dfetch(tex_kappay2, j), phiydum, ds, dt,
				s_vx0[VOFF(0,0)], s_vx0[VOFF(0,1)]);
				
				d_txy0[offset] += dt*muy*( phixdum + phiydum )
				+ staggardt2 (muy, tex1Dfetch(tex_kappax, i), tex1Dfetch(tex_kappay2, j), dt, ds,
				s_vy0[VOFF(-1,0)], s_vy0[VOFF(0,0)],
				s_vx0[VOFF(0,0)], s_vx0[VOFF(0,1)]);

				d_phivyx[npml] = phixdum;
				d_phivxy[npml] = phiydum;
			}
			d_txz0[offset] = - d_txz0[(k-1)*pitch_x*pitch_y + j*pitch_x + i]; // calcul� � l'it�ration pr�c�dente
			d_tyz0[offset] = - d_txz0[(k-1)*pitch_x*pitch_y + j*pitch_x + i]; // calcul� � l'it�ration pr�c�dente
		} else {/*}}}*/
			float mux = 0.5f*(s_mu[MUOFF(0,0)][0] + s_mu[MUOFF(1,0)][0]);
			float lamx = 0.5f*(s_lam[LAMOFF(0,0)] + s_lam[LAMOFF(1,0)]);
			float muy = 0.5f*(s_mu[MUOFF(0,0)][0] + s_mu[MUOFF(0,1)][0]);
			float b1 = 4.f * mux * (lamx + mux) / (lamx + 2.f*mux);
			float b2 = 2.f * mux * lamx / (lamx + 2.f*mux);

			d_txx0[offset] += b1*dt*(s_vx0[VOFF(1,0)]-s_vx0[VOFF(0,0)])/ds + b2*dt*(s_vy0[VOFF(0,0)] - s_vy0[VOFF(0,-1)])/ds;
			d_tyy0[offset] += b1*dt*(s_vy0[VOFF(0,0)]-s_vy0[VOFF(0,-1)])/ds	+ b2*dt*(s_vx0[VOFF(1,0)]-s_vx0[VOFF(0,0)])/ds;
			d_tzz0[offset] = 0.f;

			d_txy0[offset] += staggardt2 (muy, 1.f, 1.f, dt, ds, s_vy0[VOFF(-1,0)], s_vy0[VOFF(0,0)], s_vx0[VOFF(0,0)], s_vx0[VOFF(0,1)]);
		}
	}/*}}}*/
	return;
}
// }}}

// WRAPPER {{{
void computeStress3D (	float* d_txx0, float* d_tyy0, float* d_tzz0, float* d_txy0, float* d_txz0, float* d_tyz0,
			float* d_vx0, float* d_vy0, float* d_vz0,
			int* d_npml_tab, float* d_phivxx, float* d_phivxy, float* d_phivxz, float* d_phivyx, float* d_phivyy, float* d_phivyz, float* d_phivzx, float* d_phivzy, float* d_phivzz, 
			float* d_mu, float* d_lam, float* d_vp, 
			int sizex, int sizey, int sizez,
			int pitch_x, int pitch_y, int pitch_z, 
			float ds, float dt, int delta, int compute_external,
			int grid_x, int grid_y, int grid_z, int block_x, int block_y, int block_z, int position) {

#ifdef ENABLE_VERY_SLOW_ERROR_CHECKING
	hipDeviceSynchronize();
	printCudaErr(hipGetLastError(), "before computeStress kernel");
#endif
	dim3 grid_dim(grid_x,grid_y,grid_z);
	dim3 block_dim(block_x, block_y, block_z);
	compute_stress_3d <<< grid_dim, block_dim, 0 >>> (	d_txx0, d_tyy0, d_tzz0, d_txy0, d_txz0, d_tyz0,
							d_vx0, d_vy0, d_vz0,
							d_npml_tab, d_phivxx, d_phivxy, d_phivxz, d_phivyx, d_phivyy, d_phivyz, d_phivzx, d_phivzy, d_phivzz, 
							d_mu, d_lam, d_vp, 
							sizex, sizey, sizez,
							pitch_x, pitch_y, pitch_z, 
							ds, dt, delta, (compute_external)?true:false, position);
#ifdef ENABLE_VERY_SLOW_ERROR_CHECKING
	hipDeviceSynchronize();
	printCudaErr(hipGetLastError(), "after computeStress kernel");
#endif
}
// }}}
// }}}

// COMPUTE VELOCITY {{{
// IMPLEMENTATION {{{
__global__ void compute_veloc_3d (	float* d_txx0, float* d_tyy0, float* d_tzz0, float* d_txy0, float* d_txz0, float* d_tyz0,
							float* d_vx0, float* d_vy0, float* d_vz0,
							float* d_fx, float* d_fy, float* d_fz, 
							int* d_npml_tab, float* d_phitxxx, float* d_phitxyy, float* d_phitxzz, float *d_phitxyx, float *d_phityyy, float *d_phityzz, float *d_phitxzx, float *d_phityzy, float *d_phitzzz,
							float* d_vp, float* d_rho,
							int sizex, int sizey, int sizez,
							int pitch_x, int pitch_y, int pitch_z, 
							float ds, float dt, int delta, bool compute_external, int position)
{
	__shared__ float s_rho[(NPPDX_K2+1)*(NPPDY_K2+1)][2];
	__shared__ float s_vp[(NPPDX_K2+1)*(NPPDY_K2+1)][2];

	__shared__ float s_txx0[(NPPDX_K2+4)*(NPPDY_K2+4)];
	__shared__ float s_tyy0[(NPPDX_K2+4)*(NPPDY_K2+4)];
	__shared__ float s_tzz0[(NPPDX_K2+4)*(NPPDY_K2+4)];
	__shared__ float s_txy0[(NPPDX_K2+4)*(NPPDY_K2+4)];
	__shared__ float s_txz0[(NPPDX_K2+4)*(NPPDY_K2+4)];
	__shared__ float s_tyz0[(NPPDX_K2+4)*(NPPDY_K2+4)];
	
	// m1 pour k-1, m2 pour k-2, p1 pour k+1, p2 pour k+2
	float tzz0_m1, tzz0_p1, tzz0_p2;
	float txz0_m1, txz0_m2, txz0_p1, txz0_ip1_km1;
	float tyz0_m1, tyz0_m2, tyz0_p1, tyz0_jm1_km1;
	
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	int j = blockIdx.y*blockDim.y + threadIdx.y;
	int distance_xmin = i;
	int distance_xmax = sizex - i - 1;
	int distance_ymin = j;
	int distance_ymax = sizey - j - 1;
	int offset, offset_source, tx;
	bool last_x, last_y;

	// seulement les blocs internes/externes travaillent suivant le parametre compute_external
	// s'il n'y a qu'un point dans le dernier bloc en x ou en y, on en prend deux.
	bool active =	(compute_external && 
				(blockIdx.x == 0 || blockIdx.x >= ((blockDim.x*gridDim.x-sizex == blockDim.x-1)?gridDim.x-2:gridDim.x-1) || blockIdx.y == 0 || blockIdx.y >= ((blockDim.y*gridDim.y-sizey == blockDim.y-1)?gridDim.y-2:gridDim.y-1)))
		||	(!compute_external && 
				!(blockIdx.x == 0 || blockIdx.x >= ((blockDim.x*gridDim.x-sizex == blockDim.x-1)?gridDim.x-2:gridDim.x-1) || blockIdx.y == 0 || blockIdx.y >= ((blockDim.y*gridDim.y-sizey == blockDim.y-1)?gridDim.y-2:gridDim.y-1))); 

	//on ne calcule pas les points qui sont en dehors du domaine. Les threads correspondants ne font rien.	
	active = active && ((distance_xmax >=0 && distance_ymax >=0)?true:false);
	last_x = last_y = false;
	if (distance_xmax == 0 || (active && threadIdx.x == (NPPDX-1))) {
		last_x = true;
	}
	if (distance_ymax == 0 || (active && threadIdx.y == (NPPDY-1))) {
		last_y = true;
	}

	// ici distance pour le modele global (on ne s'interesse qu'aux bords, donc si on n'est pas pres du bord, une valeur quelconque >2 suffit)
	distance_xmin = (position & MASK_FIRST_X)?i:DUMMY_VALUE;
	distance_xmax = (position & MASK_LAST_X)?(sizex - i - 1):DUMMY_VALUE;
	distance_ymin = (position & MASK_FIRST_Y)?j:DUMMY_VALUE;
	distance_ymax = (position & MASK_LAST_Y)?(sizey - j - 1):DUMMY_VALUE;


	
	// chargement initial des valeurs dans les registres et en memoire partagee
	if (active) {/*{{{*/
		// le tableau est initialis� � z�ro et ces �l�ments ne sont jamais mis � jour
		tzz0_m1 = 0.f;
		txz0_m1 = txz0_m2 = 0.f;
		tyz0_m1 = tyz0_m2 = 0.f;
		txz0_ip1_km1 = 0.f;
		tyz0_jm1_km1 = 0.f;

		offset = pitch_x*pitch_y + j*pitch_x + i;
		tzz0_p1 = d_tzz0[offset];
		txz0_p1 = d_txz0[offset];
		tyz0_p1 = d_tyz0[offset];

		offset = 2*pitch_x*pitch_y + j*pitch_x + i;
		tzz0_p2 = d_tzz0[offset];

		// chaque thread charge une donn�e du domaine pour k=0
		tx = threadIdx.y*(NPPDX_K2+1) + threadIdx.x;
		offset = j*pitch_x + i;
		s_rho[tx][0] = d_rho[offset];
		s_vp[tx][0] = d_vp[offset];
		// maintenant, chaque thread charge une donn�e pour k=1 pour rho et vp
		offset = pitch_x*(pitch_y) + j*pitch_x + i;
		s_rho[tx][1] = d_rho[offset];
		s_vp[tx][1] = d_vp[offset];
		// maintenant, on charge les donn�es pour i+1(hors du block)
		if (last_x) {
			tx = threadIdx.y*(NPPDX_K2+1) +threadIdx.x+1;
			offset = j*pitch_x + i+1;
			s_rho[tx][0] = d_rho[offset];
			s_vp[tx][0] = d_vp[offset];
			// i+1, k+1
			offset = pitch_x*(pitch_y) + j*pitch_x + i+1;
			s_rho[tx][1] = d_rho[offset];
			s_vp[tx][1] = d_vp[offset];
		}
		// maintenant, on charge les donn�es pour j+1(hors du block) : uniquement vp et rho
		if (last_y) {
			tx = (threadIdx.y+1)*(NPPDX_K2+1) + threadIdx.x;
			offset = (j+1)*pitch_x + i;
			s_rho[tx][0] = d_rho[offset];
			s_vp[tx][0] = d_vp[offset];
			offset = pitch_x*(pitch_y) + (j+1)*pitch_x + i;
			s_rho[tx][1] = d_rho[offset];
			s_vp[tx][1] = d_vp[offset];
		}
		// maintenant, on charge la donn�e pour j+1 & i+1(hors du block) : uniquement vp et rho
		if (last_y && last_x) {
			tx = (threadIdx.y+1)*(NPPDX_K2+1) + threadIdx.x+1;
			offset = (j+1)*pitch_x + i+1;
			s_rho[tx][0] = d_rho[offset];
			s_vp[tx][0] = d_vp[offset];
			offset = pitch_x*(pitch_y) + (j+1)*pitch_x + i+1;
			s_rho[tx][1] = d_rho[offset];
			s_vp[tx][1] = d_vp[offset];
		}
		// vx0, vy0, vz0
		// chaque thread charge sa valeur en shmem
		tx = (threadIdx.y+2)*(NPPDX_K2+4) + threadIdx.x+2;
		offset = j*pitch_x + i;
		s_txx0[tx] = d_txx0[offset];
		s_tyy0[tx] = d_tyy0[offset];
		s_tzz0[tx] = d_tzz0[offset];
		s_txy0[tx] = d_txy0[offset];
		s_txz0[tx] = d_txz0[offset];
		s_tyz0[tx] = d_tyz0[offset];
		// on charge les deux rang�es i-1 et i-2
		if (threadIdx.x == 0) {
			// i-2
			tx = (threadIdx.y+2)*(NPPDX_K2+4) + 0;
			offset = j*pitch_x + i-2;
			s_txx0[tx] = d_txx0[offset];
			s_tyy0[tx] = d_tyy0[offset];
			s_tzz0[tx] = d_tzz0[offset];
			s_txy0[tx] = d_txy0[offset];
			s_txz0[tx] = d_txz0[offset];
			s_tyz0[tx] = d_tyz0[offset];
			// i-1
			tx = (threadIdx.y+2)*(NPPDX_K2+4) + 1;
			offset = j*pitch_x + i-1;
			s_txx0[tx] = d_txx0[offset];
			s_tyy0[tx] = d_tyy0[offset];
			s_tzz0[tx] = d_tzz0[offset];
			s_txy0[tx] = d_txy0[offset];
			s_txz0[tx] = d_txz0[offset];
			s_tyz0[tx] = d_tyz0[offset];
		}
		// on charge les deux rang�es i+1 et i+2
		if (last_x) {
			// i+1
			tx = (threadIdx.y+2)*(NPPDX_K2+4) + threadIdx.x + 3;
			offset = j*pitch_x + i + 1;
			s_txx0[tx] = d_txx0[offset];
			s_tyy0[tx] = d_tyy0[offset];
			s_tzz0[tx] = d_tzz0[offset];
			s_txy0[tx] = d_txy0[offset];
			s_txz0[tx] = d_txz0[offset];
			s_tyz0[tx] = d_tyz0[offset];
			// i+2
			tx = (threadIdx.y+2)*(NPPDX_K2+4) + threadIdx.x + 4;
			offset = j*pitch_x + i + 2;
			s_txx0[tx] = d_txx0[offset];
			s_tyy0[tx] = d_tyy0[offset];
			s_tzz0[tx] = d_tzz0[offset];
			s_txy0[tx] = d_txy0[offset];
			s_txz0[tx] = d_txz0[offset];
			s_tyz0[tx] = d_tyz0[offset];
		}
		// on charge les rang�es j-1 et j-2
		if (threadIdx.y == 0) {
			// j-2
			tx = threadIdx.x+2;
			offset = (j-2)*pitch_x + i;
			s_txx0[tx] = d_txx0[offset];
			s_tyy0[tx] = d_tyy0[offset];
			s_tzz0[tx] = d_tzz0[offset];
			s_txy0[tx] = d_txy0[offset];
			s_txz0[tx] = d_txz0[offset];
			s_tyz0[tx] = d_tyz0[offset];
			// j-1
			tx = (NPPDX_K2+4) + threadIdx.x+2;
			offset = (j-1)*pitch_x + i;
			s_txx0[tx] = d_txx0[offset];
			s_tyy0[tx] = d_tyy0[offset];
			s_tzz0[tx] = d_tzz0[offset];
			s_txy0[tx] = d_txy0[offset];
			s_txz0[tx] = d_txz0[offset];
			s_tyz0[tx] = d_tyz0[offset];
		}
		// on charge les rang�es j+1 et j+2
		if (last_y) {
			// j+2
			tx = (threadIdx.y + 4)*(NPPDX_K2+4) + threadIdx.x+2;
			offset = (j+2)*pitch_x + i;
			s_txx0[tx] = d_txx0[offset];
			s_tyy0[tx] = d_tyy0[offset];
			s_tzz0[tx] = d_tzz0[offset];
			s_txy0[tx] = d_txy0[offset];
			s_txz0[tx] = d_txz0[offset];
			s_tyz0[tx] = d_tyz0[offset];
			// j+1
			tx = (threadIdx.y + 3)*(NPPDX_K2+4) + threadIdx.x+2;
			offset = (j+1)*pitch_x + i;
			s_txx0[tx] = d_txx0[offset];
			s_tyy0[tx] = d_tyy0[offset];
			s_tzz0[tx] = d_tzz0[offset];
			s_txy0[tx] = d_txy0[offset];
			s_txz0[tx] = d_txz0[offset];
			s_tyz0[tx] = d_tyz0[offset];
		}
	}/*}}}*/
	int npml=-2;
	float rhoxy, rhoxz;
	// boucle sur z
	for (int k = 0; k < sizez; k++) {/*{{{*/
		int distance_zmin = k;
		int distance_zmax = sizez - k - 1;
		
		// apres decalage de la fenetre, on decalle les valeurs selon l'axe des Z.
		if (active) {/*{{{*/
			if (k>0) {
				// chaque thread charge une donn�e du domaine
				tx = threadIdx.y*(NPPDX_K2+1) + threadIdx.x;
				// on a d�j� lu les donn�es pour k � l'it�ration pr�c�dente pour rho et vp
				s_rho[tx][0] = s_rho[tx][1];
				s_vp[tx][0] = s_vp[tx][1];
				// maintenant, chaque thread charge une donn�e pour k+1 pour rho et vp
				offset = (k+1)*pitch_x*(pitch_y) + j*pitch_x + i;
				s_rho[tx][1] = d_rho[offset];
				s_vp[tx][1] = d_vp[offset];
				// maintenant, on charge les donn�es pour i+1(hors du block)
				if (last_x) {
					// i+1
					tx = threadIdx.y*(NPPDX_K2+1) +threadIdx.x+1;
					s_rho[tx][0] = s_rho[tx][1];
					s_vp[tx][0] = s_vp[tx][1];
					// i+1, k+1
					offset = (k+1)*pitch_x*(pitch_y)+j*pitch_x + i+1;
					s_rho[tx][1] = d_rho[offset];
					s_vp[tx][1] = d_vp[offset];
				}
				// maintenant, on charge les donn�es pour j+1(hors du block) : uniquement vp et rho
				if (last_y) {
					tx = (threadIdx.y+1)*(NPPDX_K2+1) +threadIdx.x;
					s_rho[tx][0] = s_rho[tx][1];
					s_vp[tx][0] = s_vp[tx][1];
					offset = (k+1)*pitch_x*(pitch_y) + (j+1)*pitch_x + i;
					s_rho[tx][1] = d_rho[offset];
					s_vp[tx][1] = d_vp[offset];
				}
				// maintenant, on charge la donn�e pour j+1 & i+1(hors du block) : uniquement vp et rho
				if (last_y && last_x) {
					tx = (threadIdx.y+1)*(NPPDX_K2+1) + threadIdx.x+1;
					s_rho[tx][0] = s_rho[tx][1];
					s_vp[tx][0] = s_vp[tx][1];
					offset = (k+1)*pitch_x*(pitch_y) + (j+1)*pitch_x + i+1;
					s_rho[tx][1] = d_rho[offset];
					s_vp[tx][1] = d_vp[offset];
				}
				// d�calage des donn�es selon l'axe Z
				txz0_ip1_km1 = s_txz0[(threadIdx.y+2)*(NPPDX_K2+4) + threadIdx.x+2+1];
				tyz0_jm1_km1 = s_tyz0[(threadIdx.y+2-1)*(NPPDX_K2+4) + threadIdx.x+2];
			}
		}/*}}}*/
		// synchro avant d'ecraser s_txz0 et s_tyz0
		__syncthreads();
		if (active) {/*{{{*/
			if (k>0) {/*{{{*/
				tx = (threadIdx.y+2)*(NPPDX_K2+4) + threadIdx.x+2;
				tzz0_m1 = s_tzz0[tx];
				s_tzz0[tx] = tzz0_p1;
				tzz0_p1 = tzz0_p2;
				
				txz0_m2 = txz0_m1;
				txz0_m1 = s_txz0[tx];
				s_txz0[tx] = txz0_p1;
				
				tyz0_m2 = tyz0_m1;
				tyz0_m1 = s_tyz0[tx];
				s_tyz0[tx] = tyz0_p1;

				// remarque : le tableau d�passe de un point ds chaque direction du domaine calcul�, donc si on est au bord du domaine,
				// i+1 est d�fini et la valeur � ce point est nulle (car hors du domaine)
				// i+2 correspond � la valeur pour j+1 et i=-1, cad 0 car hors du domaine => �a reste coh�rent
				// idem pour i - 1 et i - 2 (i-2 toujours d�fini � cause du padding d'alignement)
				tx = (threadIdx.y+2)*(NPPDX_K2+4) + threadIdx.x+2;
				offset = k*pitch_x*pitch_y + j*pitch_x + i;
				s_txx0[tx] = d_txx0[offset];
				s_tyy0[tx] = d_tyy0[offset];
				// ! ici !!! : txy a la place de tyz
				s_txy0[tx] = d_txy0[offset];
				// on charge les deux rang�es i-1 et i-2
				if (threadIdx.x == 0) {
					// i-2
					tx = (threadIdx.y+2)*(NPPDX_K2+4) + 0;
					offset = k*pitch_x*pitch_y + j*pitch_x + i-2;
					s_txx0[tx] = d_txx0[offset];
					s_tyy0[tx] = d_tyy0[offset];
					s_tzz0[tx] = d_tzz0[offset];
					s_txy0[tx] = d_txy0[offset];
					s_txz0[tx] = d_txz0[offset];
					s_tyz0[tx] = d_tyz0[offset];
					// i-1
					tx = (threadIdx.y+2)*(NPPDX_K2+4) + 1;
					offset = k*pitch_x*pitch_y + j*pitch_x + i-1;
					s_txx0[tx] = d_txx0[offset];
					s_tyy0[tx] = d_tyy0[offset];
					s_tzz0[tx] = d_tzz0[offset];
					s_txy0[tx] = d_txy0[offset];
					s_txz0[tx] = d_txz0[offset];
					s_tyz0[tx] = d_tyz0[offset];
				}
				// on charge les deux rang�es i+1 et i+2
				if (last_x) {
					// i+1
					tx = (threadIdx.y+2)*(NPPDX_K2+4) + threadIdx.x + 3;
					offset = k*pitch_x*pitch_y + j*pitch_x + i + 1;
					s_txx0[tx] = d_txx0[offset];
					s_tyy0[tx] = d_tyy0[offset];
					s_tzz0[tx] = d_tzz0[offset];
					s_txy0[tx] = d_txy0[offset];
					s_txz0[tx] = d_txz0[offset];
					s_tyz0[tx] = d_tyz0[offset];
					// i+2
					tx = (threadIdx.y+2)*(NPPDX_K2+4) + threadIdx.x + 4;
					offset = k*pitch_x*pitch_y + j*pitch_x + i + 2;
					s_txx0[tx] = d_txx0[offset];
					s_tyy0[tx] = d_tyy0[offset];
					s_tzz0[tx] = d_tzz0[offset];
					s_txy0[tx] = d_txy0[offset];
					s_txz0[tx] = d_txz0[offset];
					s_tyz0[tx] = d_tyz0[offset];
				}
				// on charge les rang�es j-1 et j-2
				if (threadIdx.y == 0) {
					// j-2
					tx = threadIdx.x+2;
					offset = k*pitch_x*pitch_y + (j-2)*pitch_x + i;
					s_txx0[tx] = d_txx0[offset];
					s_tyy0[tx] = d_tyy0[offset];
					s_tzz0[tx] = d_tzz0[offset];
					s_txy0[tx] = d_txy0[offset];
					s_txz0[tx] = d_txz0[offset];
					s_tyz0[tx] = d_tyz0[offset];
					// j-1
					tx = (NPPDX_K2+4) + threadIdx.x+2;
					offset = k*pitch_x*pitch_y + (j-1)*pitch_x + i;
					s_txx0[tx] = d_txx0[offset];
					s_tyy0[tx] = d_tyy0[offset];
					s_tzz0[tx] = d_tzz0[offset];
					s_txy0[tx] = d_txy0[offset];
					s_txz0[tx] = d_txz0[offset];
					s_tyz0[tx] = d_tyz0[offset];
				}
				// on charge les rang�es j+1 et j+2
				if (last_y) {
					// j+2
					tx = (threadIdx.y + 4)*(NPPDX_K2+4) + threadIdx.x+2;
					offset = k*pitch_x*pitch_y + (j+2)*pitch_x + i;
					s_txx0[tx] = d_txx0[offset];
					s_tyy0[tx] = d_tyy0[offset];
					s_tzz0[tx] = d_tzz0[offset];
					s_txy0[tx] = d_txy0[offset];
					s_txz0[tx] = d_txz0[offset];
					s_tyz0[tx] = d_tyz0[offset];
					// j+1
					tx = (threadIdx.y + 3)*(NPPDX_K2+4) + threadIdx.x+2;
					offset = k*pitch_x*pitch_y + (j+1)*pitch_x + i;
					s_txx0[tx] = d_txx0[offset];
					s_tyy0[tx] = d_tyy0[offset];
					s_tzz0[tx] = d_tzz0[offset];
					s_txy0[tx] = d_txy0[offset];
					s_txz0[tx] = d_txz0[offset];
					s_tyz0[tx] = d_tyz0[offset];
				}
				// on charge les points pour k+2 (acces en mem globale)
				if (distance_zmax < 2) {
					tzz0_p2 = 0.f;
				} else {
					tzz0_p2 = d_tzz0[(k+2)*pitch_x*pitch_y + j*pitch_x + i];
				}
				if (distance_zmax < 1) {
					txz0_p1 = 0.f;
					tyz0_p1 = 0.f;
				} else {
					txz0_p1 = d_txz0[(k+1)*pitch_x*pitch_y + j*pitch_x + i];
					tyz0_p1 = d_tyz0[(k+1)*pitch_x*pitch_y + j*pitch_x + i];
				}
			}/*}}}*/
		}/*}}}*/
		
		// synchro : on attends que tous les threads aient fini d'ecrire dans la memoire partagee
		__syncthreads();
		
		offset = k*pitch_x*pitch_y + j*pitch_x + i;
		offset_source = k*pitch_x*pitch_y + j*pitch_x + i;
	
		if (active) {/*{{{*/
			npml = d_npml_tab[k*(sizex)*(sizey) + j*(sizex) + i];
		 	if (npml >= 0) {/*{{{*/
				//CPML
				// ICI !!!!!!!!
				if (distance_zmin >= 1 && distance_ymin >= 1 && distance_xmin >= 1) {/* VX {{{*/
					/* Calculation of vx */
					if ( distance_zmax == 0 ){ /* free surface */
						float phixdum = d_phitxxx[npml];
						float phiydum = d_phitxyy[npml];
						float phizdum = d_phitxzz[npml];

						phixdum = CPML2 (s_vp[VPOFF(0,0)][0], tex1Dfetch(tex_dumpx, i), tex1Dfetch(tex_alphax, i), tex1Dfetch(tex_kappax, i), phixdum, ds, dt, s_txx0[TOFF(-1,0)], s_txx0[TOFF(0,0)] );
						phiydum = CPML2 (s_vp[VPOFF(0,0)][0], tex1Dfetch(tex_dumpy, j), tex1Dfetch(tex_alphay, j), tex1Dfetch(tex_kappay, j), phiydum, ds, dt, s_txy0[TOFF(0,-1)], s_txy0[TOFF(0,0)] );
						phizdum = CPML2 (s_vp[VPOFF(0,0)][0], tex1Dfetch(tex_dumpz, k), tex1Dfetch(tex_alphaz, k), tex1Dfetch(tex_kappaz, k), phizdum, ds, dt, txz0_m1, - txz0_m1 );

						if (distance_xmax == 0 || distance_ymax ==0) { // boundary condition
							d_vx0[offset] = 0.0f;
						} else {
							d_vx0[offset] += (dt/s_rho[RHOFF(0,0)][0])*( phixdum + phiydum + phizdum )
							+ staggardv2 (1.f/s_rho[RHOFF(0,0)][0], tex1Dfetch(tex_kappax, i), tex1Dfetch(tex_kappay, j), tex1Dfetch(tex_kappaz, k), dt, ds,
							s_txx0[TOFF(-1,0)], s_txx0[TOFF(0,0)],
							s_txy0[TOFF(0,-1)], s_txy0[TOFF(0,0)],
							txz0_m1, - txz0_m1 );
						}
						
						d_phitxxx[npml] = phixdum;
						d_phitxyy[npml] = phiydum;
						d_phitxzz[npml] = phizdum;
					} else if ( distance_zmax == 1 ){ /* in the first cell, 2nd order finite-difference instead of 4th order finite-difference */
						float phixdum = d_phitxxx[npml];
						float phiydum = d_phitxyy[npml];
						float phizdum = d_phitxzz[npml];

						phixdum = CPML2 (s_vp[VPOFF(0,0)][0], tex1Dfetch(tex_dumpx, i), tex1Dfetch(tex_alphax, i), tex1Dfetch(tex_kappax, i), phixdum, ds, dt,
						s_txx0[TOFF(-1,0)], s_txx0[TOFF(0,0)] );
						phiydum = CPML2 (s_vp[VPOFF(0,0)][0], tex1Dfetch(tex_dumpy, j), tex1Dfetch(tex_alphay, j), tex1Dfetch(tex_kappay, j), phiydum, ds, dt,
						s_txy0[TOFF(0,-1)], s_txy0[TOFF(0,0)] );
						phizdum = CPML2 (s_vp[VPOFF(0,0)][0], tex1Dfetch(tex_dumpz, k), tex1Dfetch(tex_alphaz, k), tex1Dfetch(tex_kappaz, k), phizdum, ds, dt,
						txz0_m1, s_txz0[TOFF(0,0)] );
						
						if (distance_xmax == 0 || distance_ymax ==0) { // boundary condition
							d_vx0[offset] = 0.0f;
						} else {
							d_vx0[offset] += (dt/s_rho[RHOFF(0,0)][0])*( phixdum + phiydum + phizdum )
							+ staggardv2 (1.f/s_rho[RHOFF(0,0)][0], tex1Dfetch(tex_kappax, i), tex1Dfetch(tex_kappay, j), tex1Dfetch(tex_kappaz, k), dt, ds,
							s_txx0[TOFF(-1,0)], s_txx0[TOFF(0,0)],
							s_txy0[TOFF(0,-1)], s_txy0[TOFF(0,0)],
							txz0_m1, s_txz0[TOFF(0,0)] );
						}
						
						d_phitxxx[npml] = phixdum;
						d_phitxyy[npml] = phiydum;
						d_phitxzz[npml] = phizdum;
					} else { /* regular domain */
						// ICI !!!!!!!!
						float phixdum = d_phitxxx[npml];
						float phiydum = d_phitxyy[npml];
						float phizdum = d_phitxzz[npml];

						phixdum = CPML4 (s_vp[VPOFF(0,0)][0], tex1Dfetch(tex_dumpx, i), tex1Dfetch(tex_alphax, i), tex1Dfetch(tex_kappax, i), phixdum, ds, dt,
						s_txx0[TOFF(-1,0)], s_txx0[TOFF(0,0)],
						s_txx0[TOFF(-2,0)], s_txx0[TOFF(1,0)] );
						phiydum = CPML4 (s_vp[VPOFF(0,0)][0], tex1Dfetch(tex_dumpy, j), tex1Dfetch(tex_alphay, j), tex1Dfetch(tex_kappay, j), phiydum, ds, dt,
						s_txy0[TOFF(0,-1)], s_txy0[TOFF(0,0)],
						s_txy0[TOFF(0,-2)], s_txy0[TOFF(0,1)] );
						phizdum = CPML4 (s_vp[VPOFF(0,0)][0], tex1Dfetch(tex_dumpz, k), tex1Dfetch(tex_alphaz, k), tex1Dfetch(tex_kappaz, k), phizdum, ds, dt,
						txz0_m1, s_txz0[TOFF(0,0)],
						txz0_m2, txz0_p1 );

						if (distance_xmax == 0 || distance_ymax ==0) { // boundary condition
							d_vx0[offset] = 0.0f;
						} else {
							d_vx0[offset] += (dt/s_rho[RHOFF(0,0)][0])*( phixdum + phiydum + phizdum )
							+ staggardv4 (1.f/s_rho[RHOFF(0,0)][0], tex1Dfetch(tex_kappax, i), tex1Dfetch(tex_kappay, j), tex1Dfetch(tex_kappaz, k), dt, ds,
							s_txx0[TOFF(-1,0)], s_txx0[TOFF(0,0)],
							s_txx0[TOFF(-2,0)], s_txx0[TOFF(1,0)],
							s_txy0[TOFF(0,-1)], s_txy0[TOFF(0,0)],
							s_txy0[TOFF(0,-2)], s_txy0[TOFF(0,1)],
							txz0_m1, s_txz0[TOFF(0,0)],
							txz0_m2, txz0_p1 );
						}
							
						d_phitxxx[npml] = phixdum;
						d_phitxyy[npml] = phiydum;
						d_phitxzz[npml] = phizdum;
					} /* end of if "free surface" */
				}/*}}}*/
				/* Calculation of vy */
				if ( distance_zmin >= 1 && distance_ymax >= 1 && distance_xmax >= 1 ){/*VY{{{*/
					rhoxy = 0.25f*(s_rho[RHOFF(0,0)][0] + s_rho[RHOFF(0,1)][0]
								+ s_rho[RHOFF(1,0)][0] + s_rho[RHOFF(1,1)][0]);
					float vpxy = 0.25f*(s_vp[VPOFF(0,0)][0] + s_vp[VPOFF(0,1)][0]
							   + s_vp[VPOFF(1,0)][0] + s_vp[VPOFF(1,1)][0]);
					if ( distance_zmax == 0 ){ /* free surface */
						float phixdum = d_phitxyx[npml];
						float phiydum = d_phityyy[npml];
						float phizdum = d_phityzz[npml];

						phixdum = CPML2 (vpxy, tex1Dfetch(tex_dumpx2, i), tex1Dfetch(tex_alphax2, i), tex1Dfetch(tex_kappax2, i), phixdum, ds, dt,
						s_txy0[TOFF(0,0)], s_txy0[TOFF(1,0)] );
						phiydum = CPML2 (vpxy, tex1Dfetch(tex_dumpy2, j), tex1Dfetch(tex_alphay2, j), tex1Dfetch(tex_kappay2, j), phiydum, ds, dt,
						s_tyy0[TOFF(0,0)], s_tyy0[TOFF(0,1)] );
						phizdum = CPML2 (vpxy, tex1Dfetch(tex_dumpz, k), tex1Dfetch(tex_alphaz, k), tex1Dfetch(tex_kappaz, k), phizdum, ds, dt,
						tyz0_m1, - tyz0_m1 );
						
						if (distance_xmin == 0 || distance_ymin == 0) { // boundary condition
							d_vy0[offset] = 0.0f;
						} else {
							d_vy0[offset] += (dt/rhoxy)*( phixdum + phiydum + phizdum )
							+ staggardv2 (1.f/rhoxy, tex1Dfetch(tex_kappax2, i), tex1Dfetch(tex_kappay2, j), tex1Dfetch(tex_kappaz, k), dt, ds,
							s_txy0[TOFF(0,0)], s_txy0[TOFF(1,0)],
							s_tyy0[TOFF(0,0)], s_tyy0[TOFF(0,1)],
							tyz0_m1, - tyz0_m1 );
						}
						
						d_phitxyx[npml] = phixdum;
						d_phityyy[npml] = phiydum;
						d_phityzz[npml] = phizdum;
					} else if ( distance_zmax == 1 ){ /* in the first cell, 2nd order finite-difference instead of 4th order finite-difference */
						float phixdum = d_phitxyx[npml];
						float phiydum = d_phityyy[npml];
						float phizdum = d_phityzz[npml];

						phixdum = CPML2 (vpxy, tex1Dfetch(tex_dumpx2, i), tex1Dfetch(tex_alphax2, i), tex1Dfetch(tex_kappax2, i), phixdum, ds, dt,
						s_txy0[TOFF(0,0)], s_txy0[TOFF(1,0)] );
						phiydum = CPML2 (vpxy, tex1Dfetch(tex_dumpy2, j), tex1Dfetch(tex_alphay2, j), tex1Dfetch(tex_kappay2, j), phiydum, ds, dt,
						s_tyy0[TOFF(0,0)], s_tyy0[TOFF(0,1)] );
						phizdum = CPML2 (vpxy, tex1Dfetch(tex_dumpz, k), tex1Dfetch(tex_alphaz, k), tex1Dfetch(tex_kappaz, k), phizdum, ds, dt,
						tyz0_m1, s_tyz0[TOFF(0,0)] );

						if (distance_xmin == 0 || distance_ymin == 0) { // boundary condition
							d_vy0[offset] = 0.0f;
						} else {
							d_vy0[offset] += (dt/rhoxy)*( phixdum + phiydum + phizdum )
							+ staggardv2 (1.f/rhoxy, tex1Dfetch(tex_kappax2, i), tex1Dfetch(tex_kappay2, j), tex1Dfetch(tex_kappaz, k), dt, ds,
							s_txy0[TOFF(0,0)], s_txy0[TOFF(1,0)],
							s_tyy0[TOFF(0,0)], s_tyy0[TOFF(0,1)],
							tyz0_m1, s_tyz0[TOFF(0,0)] );
						}

						d_phitxyx[npml] = phixdum;
						d_phityyy[npml] = phiydum;
						d_phityzz[npml] = phizdum;
					} else { /* regular domain */
						float phixdum = d_phitxyx[npml];
						float phiydum = d_phityyy[npml];
						float phizdum = d_phityzz[npml];
						
						phixdum = CPML4 (vpxy, tex1Dfetch(tex_dumpx2, i), tex1Dfetch(tex_alphax2, i), tex1Dfetch(tex_kappax2, i), phixdum, ds, dt,
						s_txy0[TOFF(0,0)], s_txy0[TOFF(1,0)],
						s_txy0[TOFF(-1,0)], s_txy0[TOFF(2,0)] );
						phiydum = CPML4 (vpxy, tex1Dfetch(tex_dumpy2, j), tex1Dfetch(tex_alphay2, j), tex1Dfetch(tex_kappay2, j), phiydum, ds, dt,
						s_tyy0[TOFF(0,0)], s_tyy0[TOFF(0,1)],
						s_tyy0[TOFF(0,-1)], s_tyy0[TOFF(0,2)] );
						phizdum = CPML4 (vpxy, tex1Dfetch(tex_dumpz, k), tex1Dfetch(tex_alphaz, k), tex1Dfetch(tex_kappaz, k), phizdum, ds, dt,
						tyz0_m1, s_tyz0[TOFF(0,0)],
						tyz0_m2, tyz0_p1 );

						if (distance_xmin == 0 || distance_ymin == 0) { // boundary condition
							d_vy0[offset] = 0.0f;
						} else {
							d_vy0[offset] += (dt/rhoxy)*( phixdum + phiydum + phizdum )
							+ staggardv4 (1.f/rhoxy, tex1Dfetch(tex_kappax2, i), tex1Dfetch(tex_kappay2, j), tex1Dfetch(tex_kappaz, k), dt, ds,
							s_txy0[TOFF(0,0)], s_txy0[TOFF(1,0)],
							s_txy0[TOFF(-1,0)], s_txy0[TOFF(2,0)],
							s_tyy0[TOFF(0,0)], s_tyy0[TOFF(0,1)],
							s_tyy0[TOFF(0,-1)], s_tyy0[TOFF(0,2)],
							tyz0_m1, s_tyz0[TOFF(0,0)],
							tyz0_m2, tyz0_p1 );
						}
						
						d_phitxyx[npml] = phixdum;
						d_phityyy[npml] = phiydum;
						d_phityzz[npml] = phizdum;
					} /* end of if "free surface" */
				}/*}}}*/
				/* Calculation of vz */
				if ( distance_ymin >= 1 && distance_xmax >= 1 ){/*VZ{{{*/
					rhoxz = 0.25f*(s_rho[RHOFF(0,0)][0] + s_rho[RHOFF(0,0)][1]
								+ s_rho[RHOFF(1,0)][0] + s_rho[RHOFF(1,0)][1]);
					float vpxz = 0.25f*(s_vp[VPOFF(0,0)][0] + s_vp[VPOFF(0,0)][1]
							   + s_vp[VPOFF(1,0)][0] + s_vp[VPOFF(1,0)][1]);
					if ( distance_zmax == 0 ){ /* free surface */
						float phixdum = d_phitxzx[npml];
						float phiydum = d_phityzy[npml];
						float phizdum = d_phitzzz[npml];

						phixdum = CPML2 (vpxz, tex1Dfetch(tex_dumpx2, i), tex1Dfetch(tex_alphax2, i), tex1Dfetch(tex_kappax2, i), phixdum, ds, dt,
						- txz0_m1, - txz0_ip1_km1 );
						phiydum = CPML2 (vpxz, tex1Dfetch(tex_dumpy, j), tex1Dfetch(tex_alphay, j), tex1Dfetch(tex_kappay, j), phiydum, ds, dt,
						- tyz0_jm1_km1, - tyz0_m1 );
						phizdum = CPML2 (vpxz, tex1Dfetch(tex_dumpz2, k), tex1Dfetch(tex_alphaz2, k), tex1Dfetch(tex_kappaz2, k), phizdum, ds, dt,
						s_tzz0[TOFF(0,0)], - tzz0_m1 );

						if (distance_xmin == 0 || distance_ymax ==0) { // boundary condition
							d_vz0[offset] = 0.0f;
						} else {
							d_vz0[offset] += (dt/rhoxz)*( phixdum + phiydum + phizdum )
							+ staggardv2 (1.f/rhoxz, tex1Dfetch(tex_kappax2, i), tex1Dfetch(tex_kappay, j), tex1Dfetch(tex_kappaz2, k), dt, ds,
							- txz0_m1, - txz0_ip1_km1,
							- tyz0_jm1_km1, - tyz0_m1,
							s_tzz0[TOFF(0,0)], - tzz0_m1 );
						}
											
						d_phitxzx[npml] = phixdum;
						d_phityzy[npml] = phiydum;
						d_phitzzz[npml] = phizdum;
					} else if ( distance_zmax == 1 ){ /* in the first cell, 2nd order finite-difference instead of 4th order finite-difference */
						float phixdum = d_phitxzx[npml];
						float phiydum = d_phityzy[npml];
						float phizdum = d_phitzzz[npml];

						phixdum = CPML2 (vpxz, tex1Dfetch(tex_dumpx2, i), tex1Dfetch(tex_alphax2, i), tex1Dfetch(tex_kappax2, i), phixdum, ds, dt,
						s_txz0[TOFF(0,0)], s_txz0[TOFF(1,0)] );
						phiydum = CPML2 (vpxz, tex1Dfetch(tex_dumpy, j), tex1Dfetch(tex_alphay, j), tex1Dfetch(tex_kappay, j), phiydum, ds, dt,
						s_tyz0[TOFF(0,-1)], s_tyz0[TOFF(0,0)] );
						phizdum = CPML2 (vpxz, tex1Dfetch(tex_dumpz2, k), tex1Dfetch(tex_alphaz2, k), tex1Dfetch(tex_kappaz2, k), phizdum, ds, dt,
						s_tzz0[TOFF(0,0)], tzz0_p1 );

						
						if (distance_xmin == 0 || distance_ymax ==0) { // boundary condition
							d_vz0[offset] = 0.0f;
						} else {
							d_vz0[offset] += (dt/rhoxz)*( phixdum + phiydum + phizdum )
							+ staggardv2 (1.f/rhoxz, tex1Dfetch(tex_kappax2, i), tex1Dfetch(tex_kappay, j), tex1Dfetch(tex_kappaz2, k), dt, ds,
							s_txz0[TOFF(0,0)], s_txz0[TOFF(1,0)],
							s_tyz0[TOFF(0,-1)], s_tyz0[TOFF(0,0)],
							s_tzz0[TOFF(0,0)], tzz0_p1 );
						}
						
						d_phitxzx[npml] = phixdum;
						d_phityzy[npml] = phiydum;
						d_phitzzz[npml] = phizdum;
					} else { /* regular domain */
						float phixdum = d_phitxzx[npml];
						float phiydum = d_phityzy[npml];
						float phizdum = d_phitzzz[npml];

						phixdum = CPML4 (vpxz, tex1Dfetch(tex_dumpx2, i), tex1Dfetch(tex_alphax2, i), tex1Dfetch(tex_kappax2, i), phixdum, ds, dt,
						s_txz0[TOFF(0,0)], s_txz0[TOFF(1,0)],
						s_txz0[TOFF(-1,0)], s_txz0[TOFF(2,0)] );
						phiydum = CPML4 (vpxz, tex1Dfetch(tex_dumpy, j), tex1Dfetch(tex_alphay, j), tex1Dfetch(tex_kappay, j), phiydum, ds, dt,
						s_tyz0[TOFF(0,-1)], s_tyz0[TOFF(0,0)],
						s_tyz0[TOFF(0,-2)], s_tyz0[TOFF(0,1)] );
						phizdum = CPML4 (vpxz, tex1Dfetch(tex_dumpz2, k), tex1Dfetch(tex_alphaz2, k), tex1Dfetch(tex_kappaz2, k), phizdum, ds, dt,
						s_tzz0[TOFF(0,0)], tzz0_p1,
						tzz0_m1, tzz0_p2 );

						
						if (distance_xmin == 0 || distance_ymax ==0 || distance_zmin == 0) { // boundary condition
							d_vz0[offset] = 0.0f;
						} else {
							d_vz0[offset] += (dt/rhoxz)*( phixdum + phiydum + phizdum )
							+ staggardv4 (1.f/rhoxz, tex1Dfetch(tex_kappax2, i), tex1Dfetch(tex_kappay, j), tex1Dfetch(tex_kappaz2, k), dt, ds,
							s_txz0[TOFF(0,0)], s_txz0[TOFF(1,0)],
							s_txz0[TOFF(-1,0)], s_txz0[TOFF(2,0)],
							s_tyz0[TOFF(0,-1)], s_tyz0[TOFF(0,0)],
							s_tyz0[TOFF(0,-2)], s_tyz0[TOFF(0,1)],
							s_tzz0[TOFF(0,0)], tzz0_p1,
							tzz0_m1, tzz0_p2 );
						}
						
						d_phitxzx[npml] = phixdum;
						d_phityzy[npml] = phiydum;
						d_phitzzz[npml] = phizdum;
					} /* end of if "free surface" */
				}/*}}}*/
			/* Normal mode }}}*/
			} else {/*{{{*/
				rhoxy = 0.25f*(s_rho[RHOFF(0,0)][0] + s_rho[RHOFF(0,1)][0]
						  + s_rho[RHOFF(1,0)][0] + s_rho[RHOFF(1,1)][0]);
				rhoxz = 0.25f*(s_rho[RHOFF(0,0)][0] + s_rho[RHOFF(0,0)][1]
						  + s_rho[RHOFF(1,0)][0] + s_rho[RHOFF(1,0)][1]);
				if (distance_xmin == 0 || distance_xmax == 0 || distance_ymin == 0 || distance_ymax == 0 || distance_zmin == 0) {
                                       d_vx0[offset] = 0.f;
                                       d_vy0[offset] = 0.f;
                                       d_vz0[offset] = 0.f;
				} else if ( distance_zmax == 0 ){ /* free surface */
					d_vx0[offset] += (1.f/s_rho[RHOFF(0,0)][0])*d_fx[offset_source]*dt/ds
					+ staggardv2 (1.f/s_rho[RHOFF(0,0)][0], 1.f, 1.f, 1.f, dt, ds,
					s_txx0[TOFF(-1,0)], s_txx0[TOFF(0,0)],
					s_txy0[TOFF(0,-1)], s_txy0[TOFF(0,0)],
					txz0_m1, - txz0_m1 );

					d_vy0[offset] += (1.f/rhoxy)*d_fy[offset_source]*dt/ds
					+ staggardv2 (1.f/rhoxy, 1.f, 1.f, 1.f, dt, ds,
					s_txy0[TOFF(0,0)], s_txy0[TOFF(1,0)],
					s_tyy0[TOFF(0,0)], s_tyy0[TOFF(0,1)],
					tyz0_m1, - tyz0_m1 );

					d_vz0[offset] += (1.f/rhoxz)*d_fz[offset_source]*dt/ds
					+ staggardv2 (1.f/rhoxz, 1.f, 1.f, 1.f, dt, ds,
					- txz0_m1, - txz0_ip1_km1,
					- tyz0_jm1_km1, - tyz0_m1,
					s_tzz0[TOFF(0,0)], - tzz0_m1 );
				} else if ( distance_zmax == 1 ){ /* in the first cell, 2nd order finite-difference instead of 4th order finite-difference */
					d_vx0[offset] += (1.f/s_rho[RHOFF(0,0)][0])*d_fx[offset_source]*dt/ds
					+ staggardv2 (1.f/s_rho[RHOFF(0,0)][0], 1.f, 1.f, 1.f, dt, ds,
					s_txx0[TOFF(-1,0)], s_txx0[TOFF(0,0)],
					s_txy0[TOFF(0,-1)], s_txy0[TOFF(0,0)],
					txz0_m1, s_txz0[TOFF(0,0)] );

					d_vy0[offset] += (1.f/rhoxy)*d_fy[offset_source]*dt/ds
					+ staggardv2 (1.f/rhoxy, 1.f, 1.f, 1.f, dt, ds,
					s_txy0[TOFF(0,0)], s_txy0[TOFF(1,0)],
					s_tyy0[TOFF(0,0)], s_tyy0[TOFF(0,1)],
					tyz0_m1, s_tyz0[TOFF(0,0)] );

					d_vz0[offset] += (1.f/rhoxz)*d_fz[offset_source]*dt/ds
					+ staggardv2 (1.f/rhoxz, 1.f, 1.f, 1.f, dt, ds,
					s_txz0[TOFF(0,0)], s_txz0[TOFF(1,0)],
					s_tyz0[TOFF(0,-1)], s_tyz0[TOFF(0,0)],
					s_tzz0[TOFF(0,0)], tzz0_p1 );
				} else { /* regular domain */

					d_vx0[offset] += (1.f/s_rho[RHOFF(0,0)][0])*d_fx[offset_source]*dt/ds
					+ staggardv4 (1.f/s_rho[RHOFF(0,0)][0], 1.f, 1.f, 1.f, dt, ds,
					s_txx0[TOFF(-1,0)], s_txx0[TOFF(0,0)],
					s_txx0[TOFF(-2,0)], s_txx0[TOFF(1,0)],
					s_txy0[TOFF(0,-1)], s_txy0[TOFF(0,0)],
					s_txy0[TOFF(0,-2)], s_txy0[TOFF(0,1)],
					txz0_m1, s_txz0[TOFF(0,0)],
					txz0_m2, txz0_p1 );

					d_vy0[offset] += (1.f/rhoxy)*d_fy[offset_source]*dt/ds
					+ staggardv4 (1.f/rhoxy, 1.f, 1.f, 1.f, dt, ds,
					s_txy0[TOFF(0,0)], s_txy0[TOFF(1,0)],
					s_txy0[TOFF(-1,0)], s_txy0[TOFF(2,0)],
					s_tyy0[TOFF(0,0)], s_tyy0[TOFF(0,1)],
					s_tyy0[TOFF(0,-1)], s_tyy0[TOFF(0,2)],
					tyz0_m1, s_tyz0[TOFF(0,0)],
					tyz0_m2, tyz0_p1 );

					d_vz0[offset] += (1.f/rhoxz)*d_fz[offset_source]*dt/ds
					+ staggardv4 (1.f/rhoxz, 1.f, 1.f, 1.f, dt, ds,
					s_txz0[TOFF(0,0)], s_txz0[TOFF(1,0)],
					s_txz0[TOFF(-1,0)], s_txz0[TOFF(2,0)],
					s_tyz0[TOFF(0,-1)], s_tyz0[TOFF(0,0)],
					s_tyz0[TOFF(0,-2)], s_tyz0[TOFF(0,1)],
					s_tzz0[TOFF(0,0)], tzz0_p1,
					tzz0_m1, tzz0_p2 );
				} /* end of if "free surface" */
			}/*}}}*/ /* end of normal mode */
		} // end of active/*}}}*/
		__syncthreads();
	}/*}}}*/
}
// }}}

// WRAPPER {{{
void computeVeloc3D (	float* d_txx0, float* d_tyy0, float* d_tzz0, float* d_txy0, float* d_txz0, float* d_tyz0,
			float* d_vx0, float* d_vy0, float* d_vz0,
			float* d_fx, float* d_fy, float* d_fz, 
			int* d_npml_tab, float* d_phitxxx, float* d_phitxyy, float* d_phitxzz, float *d_phitxyx, float *d_phityyy, float *d_phityzz, float *d_phitxzx, float *d_phityzy, float *d_phitzzz,
			float* d_vp, float* d_rho,
			int sizex, int sizey, int sizez,
			int pitch_x, int pitch_y, int pitch_z, 
			float ds, float dt, int delta, int compute_external,
			int grid_x, int grid_y, int grid_z, int block_x, int block_y, int block_z, int position) {

#ifdef ENABLE_VERY_SLOW_ERROR_CHECKING
	hipDeviceSynchronize();
	printCudaErr(hipGetLastError(), "before computeVeloc kernel");
#endif
	dim3 grid_dim(grid_x,grid_y,grid_z);
	dim3 block_dim(block_x, block_y, block_z);
	compute_veloc_3d <<< grid_dim, block_dim, 0 >>> (	d_txx0, d_tyy0, d_tzz0, d_txy0, d_txz0, d_tyz0,
							d_vx0, d_vy0, d_vz0,
							d_fx, d_fy, d_fz, 
							d_npml_tab, d_phitxxx, d_phitxyy, d_phitxzz, d_phitxyx, d_phityyy, d_phityzz, d_phitxzx, d_phityzy, d_phitzzz,
							d_vp, d_rho,
							sizex, sizey, sizez,
							pitch_x, pitch_y, pitch_z, 
							ds, dt, delta, (compute_external)?true:false, position);

#ifdef ENABLE_VERY_SLOW_ERROR_CHECKING
	hipDeviceSynchronize();
	printCudaErr(hipGetLastError(), "after computeVeloc kernel");
#endif

}
// }}}
// }}}
// }}}
// }}}

// KERNELS FOR MPI BUFFERS MANAGEMENT {{{
// GET BUFFERS STRESS {{{
// IMPLEMENTATION {{{
__global__ void get_buffers_stress(	float* d_buff_x_min, float* d_buff_x_max, float* d_buff_y_min, float* d_buff_y_max, 
					int size_x, int size_y, int size_z, int pitch_x, int pitch_y, int pitch_z, 
					float* d_Txx, float* d_Tyy, float* d_Tzz, float* d_Txy, float* d_Txz, float* d_Tyz, int size_buf_x, int size_buf_y, int position) {
	bool active;
	int size, coord, z_coord, offset, index, size_buf;
	float* p_buf;
	bool fill_buffer=true;

	// coord in X or Y direction
	coord = blockIdx.x*blockDim.x+threadIdx.x;
	// coord in Z direction
	int face = (int) (blockIdx.y/size_z);
	z_coord = blockIdx.y - (face*size_z);

	if (face < 2) {
		if (face == 0) {	// ymin
			offset = 0;
			p_buf = d_buff_y_min;
			if (position & MASK_FIRST_Y) fill_buffer = false;
		} else {		// ymax
			offset = (size_y-2)*pitch_x;
			p_buf = d_buff_y_max;
			if (position & MASK_LAST_Y) fill_buffer = false;
		}
		size = size_x;
		index = z_coord*pitch_x*pitch_y + threadIdx.y*pitch_x + offset + coord;
		size_buf = size_buf_y;
	} else {
		if (face == 2) {	// xmin
			offset = 0;
			p_buf = d_buff_x_min;
			if (position & MASK_FIRST_X) fill_buffer = false;
		} else {		// xmax
			offset = (size_x-2);
			p_buf = d_buff_x_max;
			if (position & MASK_LAST_X) fill_buffer = false;
		}
		size = size_y;
		index = z_coord*pitch_x*pitch_y + coord*pitch_x + threadIdx.y + offset;
		size_buf = size_buf_x;
	}

	active = (coord<size)?true:false;

	if (active && fill_buffer) {
		p_buf[0*size_buf + z_coord*2*size + threadIdx.y*size + coord] = d_Txx[index];
		p_buf[1*size_buf + z_coord*2*size + threadIdx.y*size + coord] = d_Tyy[index];
		p_buf[2*size_buf + z_coord*2*size + threadIdx.y*size + coord] = d_Tzz[index];
		p_buf[3*size_buf + z_coord*2*size + threadIdx.y*size + coord] = d_Txy[index];
		p_buf[4*size_buf + z_coord*2*size + threadIdx.y*size + coord] = d_Txz[index];
		p_buf[5*size_buf + z_coord*2*size + threadIdx.y*size + coord] = d_Tyz[index];
	}
	return;
}
// }}}

// WRAPPER {{{
void getBuffersStress(	float* d_buff_x_min, float* d_buff_x_max, float* d_buff_y_min, float* d_buff_y_max, 
			int size_x, int size_y, int size_z, int pitch_x, int pitch_y, int pitch_z, 
			float* d_Txx, float* d_Tyy, float* d_Tzz, float* d_Txy, float* d_Txz, float* d_Tyz, int size_buf_x, int size_buf_y,
			int grid_x, int grid_y, int grid_z, int block_x, int block_y, int block_z, int position){

#ifdef ENABLE_VERY_SLOW_ERROR_CHECKING
	hipDeviceSynchronize();
	printCudaErr(hipGetLastError(), "before getBuffersStress kernel");
#endif
	dim3 grid_dim(grid_x,grid_y*grid_z,1);
	dim3 block_dim(block_x, block_y);
	get_buffers_stress <<< grid_dim, block_dim >>> (	d_buff_x_min, d_buff_x_max, d_buff_y_min, d_buff_y_max, 
								size_x, size_y, size_z, pitch_x, pitch_y, pitch_z, 
								d_Txx, d_Tyy, d_Tzz, d_Txy, d_Txz, d_Tyz, size_buf_x, size_buf_y, position);

#ifdef ENABLE_VERY_SLOW_ERROR_CHECKING
	hipDeviceSynchronize();
	printCudaErr(hipGetLastError(), "after getBuffersStress kernel");
#endif
}
// }}}
// }}}

// GET BUFFERS VELOC {{{
// IMPLEMENTATION {{{
__global__ void get_buffers_veloc(	float* d_buff_x_min, float* d_buff_x_max, float* d_buff_y_min, float* d_buff_y_max, 
					int size_x, int size_y, int size_z, int pitch_x, int pitch_y, int pitch_z, 
					float* d_Vx, float* d_Vy, float* d_Vz, int size_buf_x, int size_buf_y, int position) {

	bool active;
	int size, coord, z_coord, offset, index, size_buf;
	float* p_buf;
	bool fill_buffer=true;

	// coord in X or Y direction
	coord = blockIdx.x*blockDim.x+threadIdx.x;
	// coord in Z direction
	int face = (int) (blockIdx.y/size_z);
	z_coord = blockIdx.y - (face*size_z);

	if (face < 2) {
		if (face == 0) {	// ymin
			offset = 0;
			p_buf = d_buff_y_min;
			if (position & MASK_FIRST_Y) fill_buffer = false;
		} else {		// ymax
			offset = (size_y-2)*pitch_x;
			p_buf = d_buff_y_max;
			if (position & MASK_LAST_Y) fill_buffer = false;
		}
		size = size_x;
		index = z_coord*pitch_x*pitch_y + threadIdx.y*pitch_x + offset + coord;
		size_buf = size_buf_y;
	} else {
		if (face == 2) {	// xmin
			offset = 0;
			p_buf = d_buff_x_min;
			if (position & MASK_FIRST_X) fill_buffer = false;
		} else {		// xmax
			offset = (size_x-2);
			p_buf = d_buff_x_max;
			if (position & MASK_LAST_X) fill_buffer = false;
		}
		size = size_y;
		index = z_coord*pitch_x*pitch_y + coord*pitch_x + threadIdx.y + offset;
		size_buf = size_buf_x;
	}

	active = (coord<size)?true:false;

	if (active && fill_buffer) {
		p_buf[0*size_buf + z_coord*2*size + threadIdx.y*size + coord] = d_Vx[index];
		p_buf[1*size_buf + z_coord*2*size + threadIdx.y*size + coord] = d_Vy[index];
		p_buf[2*size_buf + z_coord*2*size + threadIdx.y*size + coord] = d_Vz[index];
	}

	return;
}
// }}}

// WRAPPER {{{
void getBuffersVeloc(	float* d_buff_x_min, float* d_buff_x_max, float* d_buff_y_min, float* d_buff_y_max, 
			int size_x, int size_y, int size_z, int pitch_x, int pitch_y, int pitch_z, 
			float* d_Vx, float* d_Vy, float* d_Vz, int size_buf_x, int size_buf_y,
			int grid_x, int grid_y, int grid_z, int block_x, int block_y, int block_z, int position) {

#ifdef ENABLE_VERY_SLOW_ERROR_CHECKING
	hipDeviceSynchronize();
	printCudaErr(hipGetLastError(), "before computeStress kernel");
#endif
	dim3 grid_dim(grid_x,grid_y*grid_z,1);
	dim3 block_dim(block_x, block_y, block_z);
	get_buffers_veloc <<< grid_dim, block_dim, 0 >>> (	d_buff_x_min, d_buff_x_max, d_buff_y_min, d_buff_y_max, 
								size_x, size_y, size_z, pitch_x, pitch_y, pitch_z, 
								d_Vx, d_Vy, d_Vz, size_buf_x, size_buf_y, position);

#ifdef ENABLE_VERY_SLOW_ERROR_CHECKING
	hipDeviceSynchronize();
	printCudaErr(hipGetLastError(), "after computeStress kernel");
#endif
}
// }}}
// }}}

// UPDATE HALOS STRESS {{{
// IMPLEMENTATION {{{
__global__ void update_halo_stress(	float* d_buff_x_min, float* d_buff_x_max, float* d_buff_y_min, float* d_buff_y_max, 
					int size_x, int size_y, int size_z, int pitch_x, int pitch_y, int pitch_z, 
					float* d_Txx, float* d_Tyy, float* d_Tzz, float* d_Txy, float* d_Txz, float* d_Tyz, int size_buf_x, int size_buf_y, int position) {

	bool active;
	int size, coord, z_coord, offset, index, size_buf;
	float* p_buf;
	bool update=true;

	// coord in X or Y direction
	coord = blockIdx.x*blockDim.x+threadIdx.x;
	// coord in Z direction
	int face = (int) (blockIdx.y/size_z);
	z_coord = blockIdx.y - (face*size_z);

	if (face < 2) {
		if (face == 0) {	// ymin
			offset = -2*pitch_x;
			p_buf = d_buff_y_min;
			if (position & MASK_FIRST_Y) update = false;
		} else {		// ymax
			offset = size_y*pitch_x;
			p_buf = d_buff_y_max;
			if (position & MASK_LAST_Y) update = false;
		}
		size = size_x;
		index = z_coord*pitch_x*pitch_y + threadIdx.y*pitch_x + offset + coord;
		size_buf = size_buf_y;
	} else {
		if (face == 2) {	// xmin
			offset = -2;
			p_buf = d_buff_x_min;
			if (position & MASK_FIRST_X) update = false;
		} else {		// xmax
			offset = size_x;
			p_buf = d_buff_x_max;
			if (position & MASK_LAST_X) update = false;
		}
		size = size_y;
		index = z_coord*pitch_x*pitch_y + coord*pitch_x + threadIdx.y + offset;
		size_buf = size_buf_x;
	}

	active = (coord<size)?true:false;

	if (active && update) {
		d_Txx[index] = p_buf[0*size_buf + z_coord*2*size + threadIdx.y*size + coord];
		d_Tyy[index] = p_buf[1*size_buf + z_coord*2*size + threadIdx.y*size + coord];
		d_Tzz[index] = p_buf[2*size_buf + z_coord*2*size + threadIdx.y*size + coord];
		d_Txy[index] = p_buf[3*size_buf + z_coord*2*size + threadIdx.y*size + coord];
		d_Txz[index] = p_buf[4*size_buf + z_coord*2*size + threadIdx.y*size + coord];
		d_Tyz[index] = p_buf[5*size_buf + z_coord*2*size + threadIdx.y*size + coord];
	}
	return;
}
// }}}

// WRAPPER {{{
void updateHaloStress(	float* d_buff_x_min, float* d_buff_x_max, float* d_buff_y_min, float* d_buff_y_max, 
			int size_x, int size_y, int size_z, int pitch_x, int pitch_y, int pitch_z, 
			float* d_Txx, float* d_Tyy, float* d_Tzz, float* d_Txy, float* d_Txz, float* d_Tyz, int size_buf_x, int size_buf_y,
			int grid_x, int grid_y, int grid_z, int block_x, int block_y, int block_z, int position) {

#ifdef ENABLE_VERY_SLOW_ERROR_CHECKING
	hipDeviceSynchronize();
	printCudaErr(hipGetLastError(), "before computeStress kernel");
#endif
	dim3 grid_dim(grid_x,grid_y*grid_z,1);
	dim3 block_dim(block_x, block_y, block_z);
	update_halo_stress <<< grid_dim, block_dim, 0 >>> (	d_buff_x_min, d_buff_x_max, d_buff_y_min, d_buff_y_max, 
								size_x, size_y, size_z, pitch_x, pitch_y, pitch_z, 
								d_Txx, d_Tyy, d_Tzz, d_Txy, d_Txz, d_Tyz, size_buf_x, size_buf_y, position);

#ifdef ENABLE_VERY_SLOW_ERROR_CHECKING
	hipDeviceSynchronize();
	printCudaErr(hipGetLastError(), "after computeStress kernel");
#endif
}
// }}}
// }}}

// UPDATE HALOS VELOCITY {{{
// IMPLEMENTATION {{{
__global__ void update_halo_veloc(	float* d_buff_x_min, float* d_buff_x_max, float* d_buff_y_min, float* d_buff_y_max, 
					int size_x, int size_y, int size_z, int pitch_x, int pitch_y, int pitch_z, 
					float* d_Vx, float* d_Vy, float* d_Vz, int size_buf_x, int size_buf_y, int position) {

	bool active;
	int size, coord, z_coord, offset, index, size_buf;
	float* p_buf;
	bool update=true;

	// coord in X or Y direction
	coord = blockIdx.x*blockDim.x+threadIdx.x;
	// coord in Z direction
	int face = (int) (blockIdx.y/size_z);
	z_coord = blockIdx.y - (face*size_z);

	if (face < 2) {
		if (face == 0) {	// ymin
			offset = -2*pitch_x;
			p_buf = d_buff_y_min;
			if (position & MASK_FIRST_Y) update = false;
		} else {		// ymax
			offset = size_y*pitch_x;
			p_buf = d_buff_y_max;
			if (position & MASK_LAST_Y) update = false;
		}
		size = size_x;
		index = z_coord*pitch_x*pitch_y + threadIdx.y*pitch_x + offset + coord;
		size_buf = size_buf_y;
	} else {
		if (face == 2) {	// xmin
			offset = -2;
			p_buf = d_buff_x_min;
			if (position & MASK_FIRST_X) update = false;
		} else {		// xmax
			offset = size_x;
			p_buf = d_buff_x_max;
			if (position & MASK_LAST_X) update = false;
		}
		size = size_y;
		index = z_coord*pitch_x*pitch_y + coord*pitch_x + threadIdx.y + offset;
		size_buf = size_buf_x;
	}

	active = (coord<size)?true:false;

	if (active && update) {
		d_Vx[index] = p_buf[0*size_buf + z_coord*2*size + threadIdx.y*size + coord];
		d_Vy[index] = p_buf[1*size_buf + z_coord*2*size + threadIdx.y*size + coord];
		d_Vz[index] = p_buf[2*size_buf + z_coord*2*size + threadIdx.y*size + coord];
	}
	return;
}
// }}}

// WRAPPER {{{
void updateHaloVeloc(	float* d_buff_x_min, float* d_buff_x_max, float* d_buff_y_min, float* d_buff_y_max, 
			int size_x, int size_y, int size_z, int pitch_x, int pitch_y, int pitch_z, 
			float* d_Vx, float* d_Vy, float* d_Vz, int size_buf_x, int size_buf_y,
			int grid_x, int grid_y, int grid_z, int block_x, int block_y, int block_z, int position) {

#ifdef ENABLE_VERY_SLOW_ERROR_CHECKING
	hipDeviceSynchronize();
	printCudaErr(hipGetLastError(), "before computeStress kernel");
#endif
	dim3 grid_dim(grid_x,grid_y*grid_z,1);
	dim3 block_dim(block_x, block_y, block_z);
	update_halo_veloc <<< grid_dim, block_dim, 0 >>> (	d_buff_x_min, d_buff_x_max, d_buff_y_min, d_buff_y_max, 
								size_x, size_y, size_z, pitch_x, pitch_y, pitch_z, 
								d_Vx, d_Vy, d_Vz, size_buf_x, size_buf_y, position);
#ifdef ENABLE_VERY_SLOW_ERROR_CHECKING
	hipDeviceSynchronize();
	printCudaErr(hipGetLastError(), "after computeStress kernel");
#endif
}
// }}}
// }}}
// }}}
// }}}

// MISC {{{
void cuda_device_info(char* decal)
{
   int  ndev;
   hipGetDeviceCount(&ndev);
   hipDeviceSynchronize();
   printf("%sThere are %d GPUs.\n",decal,ndev);
 
   for(int i=0;i<ndev;i++) {
     hipDeviceProp_t pdev;
     hipGetDeviceProperties(&pdev,i);
     hipDeviceSynchronize();
     printf("%sName         : %s\n",decal,pdev.name);
     printf("%sCapability   : %d %d\n",decal,pdev.major,pdev.minor);
     printf("%sMemory Global: %d Mb\n",decal,(pdev.totalGlobalMem+1024*1024)/1024/1024);
     printf("%sMemory Const : %d Kb\n",decal,pdev.totalConstMem/1024);
     printf("%sMemory Shared: %d Kb\n",decal,pdev.sharedMemPerBlock/1024);
     printf("%sClock        : %.3f GHz\n",decal,pdev.clockRate/1000000.f);
     printf("%sProcessors   : %d\n",decal,pdev.multiProcessorCount);
     printf("%sCores        : %d\n",decal,8*pdev.multiProcessorCount);
     printf("%sWarp         : %d\n",decal,pdev.warpSize);
     printf("%sMax Thr/Blk  : %d\n",decal,pdev.maxThreadsPerBlock);
     printf("%sMax Blk Size : %d %d %d\n",decal,pdev.maxThreadsDim[0],pdev.maxThreadsDim[1],pdev.maxThreadsDim[2]);
     printf("%sMax Grid Size: %d %d %d\n",decal,pdev.maxGridSize[0],pdev.maxGridSize[1],pdev.maxGridSize[2]);
   }
}
// }}}

